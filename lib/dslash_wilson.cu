#include <gauge_field.h>
#include <color_spinor_field.h>
#include <dslash.h>
#include <worker.h>

#include <dslash_policy.cuh>
#include <kernels/dslash_wilson.cuh>

/**
   This is the basic gauged Wilson operator

   TODO
   - gauge fix support
   - ghost texture support in accessors
   - CPU support
*/

namespace quda
{

  /**
     @brief This is a helper class that is used to instantiate the
     correct templated kernel for the dslash.
   */
  template <typename Float, int nDim, int nColor, int nParity, bool dagger, bool xpay, KernelType kernel_type, typename Arg>
  struct WilsonLaunch {
    static constexpr const char *kernel = "quda::wilsonGPU"; // kernel name for jit compilation
    template <typename Dslash>
    inline static void launch(Dslash &dslash, TuneParam &tp, Arg &arg, const hipStream_t &stream)
    {
      dslash.launch(wilsonGPU<Float, nDim, nColor, nParity, dagger, xpay, kernel_type, Arg>, tp, arg, stream);
    }
  };

  template <typename Float, int nDim, int nColor, typename Arg> class Wilson : public Dslash<Float>
  {

protected:
    Arg &arg;
    const ColorSpinorField &in;

public:
    Wilson(Arg &arg, const ColorSpinorField &out, const ColorSpinorField &in) :
      Dslash<Float>(arg, out, in, "kernels/dslash_wilson.cuh"),
      arg(arg),
      in(in)
    {
    }

    virtual ~Wilson() {}

    void apply(const hipStream_t &stream)
    {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      Dslash<Float>::setParam(arg);
      Dslash<Float>::template instantiate<WilsonLaunch, nDim, nColor>(tp, arg, stream);
    }

    TuneKey tuneKey() const
    {
      return TuneKey(in.VolString(), typeid(*this).name(), Dslash<Float>::aux[arg.kernel_type]);
    }
  };

  template <typename Float, int nColor, QudaReconstructType recon> struct WilsonApply {

    inline WilsonApply(ColorSpinorField &out, const ColorSpinorField &in, const GaugeField &U, double a,
                       const ColorSpinorField &x, int parity, bool dagger, const int *comm_override, TimeProfile &profile)
    {
      constexpr int nDim = 4;
      WilsonArg<Float, nColor, recon> arg(out, in, U, a, x, parity, dagger, comm_override);
      Wilson<Float, nDim, nColor, WilsonArg<Float, nColor, recon>> wilson(arg, out, in);

      dslash::DslashPolicyTune<decltype(wilson)> policy(
        wilson, const_cast<cudaColorSpinorField *>(static_cast<const cudaColorSpinorField *>(&in)), in.VolumeCB(),
        in.GhostFaceCB(), profile);
      policy.apply(0);

      checkCudaError();
    }
  };

  // Apply the Wilson operator
  // out(x) = M*in = - a*\sum_mu U_{-\mu}(x)in(x+mu) + U^\dagger_mu(x-mu)in(x-mu)
  // Uses the a normalization for the Wilson operator.
  void ApplyWilson(ColorSpinorField &out, const ColorSpinorField &in, const GaugeField &U, double a,
                   const ColorSpinorField &x, int parity, bool dagger, const int *comm_override, TimeProfile &profile)
  {
#ifdef GPU_WILSON_DIRAC
    if (in.V() == out.V()) errorQuda("Aliasing pointers");
    if (in.FieldOrder() != out.FieldOrder())
      errorQuda("Field order mismatch in = %d, out = %d", in.FieldOrder(), out.FieldOrder());

    // check all precisions match
    checkPrecision(out, in, U);

    // check all locations match
    checkLocation(out, in, U);

    instantiate<WilsonApply, WilsonReconstruct>(out, in, U, a, x, parity, dagger, comm_override, profile);
#else
    errorQuda("Wilson dslash has not been built");
#endif // GPU_WILSON_DIRAC
  }

} // namespace quda
