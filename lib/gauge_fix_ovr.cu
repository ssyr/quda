#include "hip/hip_runtime.h"
#include <quda_internal.h>
#include <quda_matrix.h>
#include <tune_quda.h>
#include <gauge_field.h>
#include <gauge_field_order.h>
#include <launch_kernel.cuh>
#include <unitarization_links.h>
#include <comm_quda.h>
#include <gauge_fix_ovr_extra.h>
#include <gauge_fix_ovr_hit_devf.cuh>
#include <reduce_helper.h>
#include <index_helper.cuh>
#include <instantiate.h>

namespace quda {

#define LAUNCH_KERNEL_GAUGEFIX(kernel, tp, stream, arg, parity, ...)                                                   \
  if (tp.aux.x == 0) {                                                                                                 \
    switch (tp.block.x) {                                                                                              \
    case 256: qudaLaunchKernel(kernel<0, 32, __VA_ARGS__>, tp, stream, arg, parity); break;      \
    case 512: qudaLaunchKernel(kernel<0, 64, __VA_ARGS__>, tp, stream, arg, parity); break;      \
    case 768: qudaLaunchKernel(kernel<0, 96, __VA_ARGS__>, tp, stream, arg, parity); break;      \
    case 1024: qudaLaunchKernel(kernel<0, 128, __VA_ARGS__>, tp, stream, arg, parity); break;    \
    default: errorQuda("%s not implemented for %d threads", #kernel, tp.block.x);                                      \
    }                                                                                                                  \
  } else if (tp.aux.x == 1) {                                                                                          \
    switch (tp.block.x) {                                                                                              \
    case 256: qudaLaunchKernel(kernel<1, 32, __VA_ARGS__>, tp, stream, arg, parity); break;      \
    case 512: qudaLaunchKernel(kernel<1, 64, __VA_ARGS__>, tp, stream, arg, parity); break;      \
    case 768: qudaLaunchKernel(kernel<1, 96, __VA_ARGS__>, tp, stream, arg, parity); break;      \
    case 1024: qudaLaunchKernel(kernel<1, 128, __VA_ARGS__>, tp, stream, arg, parity); break;    \
    default: errorQuda("%s not implemented for %d threads", #kernel, tp.block.x);                                      \
    }                                                                                                                  \
  } else if (tp.aux.x == 2) {                                                                                          \
    switch (tp.block.x) {                                                                                              \
    case 256: qudaLaunchKernel(kernel<2, 32, __VA_ARGS__>, tp, stream, arg, parity); break;      \
    case 512: qudaLaunchKernel(kernel<2, 64, __VA_ARGS__>, tp, stream, arg, parity); break;      \
    case 768: qudaLaunchKernel(kernel<2, 96, __VA_ARGS__>, tp, stream, arg, parity); break;      \
    case 1024: qudaLaunchKernel(kernel<2, 128, __VA_ARGS__>, tp, stream, arg, parity); break;    \
    default: errorQuda("%s not implemented for %d threads", #kernel, tp.block.x);                                      \
    }                                                                                                                  \
  } else if (tp.aux.x == 3) {                                                                                          \
    switch (tp.block.x) {                                                                                              \
    case 128: qudaLaunchKernel(kernel<3, 32, __VA_ARGS__>, tp, stream, arg, parity); break;      \
    case 256: qudaLaunchKernel(kernel<3, 64, __VA_ARGS__>, tp, stream, arg, parity); break;      \
    case 384: qudaLaunchKernel(kernel<3, 96, __VA_ARGS__>, tp, stream, arg, parity); break;      \
    case 512: qudaLaunchKernel(kernel<3, 128, __VA_ARGS__>, tp, stream, arg, parity); break;     \
    case 640: qudaLaunchKernel(kernel<3, 160, __VA_ARGS__>, tp, stream, arg, parity); break;     \
    case 768: qudaLaunchKernel(kernel<3, 192, __VA_ARGS__>, tp, stream, arg, parity); break;     \
    case 896: qudaLaunchKernel(kernel<3, 224, __VA_ARGS__>, tp, stream, arg, parity); break;     \
    case 1024: qudaLaunchKernel(kernel<3, 256, __VA_ARGS__>, tp, stream, arg, parity); break;    \
    default: errorQuda("%s not implemented for %d threads", #kernel, tp.block.x);                                      \
    }                                                                                                                  \
  } else if (tp.aux.x == 4) {                                                                                          \
    switch (tp.block.x) {                                                                                              \
    case 128: qudaLaunchKernel(kernel<4, 32, __VA_ARGS__>, tp, stream, arg, parity); break;      \
    case 256: qudaLaunchKernel(kernel<4, 64, __VA_ARGS__>, tp, stream, arg, parity); break;      \
    case 384: qudaLaunchKernel(kernel<4, 96, __VA_ARGS__>, tp, stream, arg, parity); break;      \
    case 512: qudaLaunchKernel(kernel<4, 128, __VA_ARGS__>, tp, stream, arg, parity); break;     \
    case 640: qudaLaunchKernel(kernel<4, 160, __VA_ARGS__>, tp, stream, arg, parity); break;     \
    case 768: qudaLaunchKernel(kernel<4, 192, __VA_ARGS__>, tp, stream, arg, parity); break;     \
    case 896: qudaLaunchKernel(kernel<4, 224, __VA_ARGS__>, tp, stream, arg, parity); break;     \
    case 1024: qudaLaunchKernel(kernel<4, 256, __VA_ARGS__>, tp, stream, arg, parity); break;    \
    default: errorQuda("%s not implemented for %d threads", #kernel, tp.block.x);                                      \
    }                                                                                                                  \
  } else if (tp.aux.x == 5) {                                                                                          \
    switch (tp.block.x) {                                                                                              \
    case 128: qudaLaunchKernel(kernel<5, 32, __VA_ARGS__>, tp, stream, arg, parity); break;      \
    case 256: qudaLaunchKernel(kernel<5, 64, __VA_ARGS__>, tp, stream, arg, parity); break;      \
    case 384: qudaLaunchKernel(kernel<5, 96, __VA_ARGS__>, tp, stream, arg, parity); break;      \
    case 512: qudaLaunchKernel(kernel<5, 128, __VA_ARGS__>, tp, stream, arg, parity); break;     \
    case 640: qudaLaunchKernel(kernel<5, 160, __VA_ARGS__>, tp, stream, arg, parity); break;     \
    case 768: qudaLaunchKernel(kernel<5, 192, __VA_ARGS__>, tp, stream, arg, parity); break;     \
    case 896: qudaLaunchKernel(kernel<5, 224, __VA_ARGS__>, tp, stream, arg, parity); break;     \
    case 1024: qudaLaunchKernel(kernel<5, 256, __VA_ARGS__>, tp, stream, arg, parity); break;    \
    default: errorQuda("%s not implemented for %d threads", #kernel, tp.block.x);                                      \
    }                                                                                                                  \
  } else {                                                                                                             \
    errorQuda("Not implemented for %d", tp.aux.x);                                                                     \
  }

  /**
   * @brief container to pass parameters for the gauge fixing quality kernel
   */
  template <typename Gauge>
  struct GaugeFixQualityArg : public ReduceArg<double2> {
    int threads; // number of active threads required
    int X[4]; // grid dimensions
#ifdef MULTI_GPU
    int border[4];
#endif
    Gauge dataOr;
    double2 result;
    GaugeFixQualityArg(const Gauge &dataOr, const GaugeField &data)
      : ReduceArg<double2>(), dataOr(dataOr) {

      for ( int dir = 0; dir < 4; ++dir ) {
        X[dir] = data.X()[dir] - data.R()[dir] * 2;
      #ifdef MULTI_GPU
        border[dir] = data.R()[dir];
      #endif
      }
      threads = X[0]*X[1]*X[2]*X[3]/2;
    }
    double getAction(){ return result.x; }
    double getTheta(){ return result.y; }
  };

  /**
   * @brief Measure gauge fixing quality
   */
  template<int blockSize, typename Float, typename Gauge, int gauge_dir>
  __global__ void computeFix_quality(GaugeFixQualityArg<Gauge> argQ){
    typedef complex<Float> Cmplx;

    int idx_cb = threadIdx.x + blockIdx.x * blockDim.x;
    int parity = threadIdx.y;

    double2 data = make_double2(0.0,0.0);
    while (idx_cb < argQ.threads) {
      int X[4];
#pragma unroll
      for ( int dr = 0; dr < 4; ++dr ) X[dr] = argQ.X[dr];

      int x[4];
      getCoords(x, idx_cb, X, parity);
#ifdef MULTI_GPU
    #pragma unroll
      for ( int dr = 0; dr < 4; ++dr ) {
        x[dr] += argQ.border[dr];
        X[dr] += 2 * argQ.border[dr];
      }
#endif
      Matrix<Cmplx,3> delta;
      setZero(&delta);
      //load upward links
      for ( int mu = 0; mu < gauge_dir; mu++ ) {
        Matrix<Cmplx,3> U = argQ.dataOr(mu, linkIndex(x, X), parity);
        delta -= U;
      }
      //18*gauge_dir
      data.x += -delta(0, 0).x - delta(1, 1).x - delta(2, 2).x;
      //2
      //load downward links
      for ( int mu = 0; mu < gauge_dir; mu++ ) {
        Matrix<Cmplx,3> U = argQ.dataOr(mu, linkIndexM1(x,X,mu), 1 - parity);
        delta += U;
      }
      //18*gauge_dir
      delta -= conj(delta);
      //18
      SubTraceUnit(delta);
      //12
      data.y += getRealTraceUVdagger(delta, delta);
      //35
      //T=36*gauge_dir+65

      idx_cb += blockDim.x * gridDim.x;
    }
    argQ.template reduce2d<blockSize,2>(data);
  }

  /**
   * @brief Tunable object for the gauge fixing quality kernel
   */
  template<typename Float, typename Gauge, int gauge_dir>
  class GaugeFixQuality : TunableLocalParityReduction {
    GaugeFixQualityArg<Gauge> &arg;
    const GaugeField &meta;

  public:
    GaugeFixQuality(GaugeFixQualityArg<Gauge> &arg, const GaugeField &meta) :
      arg(arg),
      meta(meta)
    { }

    void apply(const qudaStream_t &stream)
    {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      LAUNCH_KERNEL_LOCAL_PARITY(computeFix_quality, (*this), tp, stream, arg, Float, Gauge, gauge_dir);
      auto reset = true; // apply is called multiple times with the same arg instance so we need to reset
      arg.complete(arg.result, stream, reset);
      if (!activeTuning()) {
        comm_allreduce_array((double*)&arg.result, 2);
        arg.result.x /= (double)(3 * gauge_dir * 2 * arg.threads * comm_size());
        arg.result.y /= (double)(3 * 2 * arg.threads * comm_size());
      }
    }

    TuneKey tuneKey() const { return TuneKey(meta.VolString(), typeid(*this).name(), meta.AuxString()); }
    long long flops() const { return (36LL * gauge_dir + 65LL) * meta.Volume(); }
    //long long bytes() const { return (1)*2*gauge_dir*arg.Bytes(); }
    long long bytes() const { return 2LL * gauge_dir * meta.Volume() * meta.Reconstruct() * sizeof(Float); }
  };

  /**
   * @brief container to pass parameters for the gauge fixing kernel
   */
  template <typename Float, typename Gauge>
  struct GaugeFixArg {
    int threads; // number of active threads required
    int X[4]; // grid dimensions
#ifdef MULTI_GPU
    int border[4];
#endif
    Gauge dataOr;
    GaugeField &data;
    const Float relax_boost;

    GaugeFixArg(Gauge & dataOr, GaugeField & data, const Float relax_boost)
      : dataOr(dataOr), data(data), relax_boost(relax_boost) {

      for ( int dir = 0; dir < 4; ++dir ) {
        X[dir] = data.X()[dir] - data.R()[dir] * 2;
      #ifdef MULTI_GPU
        border[dir] = data.R()[dir];
      #endif
      }
      threads = X[0] * X[1] * X[2] * X[3] >> 1;
    }
  };


  /**
   * @brief Kernel to perform gauge fixing with overrelaxation for single-GPU
   */
  template<int ImplementationType, int blockSize, typename Float, typename Gauge, int gauge_dir>
  __global__ void computeFix(GaugeFixArg<Float, Gauge> arg, int parity)
  {
    typedef complex<Float> Cmplx;
    int tid = (threadIdx.x + blockSize) % blockSize;
    int idx = blockIdx.x * blockSize + tid;

    if ( idx >= arg.threads ) return;

    // 8 threads per lattice site
    if ( ImplementationType < 3 ) {
      int X[4];
    #pragma unroll
      for ( int dr = 0; dr < 4; ++dr ) X[dr] = arg.X[dr];

      int x[4];
      getCoords(x, idx, X, parity);
  #ifdef MULTI_GPU
    #pragma unroll
      for ( int dr = 0; dr < 4; ++dr ) {
        x[dr] += arg.border[dr];
        X[dr] += 2 * arg.border[dr];
      }
  #endif
      int mu = (threadIdx.x / blockSize);
      int oddbit = parity;
      if ( threadIdx.x >= blockSize * 4 ) {
        mu -= 4;
        x[mu] = (x[mu] - 1 + X[mu]) % X[mu];
        oddbit = 1 - parity;
      }
      idx = (((x[3] * X[2] + x[2]) * X[1] + x[1]) * X[0] + x[0]) >> 1;
      Matrix<Cmplx,3> link = arg.dataOr(mu, idx, oddbit);
      // 8 treads per lattice site, the reduction is performed by shared memory without using atomicadd.
      // this implementation needs 8x more shared memory than the implementation using atomicadd
      if ( ImplementationType == 0 ) GaugeFixHit_NoAtomicAdd<blockSize, Float, gauge_dir, 3>(link, arg.relax_boost, tid);
      // 8 treads per lattice site, the reduction is performed by shared memory using atomicadd
      if ( ImplementationType == 1 ) GaugeFixHit_AtomicAdd<blockSize, Float, gauge_dir, 3>(link, arg.relax_boost, tid);
      // 8 treads per lattice site, the reduction is performed by shared memory without using atomicadd.
      // uses the same amount of shared memory as the atomicadd implementation with more thread block synchronization
      if ( ImplementationType == 2 ) GaugeFixHit_NoAtomicAdd_LessSM<blockSize, Float, gauge_dir, 3>(link, arg.relax_boost, tid);
      arg.dataOr(mu, idx, oddbit) = link;
    }
    // 4 threads per lattice site
    else{
      int X[4];
    #pragma unroll
      for ( int dr = 0; dr < 4; ++dr ) X[dr] = arg.X[dr];

      int x[4];
      getCoords(x, idx, X, parity);
  #ifdef MULTI_GPU
    #pragma unroll
      for ( int dr = 0; dr < 4; ++dr ) {
        x[dr] += arg.border[dr];
        X[dr] += 2 * arg.border[dr];
      }
  #endif
      int mu = (threadIdx.x / blockSize);
      idx = (((x[3] * X[2] + x[2]) * X[1] + x[1]) * X[0] + x[0]) >> 1;
      //load upward link
      Matrix<Cmplx,3> link = arg.dataOr(mu, idx, parity);

      x[mu] = (x[mu] - 1 + X[mu]) % X[mu];
      int idx1 = (((x[3] * X[2] + x[2]) * X[1] + x[1]) * X[0] + x[0]) >> 1;
      //load downward link
      Matrix<Cmplx,3> link1 = arg.dataOr(mu, idx1, 1 - parity);

      // 4 treads per lattice site, the reduction is performed by shared memory without using atomicadd.
      // this implementation needs 4x more shared memory than the implementation using atomicadd
      if ( ImplementationType == 3 ) GaugeFixHit_NoAtomicAdd<blockSize, Float, gauge_dir, 3>(link, link1, arg.relax_boost, tid);
      // 4 treads per lattice site, the reduction is performed by shared memory using atomicadd
      if ( ImplementationType == 4 ) GaugeFixHit_AtomicAdd<blockSize, Float, gauge_dir, 3>(link, link1, arg.relax_boost, tid);
      // 4 treads per lattice site, the reduction is performed by shared memory without using atomicadd.
      // uses the same amount of shared memory as the atomicadd implementation with more thread block synchronization
      if ( ImplementationType == 5 ) GaugeFixHit_NoAtomicAdd_LessSM<blockSize, Float, gauge_dir, 3>(link, link1, arg.relax_boost, tid);

      arg.dataOr(mu, idx, parity) = link;
      arg.dataOr(mu, idx1, 1 - parity) = link1;
    }
  }

  /**
   * @brief Tunable object for the gauge fixing kernel
   */
  template<typename Float, typename Gauge, int gauge_dir>
  class GaugeFix : Tunable {
    GaugeFixArg<Float, Gauge> &arg;
    const GaugeField &meta;
    int parity;

    dim3 createGrid(const TuneParam &param) const
    {
      unsigned int blockx = param.block.x / 8;
      if (param.aux.x > 2) blockx = param.block.x / 4;
      unsigned int gx  = std::max((arg.threads + blockx - 1) / blockx, 1u);
      return dim3(gx, 1, 1);
    }

    bool advanceBlockDim (TuneParam &param) const
    {
      // Use param.aux.x to tune and save state for best kernel option
      // to make use or not of atomicAdd operations and 4 or 8 threads per lattice site!!!
      const unsigned int min_threads0 = 32 * 8;
      const unsigned int min_threads1 = 32 * 4;
      const unsigned int max_threads = 1024; // FIXME: use deviceProp.maxThreadsDim[0];
      const unsigned int atmadd = 0;
      unsigned int min_threads = min_threads0;
      param.aux.x += atmadd; // USE TO SELECT BEST KERNEL OPTION WITH/WITHOUT USING ATOMICADD
      if (param.aux.x > 2) min_threads = 32 * 4;
      param.block.x += min_threads;
      param.block.y = 1;
      param.grid = createGrid(param);

      if ((param.block.x >= min_threads) && (param.block.x <= max_threads)) {
        param.shared_bytes = sharedBytesPerBlock(param);
        return true;
      } else if (param.aux.x == 0) {
        param.block.x = min_threads0;
        param.block.y = 1;
        param.aux.x = 1; // USE FOR ATOMIC ADD
        param.grid = createGrid(param);
        param.shared_bytes = param.block.x * 4 * sizeof(Float) / 8;
        return true;
      } else if (param.aux.x == 1) {
        param.block.x = min_threads0;
        param.block.y = 1;
        param.aux.x = 2; // USE FOR NO ATOMIC ADD and LESS SHARED MEM
        param.grid = createGrid(param);
        param.shared_bytes = param.block.x * 4 * sizeof(Float) / 8;
        return true;
      } else if (param.aux.x == 2) {
        param.block.x = min_threads1;
        param.block.y = 1;
        param.aux.x = 3; // USE FOR NO ATOMIC ADD
        param.grid = createGrid(param);
        param.shared_bytes = param.block.x * 4 * sizeof(Float);
        return true;
      } else if (param.aux.x == 3) {
        param.block.x = min_threads1;
        param.block.y = 1;
        param.aux.x = 4;
        param.grid = createGrid(param);
        param.shared_bytes = param.block.x * sizeof(Float);
        return true;
      } else if (param.aux.x == 4) {
        param.block.x = min_threads1;
        param.block.y = 1;
        param.aux.x = 5;
        param.grid = createGrid(param);
        param.shared_bytes = param.block.x * sizeof(Float);
        return true;
      } else {
        return false;
      }
    }

    unsigned int sharedBytesPerThread() const { return 0; }
    unsigned int sharedBytesPerBlock(const TuneParam &param) const {
      switch (param.aux.x) {
      case 0: return param.block.x * 4 * sizeof(Float);
      case 1: return param.block.x * 4 * sizeof(Float) / 8;
      case 2: return param.block.x * 4 * sizeof(Float) / 8;
      case 3: return param.block.x * 4 * sizeof(Float);
      default: return param.block.x * sizeof(Float);
      }
    }

    bool tuneSharedBytes() const { return false; }
    bool tuneGridDim() const { return false; }
    unsigned int minThreads() const { return arg.threads; }

public:
    GaugeFix(GaugeFixArg<Float, Gauge> &arg, const GaugeField &meta) :
      arg(arg),
      meta(meta),
      parity(0) { }

    void setParity(const int par) { parity = par; }

    void apply(const qudaStream_t &stream){
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      LAUNCH_KERNEL_GAUGEFIX(computeFix, tp, stream, arg, parity, Float, Gauge, gauge_dir);
    }

    virtual void initTuneParam(TuneParam &param) const
    {
      param.block = dim3(256, 1, 1);
      param.aux.x = 0;
      param.grid = createGrid(param);
      param.shared_bytes = sharedBytesPerBlock(param);
    }

    virtual void defaultTuneParam(TuneParam &param) const { initTuneParam(param); }

    TuneKey tuneKey() const { return TuneKey(meta.VolString(), typeid(*this).name(), meta.AuxString()); }

    std::string paramString(const TuneParam &param) const {
      std::stringstream ps(Tunable::paramString(param));
      ps << ", atomicadd=" << param.aux.x;
      return ps.str();
    }

    void preTune() { arg.data.backup(); }
    void postTune() { arg.data.restore(); }
    long long flops() const { return 3LL * (22 + 28 * gauge_dir + 224 * 3) * arg.threads; }
    long long bytes() const { return 8LL * 2 * arg.threads * meta.Reconstruct() * sizeof(Float);  }
  };

#ifdef MULTI_GPU
  template <typename Float, typename Gauge>
  struct GaugeFixInteriorPointsArg {
    int threads; // number of active threads required
    int X[4]; // grid dimensions
#ifdef MULTI_GPU
    int border[4];
#endif
    Gauge dataOr;
    GaugeField &data;
    const Float relax_boost;
    GaugeFixInteriorPointsArg(Gauge & dataOr, GaugeField & data, const Float relax_boost)
      : dataOr(dataOr), data(data), relax_boost(relax_boost) {

#ifdef MULTI_GPU
      for ( int dir = 0; dir < 4; ++dir ) {
        if ( comm_dim_partitioned(dir)) border[dir] = data.R()[dir] + 1;  //skip BORDER_RADIUS + face border point
        else border[dir] = 0;
      }
      for ( int dir = 0; dir < 4; ++dir ) X[dir] = data.X()[dir] - border[dir] * 2;
#else
      for ( int dir = 0; dir < 4; ++dir ) X[dir] = data.X()[dir];
#endif
      threads = X[0] * X[1] * X[2] * X[3] >> 1;
      if (this->threads == 0) errorQuda("Local volume is too small");
    }
  };


  /**
   * @brief Kernel to perform gauge fixing with overrelaxation in the interior points for multi-GPU implementation
   */
  template<int ImplementationType, int blockSize, typename Float, typename Gauge, int gauge_dir>
  __global__ void computeFixInteriorPoints(GaugeFixInteriorPointsArg<Float, Gauge> arg, int parity){
    int tid = (threadIdx.x + blockSize) % blockSize;
    int idx = blockIdx.x * blockSize + tid;
    if ( idx >= arg.threads ) return;
    typedef complex<Float> Complex;
    int X[4];
#pragma unroll
    for ( int dr = 0; dr < 4; ++dr ) X[dr] = arg.X[dr];
    int x[4];
#ifdef MULTI_GPU
    int za = (idx / (X[0] / 2));
    int zb =  (za / X[1]);
    x[1] = za - zb * X[1];
    x[3] = (zb / X[2]);
    x[2] = zb - x[3] * X[2];
    int p = 0; for ( int dr = 0; dr < 4; ++dr ) p += arg.border[dr];
    p = p & 1;
    int x1odd = (x[1] + x[2] + x[3] + parity + p) & 1;
    //int x1odd = (x[1] + x[2] + x[3] + parity) & 1;
    x[0] = (2 * idx + x1odd)  - za * X[0];
    for ( int dr = 0; dr < 4; ++dr ) {
      x[dr] += arg.border[dr];
      X[dr] += 2 * arg.border[dr];
    }
#else
    getCoords(x, idx, X, parity);
#endif
    int mu = (threadIdx.x / blockSize);

    // 8 threads per lattice site
    if ( ImplementationType < 3 ) {
      if ( threadIdx.x >= blockSize * 4 ) {
        mu -= 4;
        x[mu] = (x[mu] - 1 + X[mu]) % X[mu];
        parity = 1 - parity;
      }
      idx = (((x[3] * X[2] + x[2]) * X[1] + x[1]) * X[0] + x[0]) >> 1;
      Matrix<Complex,3> link = arg.dataOr(mu, idx, parity);
      // 8 treads per lattice site, the reduction is performed by shared memory without using atomicadd.
      // this implementation needs 8x more shared memory than the implementation using atomicadd
      if ( ImplementationType == 0 ) GaugeFixHit_NoAtomicAdd<blockSize, Float, gauge_dir, 3>(link, arg.relax_boost, tid);
      // 8 treads per lattice site, the reduction is performed by shared memory using atomicadd
      if ( ImplementationType == 1 ) GaugeFixHit_AtomicAdd<blockSize, Float, gauge_dir, 3>(link, arg.relax_boost, tid);
      // 8 treads per lattice site, the reduction is performed by shared memory without using atomicadd.
      // uses the same amount of shared memory as the atomicadd implementation with more thread block synchronization
      if ( ImplementationType == 2 ) GaugeFixHit_NoAtomicAdd_LessSM<blockSize, Float, gauge_dir, 3>(link, arg.relax_boost, tid);
      arg.dataOr(mu, idx, parity) = link;
    }
    // 4 threads per lattice site
    else{
      idx = (((x[3] * X[2] + x[2]) * X[1] + x[1]) * X[0] + x[0]) >> 1;
      Matrix<Complex,3> link = arg.dataOr(mu, idx, parity);


      x[mu] = (x[mu] - 1 + X[mu]) % X[mu];
      int idx1 = (((x[3] * X[2] + x[2]) * X[1] + x[1]) * X[0] + x[0]) >> 1;
      Matrix<Complex,3> link1 = arg.dataOr(mu, idx1, 1 - parity);

      // 4 treads per lattice site, the reduction is performed by shared memory without using atomicadd.
      // this implementation needs 4x more shared memory than the implementation using atomicadd
      if ( ImplementationType == 3 ) GaugeFixHit_NoAtomicAdd<blockSize, Float, gauge_dir, 3>(link, link1, arg.relax_boost, tid);
      // 4 treads per lattice site, the reduction is performed by shared memory using atomicadd
      if ( ImplementationType == 4 ) GaugeFixHit_AtomicAdd<blockSize, Float, gauge_dir, 3>(link, link1, arg.relax_boost, tid);
      // 4 treads per lattice site, the reduction is performed by shared memory without using atomicadd.
      // uses the same amount of shared memory as the atomicadd implementation with more thread block synchronization
      if ( ImplementationType == 5 ) GaugeFixHit_NoAtomicAdd_LessSM<blockSize, Float, gauge_dir, 3>(link, link1, arg.relax_boost, tid);

      arg.dataOr(mu, idx, parity) = link;
      arg.dataOr(mu, idx1, 1 - parity) = link1;
    }
  }

  /**
   * @brief Tunable object for the interior points of the gauge fixing
   * kernel in multi-GPU implementation
   */
  template<typename Float, typename Gauge, int gauge_dir>
  class GaugeFixInteriorPoints : Tunable {
    GaugeFixInteriorPointsArg<Float, Gauge> &arg;
    const GaugeField &meta;
    int parity;

    dim3 createGrid(const TuneParam &param) const
    {
      unsigned int blockx = param.block.x / 8;
      if (param.aux.x > 2) blockx = param.block.x / 4;
      unsigned int gx  = (arg.threads + blockx - 1) / blockx;
      return dim3(gx, 1, 1);
    }

    bool advanceBlockDim(TuneParam &param) const
    {
      // Use param.aux.x to tune and save state for best kernel option
      // to make use or not of atomicAdd operations and 4 or 8 threads per lattice site!!!
      const unsigned int min_threads0 = 32 * 8;
      const unsigned int min_threads1 = 32 * 4;
      const unsigned int max_threads = 1024; // FIXME: use deviceProp.maxThreadsDim[0];
      const unsigned int atmadd = 0;
      unsigned int min_threads = min_threads0;
      param.aux.x += atmadd; // USE TO SELECT BEST KERNEL OPTION WITH/WITHOUT USING ATOMICADD
      if (param.aux.x > 2) min_threads = 32 * 4;
      param.block.x += min_threads;
      param.block.y = 1;
      param.grid = createGrid(param);

      if ((param.block.x >= min_threads) && (param.block.x <= max_threads)) {
        param.shared_bytes = sharedBytesPerBlock(param);
        return true;
      } else if (param.aux.x == 0) {
        param.block.x = min_threads0;
        param.block.y = 1;
        param.aux.x = 1; // USE FOR ATOMIC ADD
        param.grid = createGrid(param);
        param.shared_bytes = param.block.x * 4 * sizeof(Float) / 8;
        return true;
      } else if (param.aux.x == 1) {
        param.block.x = min_threads0;
        param.block.y = 1;
        param.aux.x = 2; // USE FOR NO ATOMIC ADD and LESS SHARED MEM
        param.grid = createGrid(param);
        param.shared_bytes = param.block.x * 4 * sizeof(Float) / 8;
        return true;
      } else if (param.aux.x == 2) {
        param.block.x = min_threads1;
        param.block.y = 1;
        param.aux.x = 3; // USE FOR NO ATOMIC ADD
        param.grid = createGrid(param);
        param.shared_bytes = param.block.x * 4 * sizeof(Float);
        return true;
      } else if (param.aux.x == 3) {
        param.block.x = min_threads1;
        param.block.y = 1;
        param.aux.x = 4;
        param.grid = createGrid(param);
        param.shared_bytes = param.block.x * sizeof(Float);
        return true;
      } else if (param.aux.x == 4) {
        param.block.x = min_threads1;
        param.block.y = 1;
        param.aux.x = 5;
        param.grid = createGrid(param);
        param.shared_bytes = param.block.x * sizeof(Float);
        return true;
      } else {
        return false;
      }
    }

    unsigned int sharedBytesPerThread() const { return 0; }
    unsigned int sharedBytesPerBlock(const TuneParam &param) const {
      switch (param.aux.x) {
      case 0: return param.block.x * 4 * sizeof(Float);
      case 1: return param.block.x * 4 * sizeof(Float) / 8;
      case 2: return param.block.x * 4 * sizeof(Float) / 8;
      case 3: return param.block.x * 4 * sizeof(Float);
      default: return param.block.x * sizeof(Float);
      }
    }

    bool tuneSharedBytes() const { return false; }
    bool tuneGridDim() const { return false; }
    unsigned int minThreads() const { return arg.threads; }

public:
    GaugeFixInteriorPoints(GaugeFixInteriorPointsArg<Float, Gauge> &arg, const GaugeField &meta) :
      arg(arg),
      meta(meta),
      parity(0) {}

    void setParity(const int par) { parity = par; }

    void apply(const qudaStream_t &stream)
    {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      LAUNCH_KERNEL_GAUGEFIX(computeFixInteriorPoints, tp, stream, arg, parity, Float, Gauge, gauge_dir);
    }

    virtual void initTuneParam(TuneParam &param) const
    {
      param.block = dim3(256, 1, 1);
      param.aux.x = 0;
      param.grid = createGrid(param);
      param.shared_bytes = sharedBytesPerBlock(param);
    }

    virtual void defaultTuneParam(TuneParam &param) const { initTuneParam(param); }

    TuneKey tuneKey() const { return TuneKey(meta.VolString(), typeid(*this).name(), meta.AuxString()); }

    std::string paramString(const TuneParam &param) const {
      std::stringstream ps(Tunable::paramString(param));
      ps << ", atomicadd=" << param.aux.x;
      return ps.str();
    }

    void preTune() { arg.data.backup(); }
    void postTune() { arg.data.restore(); }
    long long flops() const { return 3LL * (22 + 28 * gauge_dir + 224 * 3) * arg.threads; }
    long long bytes() const { return 8LL * 2 * arg.threads * meta.Reconstruct() * sizeof(Float); }
  };

  template <typename Float, typename Gauge>
  struct GaugeFixBorderPointsArg {
    int threads; // number of active threads required
    int X[4]; // grid dimensions
    int border[4];
    int *borderpoints[2];
    int *faceindicessize[2];
    size_t faceVolume[4];
    size_t faceVolumeCB[4];
    Gauge dataOr;
    GaugeField &data;
    const Float relax_boost;

    GaugeFixBorderPointsArg(Gauge & dataOr, GaugeField & data, const Float relax_boost, size_t faceVolume_[4], size_t faceVolumeCB_[4])
      : dataOr(dataOr), data(data), relax_boost(relax_boost)
    {
      for ( int dir = 0; dir < 4; ++dir ) {
        X[dir] = data.X()[dir] - data.R()[dir] * 2;
        border[dir] = data.R()[dir];
      }

      /*for(int dir=0; dir<4; ++dir){
         if(comm_dim_partitioned(dir)) border[dir] = BORDER_RADIUS;
         else border[dir] = 0;
         }
         for(int dir=0; dir<4; ++dir) X[dir] = data.X()[dir] - border[dir]*2;*/
      for ( int dir = 0; dir < 4; ++dir ) {
        faceVolume[dir] = faceVolume_[dir];
        faceVolumeCB[dir] = faceVolumeCB_[dir];
      }
      if ( comm_partitioned() ) PreCalculateLatticeIndices(faceVolume, faceVolumeCB, X, border, threads, borderpoints);
    }
  };

  /**
   * @brief Kernel to perform gauge fixing with overrelaxation in the border points for multi-GPU implementation
  */
  template<int ImplementationType, int blockSize, typename Float, typename Gauge, int gauge_dir>
  __global__ void computeFixBorderPoints(GaugeFixBorderPointsArg<Float, Gauge> arg, int parity){
    typedef complex<Float> Cmplx;

    int tid = (threadIdx.x + blockSize) % blockSize;
    int idx = blockIdx.x * blockSize + tid;
    if ( idx >= arg.threads ) return;
    int mu = (threadIdx.x / blockSize);
    idx = arg.borderpoints[parity][idx];
    int X[4], x[4];
    x[3] = idx / (arg.X[0] * arg.X[1]  * arg.X[2]);
    x[2] = (idx / (arg.X[0] * arg.X[1])) % arg.X[2];
    x[1] = (idx / arg.X[0]) % arg.X[1];
    x[0] = idx % arg.X[0];
  #pragma unroll
    for ( int dr = 0; dr < 4; ++dr ) x[dr] += arg.border[dr];
  #pragma unroll
    for ( int dr = 0; dr < 4; ++dr ) X[dr] = arg.X[dr] + 2 * arg.border[dr];

    // 8 threads per lattice site
    if ( ImplementationType < 3 ) {
      if ( threadIdx.x >= blockSize * 4 ) {
        mu -= 4;
        x[mu] = (x[mu] - 1 + X[mu]) % X[mu];
        parity = 1 - parity;
      }
      idx = (((x[3] * X[2] + x[2]) * X[1] + x[1]) * X[0] + x[0]) >> 1;
      Matrix<Cmplx,3> link = arg.dataOr(mu, idx, parity);
      // 8 treads per lattice site, the reduction is performed by shared memory without using atomicadd.
      // this implementation needs 8x more shared memory than the implementation using atomicadd
      if ( ImplementationType == 0 ) GaugeFixHit_NoAtomicAdd<blockSize, Float, gauge_dir, 3>(link, arg.relax_boost, tid);
      // 8 treads per lattice site, the reduction is performed by shared memory using atomicadd
      if ( ImplementationType == 1 ) GaugeFixHit_AtomicAdd<blockSize, Float, gauge_dir, 3>(link, arg.relax_boost, tid);
      // 8 treads per lattice site, the reduction is performed by shared memory without using atomicadd.
      // uses the same amount of shared memory as the atomicadd implementation with more thread block synchronization
      if ( ImplementationType == 2 ) GaugeFixHit_NoAtomicAdd_LessSM<blockSize, Float, gauge_dir, 3>(link, arg.relax_boost, tid);
      arg.dataOr(mu, idx, parity) = link;
    }
    // 4 threads per lattice site
    else{
      idx = (((x[3] * X[2] + x[2]) * X[1] + x[1]) * X[0] + x[0]) >> 1;
      Matrix<Cmplx,3> link = arg.dataOr(mu, idx, parity);


      x[mu] = (x[mu] - 1 + X[mu]) % X[mu];
      int idx1 = (((x[3] * X[2] + x[2]) * X[1] + x[1]) * X[0] + x[0]) >> 1;
      Matrix<Cmplx,3> link1 = arg.dataOr(mu, idx1, 1 - parity);

      // 4 treads per lattice site, the reduction is performed by shared memory without using atomicadd.
      // this implementation needs 4x more shared memory than the implementation using atomicadd
      if ( ImplementationType == 3 ) GaugeFixHit_NoAtomicAdd<blockSize, Float, gauge_dir, 3>(link, link1, arg.relax_boost, tid);
      // 4 treads per lattice site, the reduction is performed by shared memory using atomicadd
      if ( ImplementationType == 4 ) GaugeFixHit_AtomicAdd<blockSize, Float, gauge_dir, 3>(link, link1, arg.relax_boost, tid);
      // 4 treads per lattice site, the reduction is performed by shared memory without using atomicadd.
      // uses the same amount of shared memory as the atomicadd implementation with more thread block synchronization
      if ( ImplementationType == 5 ) GaugeFixHit_NoAtomicAdd_LessSM<blockSize, Float, gauge_dir, 3>(link, link1, arg.relax_boost, tid);

      arg.dataOr(mu, idx, parity) = link;
      arg.dataOr(mu, idx1, 1 - parity) = link1;
    }
  }

  /**
   * @brief Tunable object for the border points of the gauge fixing kernel in multi-GPU implementation
   */
  template<typename Float, typename Gauge, int gauge_dir>
  class GaugeFixBorderPoints : Tunable {
    GaugeFixBorderPointsArg<Float, Gauge> &arg;
    const GaugeField &meta;
    int parity;

    dim3 createGrid(const TuneParam &param) const
    {
      unsigned int blockx = param.block.x / 8;
      if (param.aux.x > 2) blockx = param.block.x / 4;
      unsigned int gx = (arg.threads + blockx - 1) / blockx;
      return dim3(gx, 1, 1);
    }

    bool advanceBlockDim(TuneParam &param) const
    {
      // Use param.aux.x to tune and save state for best kernel option
      // to make use or not of atomicAdd operations and 4 or 8 threads per lattice site!!!
      const unsigned int min_threads0 = 32 * 8;
      const unsigned int min_threads1 = 32 * 4;
      const unsigned int max_threads = 1024; // FIXME: use deviceProp.maxThreadsDim[0];
      const unsigned int atmadd = 0;
      unsigned int min_threads = min_threads0;
      param.aux.x += atmadd; // USE TO SELECT BEST KERNEL OPTION WITH/WITHOUT USING ATOMICADD
      if (param.aux.x > 2) min_threads = 32 * 4;
      param.block.x += min_threads;
      param.block.y = 1;
      param.grid = createGrid(param);

      if ((param.block.x >= min_threads) && (param.block.x <= max_threads)) {
        param.shared_bytes = sharedBytesPerBlock(param);
        return true;
      } else if (param.aux.x == 0) {
        param.block.x = min_threads0;
        param.block.y = 1;
        param.aux.x = 1; // USE FOR ATOMIC ADD
        param.grid = createGrid(param);
        param.shared_bytes = param.block.x * 4 * sizeof(Float) / 8;
        return true;
      } else if (param.aux.x == 1) {
        param.block.x = min_threads0;
        param.block.y = 1;
        param.aux.x = 2; // USE FOR NO ATOMIC ADD and LESS SHARED MEM
        param.grid = createGrid(param);
        param.shared_bytes = param.block.x * 4 * sizeof(Float) / 8;
        return true;
      } else if (param.aux.x == 2) {
        param.block.x = min_threads1;
        param.block.y = 1;
        param.aux.x = 3; // USE FOR NO ATOMIC ADD
        param.grid = createGrid(param);
        param.shared_bytes = param.block.x * 4 * sizeof(Float);
        return true;
      } else if (param.aux.x == 3) {
        param.block.x = min_threads1;
        param.block.y = 1;
        param.aux.x = 4;
        param.grid = createGrid(param);
        param.shared_bytes = param.block.x * sizeof(Float);
        return true;
      } else if (param.aux.x == 4) {
        param.block.x = min_threads1;
        param.block.y = 1;
        param.aux.x = 5;
        param.grid = createGrid(param);
        param.shared_bytes = param.block.x * sizeof(Float);
        return true;
      } else {
        return false;
      }
    }

    unsigned int sharedBytesPerThread() const { return 0; }
    unsigned int sharedBytesPerBlock(const TuneParam &param) const {
      switch (param.aux.x) {
      case 0: return param.block.x * 4 * sizeof(Float);
      case 1: return param.block.x * 4 * sizeof(Float) / 8;
      case 2: return param.block.x * 4 * sizeof(Float) / 8;
      case 3: return param.block.x * 4 * sizeof(Float);
      default: return param.block.x * sizeof(Float);
      }
    }

    bool tuneSharedBytes() const { return false; }
    bool tuneGridDim() const { return false; }
    unsigned int minThreads() const { return arg.threads; }

public:
    GaugeFixBorderPoints(GaugeFixBorderPointsArg<Float, Gauge> &arg, const GaugeField &meta) :
      arg(arg),
      meta(meta),
      parity(0) { }

    ~GaugeFixBorderPoints () {
      if ( comm_partitioned() ) for ( int i = 0; i < 2; i++ ) pool_device_free(arg.borderpoints[i]);
    }

    void setParity(const int par) { parity = par; }

    void apply(const qudaStream_t &stream){
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      LAUNCH_KERNEL_GAUGEFIX(computeFixBorderPoints, tp, stream, arg, parity, Float, Gauge, gauge_dir);
    }

    virtual void initTuneParam(TuneParam &param) const
    {
      param.block = dim3(256, 1, 1);
      param.aux.x = 0;
      param.grid = createGrid(param);
      param.shared_bytes = sharedBytesPerBlock(param);
    }

    virtual void defaultTuneParam(TuneParam &param) const { initTuneParam(param); }

    TuneKey tuneKey() const { return TuneKey(meta.VolString(), typeid(*this).name(), meta.AuxString()); }

    std::string paramString(const TuneParam &param) const {
      std::stringstream ps(Tunable::paramString(param));
      ps << ", atomicadd=" << param.aux.x;
      return ps.str();
    }

    void preTune() { arg.data.backup(); }
    void postTune() { arg.data.restore(); }
    long long flops() const { return 3LL * (22 + 28 * gauge_dir + 224 * 3) * arg.threads; }
    //long long bytes() const { return (1)*8*2*arg.dataOr.Bytes(); } // Only correct if there is no link reconstruction load+save
    long long bytes() const { return 8LL * 2 * arg.threads * meta.Reconstruct() * sizeof(Float); }
  };

  template <int NElems_, typename Gauge>
  struct GaugeFixUnPackArg {
    static constexpr int NElems = NElems_;
    int X[4]; // grid dimensions
#ifdef MULTI_GPU
    int border[4];
#endif
    Gauge dataOr;
    GaugeFixUnPackArg(Gauge & dataOr, GaugeField & data)
      : dataOr(dataOr) {
      for ( int dir = 0; dir < 4; ++dir ) {
        X[dir] = data.X()[dir] - data.R()[dir] * 2;
      #ifdef MULTI_GPU
        border[dir] = data.R()[dir];
      #endif
      }
    }
  };

  template <typename Float, bool pack, typename Arg>
  __global__ void Kernel_UnPackGhost(int size, Arg arg, complex<Float> *array, int parity, int face, int dir)
  {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if ( idx >= size ) return;
    int X[4];
    for ( int dr = 0; dr < 4; ++dr ) X[dr] = arg.X[dr];
    int x[4];
    int za, xodd;
    int borderid = 0;
    parity = 1 - parity;
    switch ( face ) {
    case 0: //X FACE
      za = idx / ( X[1] / 2);
      x[3] = za / X[2];
      x[2] = za - x[3] * X[2];
      x[0] = borderid;
      xodd = (borderid + x[2] + x[3] + parity) & 1;
      x[1] = (2 * idx + xodd)  - za * X[1];
      break;
    case 1: //Y FACE
      za = idx / ( X[0] / 2);
      x[3] = za / X[2];
      x[2] = za - x[3] * X[2];
      x[1] = borderid;
      xodd = (borderid  + x[2] + x[3] + parity) & 1;
      x[0] = (2 * idx + xodd)  - za * X[0];
      break;
    case 2: //Z FACE
      za = idx / ( X[0] / 2);
      x[3] = za / X[1];
      x[1] = za - x[3] * X[1];
      x[2] = borderid;
      xodd = (borderid  + x[1] + x[3] + parity) & 1;
      x[0] = (2 * idx + xodd)  - za * X[0];
      break;
    case 3: //T FACE
      za = idx / ( X[0] / 2);
      x[2] = za / X[1];
      x[1] = za - x[2] * X[1];
      x[3] = borderid;
      xodd = (borderid  + x[1] + x[2] + parity) & 1;
      x[0] = (2 * idx + xodd)  - za * X[0];
      break;
    }
    for ( int dr = 0; dr < 4; ++dr ) {
      x[dr] += arg.border[dr];
      X[dr] += 2 * arg.border[dr];
    }
    x[face] -= 1;
    parity = 1 - parity;
    int id = (((x[3] * X[2] + x[2]) * X[1] + x[1]) * X[0] + x[0]) >> 1;
    typedef complex<Float> Cmplx;
    typedef typename mapper<Float>::type RegType;
    RegType tmp[Arg::NElems];
    Cmplx data[9];
    if ( pack ) {
      arg.dataOr.load(data, id, dir, parity);
      arg.dataOr.reconstruct.Pack(tmp, data, id);
      for ( int i = 0; i < Arg::NElems / 2; ++i ) {
        array[idx + size * i] = Cmplx(tmp[2*i+0], tmp[2*i+1]);
      }
    } else {
      for ( int i = 0; i < Arg::NElems / 2; ++i ) {
        tmp[2*i+0] = array[idx + size * i].real();
        tmp[2*i+1] = array[idx + size * i].imag();
      }
      arg.dataOr.reconstruct.Unpack(data, tmp, id, dir, 0, arg.dataOr.X, arg.dataOr.R);
      arg.dataOr.save(data, id, dir, parity);
    }
  }

  template <typename Float, bool pack, typename Arg>
  __global__ void Kernel_UnPackTop(int size, Arg arg, complex<Float> *array, int parity, int face, int dir)
  {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if ( idx >= size ) return;
    int X[4];
    for ( int dr = 0; dr < 4; ++dr ) X[dr] = arg.X[dr];
    int x[4];
    int za, xodd;
    int borderid = arg.X[face] - 1;
    switch ( face ) {
    case 0: //X FACE
      za = idx / ( X[1] / 2);
      x[3] = za / X[2];
      x[2] = za - x[3] * X[2];
      x[0] = borderid;
      xodd = (borderid + x[2] + x[3] + parity) & 1;
      x[1] = (2 * idx + xodd)  - za * X[1];
      break;
    case 1: //Y FACE
      za = idx / ( X[0] / 2);
      x[3] = za / X[2];
      x[2] = za - x[3] * X[2];
      x[1] = borderid;
      xodd = (borderid  + x[2] + x[3] + parity) & 1;
      x[0] = (2 * idx + xodd)  - za * X[0];
      break;
    case 2: //Z FACE
      za = idx / ( X[0] / 2);
      x[3] = za / X[1];
      x[1] = za - x[3] * X[1];
      x[2] = borderid;
      xodd = (borderid  + x[1] + x[3] + parity) & 1;
      x[0] = (2 * idx + xodd)  - za * X[0];
      break;
    case 3: //T FACE
      za = idx / ( X[0] / 2);
      x[2] = za / X[1];
      x[1] = za - x[2] * X[1];
      x[3] = borderid;
      xodd = (borderid  + x[1] + x[2] + parity) & 1;
      x[0] = (2 * idx + xodd)  - za * X[0];
      break;
    }
    for ( int dr = 0; dr < 4; ++dr ) {
      x[dr] += arg.border[dr];
      X[dr] += 2 * arg.border[dr];
    }
    int id = (((x[3] * X[2] + x[2]) * X[1] + x[1]) * X[0] + x[0]) >> 1;
    typedef complex<Float> Cmplx;
    typedef typename mapper<Float>::type RegType;
    RegType tmp[Arg::NElems];
    Cmplx data[9];
    if ( pack ) {
      arg.dataOr.load(data, id, dir, parity);
      arg.dataOr.reconstruct.Pack(tmp, data, id);
      for ( int i = 0; i < Arg::NElems / 2; ++i ) array[idx + size * i] = Cmplx(tmp[2*i+0], tmp[2*i+1]);
    }
    else{
      for ( int i = 0; i < Arg::NElems / 2; ++i ) {
        tmp[2*i+0] = array[idx + size * i].real();
        tmp[2*i+1] = array[idx + size * i].imag();
      }
      arg.dataOr.reconstruct.Unpack(data, tmp, id, dir, 0, arg.dataOr.X, arg.dataOr.R);
      arg.dataOr.save(data, id, dir, parity);
    }
  }
#endif


  template<typename Float, typename Gauge, int NElems, int gauge_dir>
  void gaugefixingOVR( Gauge dataOr, GaugeField& data,
		       const int Nsteps, const int verbose_interval,
		       const Float relax_boost, const double tolerance,
		       const int reunit_interval, const int stopWtheta)
  {
    TimeProfile profileInternalGaugeFixOVR("InternalGaugeFixQudaOVR", false);

    profileInternalGaugeFixOVR.TPSTART(QUDA_PROFILE_COMPUTE);
    double flop = 0;
    double byte = 0;

    printfQuda("\tOverrelaxation boost parameter: %lf\n", (double)relax_boost);
    printfQuda("\tStop criterium: %lf\n", tolerance);
    if ( stopWtheta ) printfQuda("\tStop criterium method: theta\n");
    else printfQuda("\tStop criterium method: Delta\n");
    printfQuda("\tMaximum number of iterations: %d\n", Nsteps);
    printfQuda("\tReunitarize at every %d steps\n", reunit_interval);
    printfQuda("\tPrint convergence results at every %d steps\n", verbose_interval);

    const double unitarize_eps = 1e-14;
    const double max_error = 1e-10;
    const int reunit_allow_svd = 1;
    const int reunit_svd_only  = 0;
    const double svd_rel_error = 1e-6;
    const double svd_abs_error = 1e-6;
    setUnitarizeLinksConstants(unitarize_eps, max_error,
                               reunit_allow_svd, reunit_svd_only,
                               svd_rel_error, svd_abs_error);
    int num_failures = 0;
    int* num_failures_dev = static_cast<int*>(pool_device_malloc(sizeof(int)));
    qudaMemset(num_failures_dev, 0, sizeof(int));

    GaugeFixQualityArg<Gauge> argQ(dataOr, data);
    GaugeFixQuality<Float,Gauge, gauge_dir> GaugeFixQuality(argQ, data);

    GaugeFixArg<Float, Gauge> arg(dataOr, data, relax_boost);
    GaugeFix<Float,Gauge, gauge_dir> gaugeFix(arg, data);

#ifdef MULTI_GPU
    void *send[4];
    void *recv[4];
    void *sendg[4];
    void *recvg[4];
    void *send_d[4];
    void *recv_d[4];
    void *sendg_d[4];
    void *recvg_d[4];
    void *hostbuffer_h[4];
    qudaStream_t GFStream[9];
    size_t offset[4];
    size_t bytes[4];
    size_t faceVolume[4];
    size_t faceVolumeCB[4];
    // do the exchange
    MsgHandle *mh_recv_back[4];
    MsgHandle *mh_recv_fwd[4];
    MsgHandle *mh_send_fwd[4];
    MsgHandle *mh_send_back[4];
    int X[4];
    TuneParam tp[4];

    if ( comm_partitioned() ) {

      for ( int dir = 0; dir < 4; ++dir ) {
        X[dir] = data.X()[dir] - data.R()[dir] * 2;
        if ( !commDimPartitioned(dir) && data.R()[dir] != 0 ) errorQuda("Not supported!");
      }
      for ( int i = 0; i < 4; i++ ) {
        faceVolume[i] = 1;
        for ( int j = 0; j < 4; j++ ) {
          if ( i == j ) continue;
          faceVolume[i] *= X[j];
        }
        faceVolumeCB[i] = faceVolume[i] / 2;
      }

      for ( int d = 0; d < 4; d++ ) {
        if ( !commDimPartitioned(d)) continue;
        offset[d] = faceVolumeCB[d] * NElems;
        bytes[d] =  sizeof(Float) * offset[d];
        send_d[d] = device_malloc(bytes[d]);
        recv_d[d] = device_malloc(bytes[d]);
        sendg_d[d] = device_malloc(bytes[d]);
        recvg_d[d] = device_malloc(bytes[d]);
        hipStreamCreate(&GFStream[d]);
        hipStreamCreate(&GFStream[4 + d]);
      #ifndef GPU_COMMS
        hostbuffer_h[d] = (void*)pinned_malloc(4 * bytes[d]);
      #endif
        tp[d].block = make_uint3(128, 1, 1);
        tp[d].grid = make_uint3((faceVolumeCB[d] + tp[d].block.x - 1) / tp[d].block.x, 1, 1);
      }
      hipStreamCreate(&GFStream[8]);
      for ( int d = 0; d < 4; d++ ) {
        if ( !commDimPartitioned(d)) continue;
      #ifdef GPU_COMMS
        recv[d] = recv_d[d];
        send[d] = send_d[d];
        recvg[d] = recvg_d[d];
        sendg[d] = sendg_d[d];
      #else
        recv[d] = hostbuffer_h[d];
        send[d] = static_cast<char*>(hostbuffer_h[d]) + bytes[d];
        recvg[d] = static_cast<char*>(hostbuffer_h[d]) + 3 * bytes[d];
        sendg[d] = static_cast<char*>(hostbuffer_h[d]) + 2 * bytes[d];
      #endif
        mh_recv_back[d] = comm_declare_receive_relative(recv[d], d, -1, bytes[d]);
        mh_recv_fwd[d]  = comm_declare_receive_relative(recvg[d], d, +1, bytes[d]);
        mh_send_back[d] = comm_declare_send_relative(sendg[d], d, -1, bytes[d]);
        mh_send_fwd[d]  = comm_declare_send_relative(send[d], d, +1, bytes[d]);
      }
    }
    GaugeFixUnPackArg<NElems,Gauge> dataexarg(dataOr, data);
    GaugeFixBorderPointsArg<Float, Gauge> argBorder(dataOr, data, relax_boost, faceVolume, faceVolumeCB);
    GaugeFixBorderPoints<Float,Gauge, gauge_dir> gfixBorderPoints(argBorder, data);
    GaugeFixInteriorPointsArg<Float, Gauge> argInt(dataOr, data, relax_boost);
    GaugeFixInteriorPoints<Float,Gauge, gauge_dir> gfixIntPoints(argInt, data);
  #endif

    GaugeFixQuality.apply(0);
    flop += (double)GaugeFixQuality.flops();
    byte += (double)GaugeFixQuality.bytes();
    double action0 = argQ.getAction();
    printfQuda("Step: %d\tAction: %.16e\ttheta: %.16e\n", 0, argQ.getAction(), argQ.getTheta());

    unitarizeLinks(data, data, num_failures_dev);
    qudaMemcpy(&num_failures, num_failures_dev, sizeof(int), hipMemcpyDeviceToHost);
    if ( num_failures > 0 ) {
      pool_device_free(num_failures_dev);
      errorQuda("Error in the unitarization\n");
      exit(1);
    }
    qudaMemset(num_failures_dev, 0, sizeof(int));

    int iter = 0;
    for ( iter = 0; iter < Nsteps; iter++ ) {
      for ( int p = 0; p < 2; p++ ) {
      #ifndef MULTI_GPU
        gaugeFix.setParity(p);
        gaugeFix.apply(0);
        flop += (double)gaugeFix.flops();
        byte += (double)gaugeFix.bytes();
      #else
        if ( !comm_partitioned() ) {
          gaugeFix.setParity(p);
          gaugeFix.apply(0);
          flop += (double)gaugeFix.flops();
          byte += (double)gaugeFix.bytes();
        }
        else{
          gfixIntPoints.setParity(p);
          gfixBorderPoints.setParity(p); //compute border points
          gfixBorderPoints.apply(0);
          flop += (double)gfixBorderPoints.flops();
          byte += (double)gfixBorderPoints.bytes();
          flop += (double)gfixIntPoints.flops();
          byte += (double)gfixIntPoints.bytes();
          for ( int d = 0; d < 4; d++ ) {
            if ( !commDimPartitioned(d)) continue;
            comm_start(mh_recv_back[d]);
            comm_start(mh_recv_fwd[d]);
          }
          //wait for the update to the halo points before start packing...
          qudaDeviceSynchronize();
          for ( int d = 0; d < 4; d++ ) {
            if ( !commDimPartitioned(d)) continue;
            //extract top face
            qudaLaunchKernel(Kernel_UnPackTop<Float, true, decltype(dataexarg)>, tp[d], GFStream[d],
                             faceVolumeCB[d], dataexarg, reinterpret_cast<complex<Float>*>(send_d[d]), p, d, d);
            //extract bottom ghost
            qudaLaunchKernel(Kernel_UnPackGhost<Float, true, decltype(dataexarg)>, tp[d], GFStream[4 + d],
                             faceVolumeCB[d], dataexarg, reinterpret_cast<complex<Float>*>(sendg_d[d]), 1 - p, d, d);
          }
        #ifdef GPU_COMMS
          for ( int d = 0; d < 4; d++ ) {
            if ( !commDimPartitioned(d)) continue;
            qudaStreamSynchronize(GFStream[d]);
            comm_start(mh_send_fwd[d]);
            qudaStreamSynchronize(GFStream[4 + d]);
            comm_start(mh_send_back[d]);
          }
        #else
          for ( int d = 0; d < 4; d++ ) {
            if ( !commDimPartitioned(d)) continue;
            qudaMemcpyAsync(send[d], send_d[d], bytes[d], hipMemcpyDeviceToHost, GFStream[d]);
          }
          for ( int d = 0; d < 4; d++ ) {
            if ( !commDimPartitioned(d)) continue;
            qudaMemcpyAsync(sendg[d], sendg_d[d], bytes[d], hipMemcpyDeviceToHost, GFStream[4 + d]);
          }
        #endif
          //compute interior points
          gfixIntPoints.apply(GFStream[8]);

        #ifndef GPU_COMMS
          for ( int d = 0; d < 4; d++ ) {
            if ( !commDimPartitioned(d)) continue;
            qudaStreamSynchronize(GFStream[d]);
            comm_start(mh_send_fwd[d]);
            qudaStreamSynchronize(GFStream[4 + d]);
            comm_start(mh_send_back[d]);
          }
          for ( int d = 0; d < 4; d++ ) {
            if ( !commDimPartitioned(d)) continue;
            comm_wait(mh_recv_back[d]);
            qudaMemcpyAsync(recv_d[d], recv[d], bytes[d], hipMemcpyHostToDevice, GFStream[d]);
          }
          for ( int d = 0; d < 4; d++ ) {
            if ( !commDimPartitioned(d)) continue;
            comm_wait(mh_recv_fwd[d]);
            qudaMemcpyAsync(recvg_d[d], recvg[d], bytes[d], hipMemcpyHostToDevice, GFStream[4 + d]);
          }
        #endif
          for ( int d = 0; d < 4; d++ ) {
            if ( !commDimPartitioned(d)) continue;
          #ifdef GPU_COMMS
            comm_wait(mh_recv_back[d]);
          #endif
            qudaLaunchKernel(Kernel_UnPackGhost<Float, false, decltype(dataexarg)>, tp[d], GFStream[d],
                             faceVolumeCB[d], dataexarg, reinterpret_cast<complex<Float>*>(recv_d[d]), p, d, d);
          }
          for ( int d = 0; d < 4; d++ ) {
            if ( !commDimPartitioned(d)) continue;
          #ifdef GPU_COMMS
            comm_wait(mh_recv_fwd[d]);
          #endif
            qudaLaunchKernel(Kernel_UnPackTop<Float, false, decltype(dataexarg)>, tp[d], GFStream[4 + d],
                             faceVolumeCB[d], dataexarg, reinterpret_cast<complex<Float>*>(recvg_d[d]), 1 - p, d, d);
          }
          for ( int d = 0; d < 4; d++ ) {
            if ( !commDimPartitioned(d)) continue;
            comm_wait(mh_send_back[d]);
            comm_wait(mh_send_fwd[d]);
            qudaStreamSynchronize(GFStream[d]);
            qudaStreamSynchronize(GFStream[4 + d]);
          }
          qudaStreamSynchronize(GFStream[8]);
        }
      #endif
      }
      if ((iter % reunit_interval) == (reunit_interval - 1)) {
        unitarizeLinks(data, data, num_failures_dev);
        qudaMemcpy(&num_failures, num_failures_dev, sizeof(int), hipMemcpyDeviceToHost);
        if ( num_failures > 0 ) errorQuda("Error in the unitarization\n");
        qudaMemset(num_failures_dev, 0, sizeof(int));
        flop += 4588.0 * data.X()[0]*data.X()[1]*data.X()[2]*data.X()[3];
        byte += 8.0 * data.X()[0]*data.X()[1]*data.X()[2]*data.X()[3] * dataOr.Bytes();
      }
      GaugeFixQuality.apply(0);
      flop += (double)GaugeFixQuality.flops();
      byte += (double)GaugeFixQuality.bytes();
      double action = argQ.getAction();
      double diff = abs(action0 - action);
      if ((iter % verbose_interval) == (verbose_interval - 1))
        printfQuda("Step: %d\tAction: %.16e\ttheta: %.16e\tDelta: %.16e\n", iter + 1, argQ.getAction(), argQ.getTheta(), diff);
      if ( stopWtheta ) {
        if ( argQ.getTheta() < tolerance ) break;
      }
      else{
        if ( diff < tolerance ) break;
      }
      action0 = action;
    }
    if ((iter % reunit_interval) != 0 )  {
      unitarizeLinks(data, data, num_failures_dev);
      qudaMemcpy(&num_failures, num_failures_dev, sizeof(int), hipMemcpyDeviceToHost);
      if ( num_failures > 0 ) errorQuda("Error in the unitarization\n");
      qudaMemset(num_failures_dev, 0, sizeof(int));
      flop += 4588.0 * data.X()[0]*data.X()[1]*data.X()[2]*data.X()[3];
      byte += 8.0 * data.X()[0]*data.X()[1]*data.X()[2]*data.X()[3] * dataOr.Bytes();
    }
    if ((iter % verbose_interval) != 0 ) {
      GaugeFixQuality.apply(0);
      flop += (double)GaugeFixQuality.flops();
      byte += (double)GaugeFixQuality.bytes();
      double action = argQ.getAction();
      double diff = abs(action0 - action);
      printfQuda("Step: %d\tAction: %.16e\ttheta: %.16e\tDelta: %.16e\n", iter + 1, argQ.getAction(), argQ.getTheta(), diff);
    }
    pool_device_free(num_failures_dev);
  #ifdef MULTI_GPU
    if ( comm_partitioned() ) {
      data.exchangeExtendedGhost(data.R(),false);
      for ( int d = 0; d < 4; d++ ) {
        if ( commDimPartitioned(d)) {
          comm_free(mh_send_fwd[d]);
          comm_free(mh_send_back[d]);
          comm_free(mh_recv_back[d]);
          comm_free(mh_recv_fwd[d]);
          device_free(send_d[d]);
          device_free(recv_d[d]);
          device_free(sendg_d[d]);
          device_free(recvg_d[d]);
          hipStreamDestroy(GFStream[d]);
          hipStreamDestroy(GFStream[4 + d]);
        #ifndef GPU_COMMS
          host_free(hostbuffer_h[d]);
        #endif
        }
      }
      hipStreamDestroy(GFStream[8]);
    }
  #endif
    qudaDeviceSynchronize();
    profileInternalGaugeFixOVR.TPSTOP(QUDA_PROFILE_COMPUTE);
    if (getVerbosity() > QUDA_SUMMARIZE){
      double secs = profileInternalGaugeFixOVR.Last(QUDA_PROFILE_COMPUTE);
      double gflops = (flop * 1e-9) / (secs);
      double gbytes = byte / (secs * 1e9);
      printfQuda("Time: %6.6f s, Gflop/s = %6.1f, GB/s = %6.1f\n", secs, gflops * comm_size(), gbytes * comm_size());
    }
  }

  template <typename Float, int nColor, QudaReconstructType recon> struct GaugeFixingOVR {
    GaugeFixingOVR(GaugeField& data, const int gauge_dir, const int Nsteps, const int verbose_interval,
                   const Float relax_boost, const double tolerance, const int reunit_interval, const int stopWtheta)
    {
      using Gauge = typename gauge_mapper<Float, recon>::type;
      if (gauge_dir == 4) {
        printfQuda("Starting Landau gauge fixing...\n");
        gaugefixingOVR<Float, Gauge, recon, 4>(Gauge(data), data, Nsteps, verbose_interval, relax_boost, tolerance, reunit_interval, stopWtheta);
      } else if (gauge_dir == 3) {
        printfQuda("Starting Coulomb gauge fixing...\n");
        gaugefixingOVR<Float, Gauge, recon, 3>(Gauge(data), data, Nsteps, verbose_interval, relax_boost, tolerance, reunit_interval, stopWtheta);
      } else {
        errorQuda("Unexpected gauge_dir = %d", gauge_dir);
      }
    }
  };

  /**
   * @brief Gauge fixing with overrelaxation with support for single and multi GPU.
   * @param[in,out] data, quda gauge field
   * @param[in] gauge_dir, 3 for Coulomb gauge fixing, other for Landau gauge fixing
   * @param[in] Nsteps, maximum number of steps to perform gauge fixing
   * @param[in] verbose_interval, print gauge fixing info when iteration count is a multiple of this
   * @param[in] relax_boost, gauge fixing parameter of the overrelaxation method, most common value is 1.5 or 1.7.
   * @param[in] tolerance, torelance value to stop the method, if this value is zero then the method stops when iteration reachs the maximum number of steps defined by Nsteps
   * @param[in] reunit_interval, reunitarize gauge field when iteration count is a multiple of this
   * @param[in] stopWtheta, 0 for MILC criterium and 1 to use the theta value
   */
  void gaugeFixingOVR(GaugeField& data, const int gauge_dir, const int Nsteps, const int verbose_interval, const double relax_boost,
                      const double tolerance, const int reunit_interval, const int stopWtheta) {
#ifdef GPU_GAUGE_ALG
    instantiate<GaugeFixingOVR>(data, gauge_dir, Nsteps, verbose_interval, relax_boost, tolerance, reunit_interval, stopWtheta);
#else
    errorQuda("Gauge fixing has not been built");
#endif // GPU_GAUGE_ALG
  }

}   //namespace quda
