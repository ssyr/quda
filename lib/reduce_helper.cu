#include "hip/hip_runtime.h"
#include <quda_internal.h>
#include <malloc_quda.h>
#include <reduce_helper.h>
#include <tune_quda.h>

// These are used for reduction kernels
static device_reduce_t *d_reduce = nullptr;
static device_reduce_t *h_reduce = nullptr;
static device_reduce_t *hd_reduce = nullptr;

static count_t *reduce_count = nullptr;
static hipEvent_t reduceEnd;

namespace quda
{

  namespace reducer
  {

    // FIXME need to dynamically resize these
    void *get_device_buffer() { return d_reduce; }
    void *get_mapped_buffer() { return hd_reduce; }
    void *get_host_buffer() { return h_reduce; }
    count_t *get_count() { return reduce_count; }
    hipEvent_t &get_event() { return reduceEnd; }

    size_t buffer_size()
    {
      /* we have these different reductions to cater for:

         - regular reductions (reduce_quda.cu) where are reducing to a
           single vector type (max length 4 presently), and a
           grid-stride loop with max number of blocks = 2 x SM count

         - multi-reductions where we are reducing to a matrix of size
           of size QUDA_MAX_MULTI_REDUCE of vectors (max length 4),
           and a grid-stride loop with maximum number of blocks = 2 x
           SM count
      */

      int reduce_size = 4 * sizeof(device_reduce_t);
      int max_reduce = reduce_size;
      int max_multi_reduce = max_n_reduce() * reduce_size;
      int max_reduce_blocks = 2 * deviceProp.multiProcessorCount;

      // reduction buffer size
      size_t bytes = max_reduce_blocks * std::max(max_reduce, max_multi_reduce);
      return bytes;
    }

    // need to use placement new constructor to initialize the atomic counters
    template <typename T> __global__ void init_count(T *counter)
    {
      for (int i = 0; i < max_n_reduce(); i++) new (counter + i) T {0};
    }

    void init()
    {
      auto bytes = buffer_size();
      if (!d_reduce) d_reduce = (device_reduce_t *)device_malloc(bytes);

      // these arrays are actually oversized currently (only needs to be device_reduce_t x 3)

      // if the device supports host-mapped memory then use a host-mapped array for the reduction
      if (!h_reduce) {
        h_reduce = (device_reduce_t *)mapped_malloc(bytes);
        hd_reduce = (device_reduce_t *)get_mapped_device_pointer(h_reduce); // set the matching device pointer

#ifdef HETEROGENEOUS_ATOMIC
        using system_atomic_t = device_reduce_t;
        size_t n_reduce = bytes / sizeof(system_atomic_t);
        auto *atomic_buf = reinterpret_cast<system_atomic_t *>(h_reduce);               // FIXME
        for (size_t i = 0; i < n_reduce; i++) new (atomic_buf + i) system_atomic_t {0}; // placement new constructor
#else
        memset(h_reduce, 0, bytes); // added to ensure that valgrind doesn't report h_reduce is unitialised
#endif
      }

      if (!reduce_count) {
        reduce_count = static_cast<count_t *>(device_malloc(max_n_reduce() * sizeof(decltype(*reduce_count))));
        TuneParam tp;
        tp.grid = dim3(1, 1, 1);
        tp.block = dim3(1, 1, 1);

        qudaLaunchKernel(init_count<count_t>, tp, 0, reduce_count);
      }

      hipEventCreateWithFlags(&reduceEnd, hipEventDisableTiming);

      checkCudaError();
    }

    void destroy()
    {
      hipEventDestroy(reduceEnd);

      if (reduce_count) {
        device_free(reduce_count);
        reduce_count = nullptr;
      }
      if (d_reduce) {
        device_free(d_reduce);
        d_reduce = 0;
      }
      if (h_reduce) {
        host_free(h_reduce);
        h_reduce = 0;
      }
      hd_reduce = 0;
    }

  } // namespace reducer
} // namespace quda
