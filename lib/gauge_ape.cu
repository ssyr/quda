#include <quda_internal.h>
#include <tune_quda.h>
#include <gauge_field.h>

#define  DOUBLE_TOL	1e-15
#define  SINGLE_TOL	2e-6

#include <jitify_helper.cuh>
#include <kernels/gauge_ape.cuh>

namespace quda {

#ifdef GPU_GAUGE_TOOLS

  template <typename Float, typename Arg> class GaugeAPE : TunableVectorYZ
  {
    Arg &arg;
    const GaugeField &meta;

private:
    bool tuneGridDim() const { return false; } // Don't tune the grid dimensions.
    unsigned int minThreads() const { return arg.threads; }

public:
    // (2,3): 2 for parity in the y thread dim, 3 corresponds to mapping direction to the z thread dim
    GaugeAPE(Arg &arg, const GaugeField &meta) : TunableVectorYZ(2, 3), arg(arg), meta(meta)
    {
#ifdef JITIFY
      create_jitify_program("kernels/gauge_ape.cuh");
#endif
    }
    virtual ~GaugeAPE() {}

    void apply(const hipStream_t &stream)
    {
      if (meta.Location() == QUDA_CUDA_FIELD_LOCATION) {
        TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
#ifdef JITIFY
        using namespace jitify::reflection;
        jitify_error = program->kernel("quda::computeAPEStep")
                         .instantiate(Type<Float>(), Type<Arg>())
                         .configure(tp.grid, tp.block, tp.shared_bytes, stream)
                         .launch(arg);
#else
        computeAPEStep<Float><<<tp.grid, tp.block, tp.shared_bytes>>>(arg);
#endif
      } else {
        errorQuda("CPU not supported yet\n");
        // computeAPEStepCPU(arg);
      }
    }

    TuneKey tuneKey() const
    {
      std::stringstream aux;
      aux << "threads=" << arg.threads << ",prec=" << sizeof(Float);
      return TuneKey(meta.VolString(), typeid(*this).name(), aux.str().c_str());
    }

    void preTune() { arg.dest.save(); } // defensive measure in case they alias
    void postTune() { arg.dest.load(); }

    long long flops() const { return 3 * (2 + 2 * 4) * 198ll * arg.threads; } // just counts matrix multiplication
    long long bytes() const { return 3 * ((1 + 2 * 6) * arg.origin.Bytes() + arg.dest.Bytes()) * arg.threads; }
  }; // GaugeAPE

  template<typename Float,typename GaugeOr, typename GaugeDs>
  void APEStep(GaugeOr origin, GaugeDs dest, const GaugeField& dataOr, Float alpha) {
    GaugeAPEArg<Float,GaugeOr,GaugeDs> arg(origin, dest, dataOr, alpha, dataOr.Precision() == QUDA_DOUBLE_PRECISION ? DOUBLE_TOL : SINGLE_TOL);
    GaugeAPE<Float, GaugeAPEArg<Float, GaugeOr, GaugeDs>> gaugeAPE(arg, dataOr);
    gaugeAPE.apply(0);
    qudaDeviceSynchronize();
  }

  template <typename Float> void APEStep(GaugeField &dataDs, const GaugeField &dataOr, Float alpha)
  {

    if(dataDs.Reconstruct() == QUDA_RECONSTRUCT_NO) {
      typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_NO>::type GDs;

      if(dataOr.Reconstruct() == QUDA_RECONSTRUCT_NO) {
	typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_NO>::type GOr;
	APEStep(GOr(dataOr), GDs(dataDs), dataOr, alpha);
      }else if(dataOr.Reconstruct() == QUDA_RECONSTRUCT_12){
	typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_12>::type GOr;
	APEStep(GOr(dataOr), GDs(dataDs), dataOr, alpha);
      }else if(dataOr.Reconstruct() == QUDA_RECONSTRUCT_8){
	typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_8>::type GOr;
	APEStep(GOr(dataOr), GDs(dataDs), dataOr, alpha);
      }else{
	errorQuda("Reconstruction type %d of origin gauge field not supported", dataOr.Reconstruct());
      }
    } else if(dataDs.Reconstruct() == QUDA_RECONSTRUCT_12){
      typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_12>::type GDs;
      if(dataOr.Reconstruct() == QUDA_RECONSTRUCT_NO){
	typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_NO>::type GOr;
	APEStep(GOr(dataOr), GDs(dataDs), dataOr, alpha);
      }else if(dataOr.Reconstruct() == QUDA_RECONSTRUCT_12){
	typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_12>::type GOr;
	APEStep(GOr(dataOr), GDs(dataDs), dataOr, alpha);
      }else if(dataOr.Reconstruct() == QUDA_RECONSTRUCT_8){
	typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_8>::type GOr;
	APEStep(GOr(dataOr), GDs(dataDs), dataOr, alpha);
      }else{
	errorQuda("Reconstruction type %d of origin gauge field not supported", dataOr.Reconstruct());
      }
    } else if(dataDs.Reconstruct() == QUDA_RECONSTRUCT_8){
      typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_8>::type GDs;
      if(dataOr.Reconstruct() == QUDA_RECONSTRUCT_NO){
	typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_NO>::type GOr;
	APEStep(GOr(dataOr), GDs(dataDs), dataOr, alpha);
      }else if(dataOr.Reconstruct() == QUDA_RECONSTRUCT_12){
	typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_12>::type GOr;
	APEStep(GOr(dataOr), GDs(dataDs), dataOr, alpha);
      }else if(dataOr.Reconstruct() == QUDA_RECONSTRUCT_8){
	typedef typename gauge_mapper<Float,QUDA_RECONSTRUCT_8>::type GOr;
	APEStep(GOr(dataOr), GDs(dataDs), dataOr, alpha);
      }else{
	errorQuda("Reconstruction type %d of origin gauge field not supported", dataOr.Reconstruct());
            }
    } else {
      errorQuda("Reconstruction type %d of destination gauge field not supported", dataDs.Reconstruct());
    }
  }

#endif

  void APEStep(GaugeField &dataDs, const GaugeField& dataOr, double alpha) {

#ifdef GPU_GAUGE_TOOLS

    if(dataOr.Precision() != dataDs.Precision()) {
      errorQuda("Origin and destination fields must have the same precision\n");
    }

    if(dataDs.Precision() == QUDA_HALF_PRECISION){
      errorQuda("Half precision not supported\n");
    }

    if (!dataOr.isNative())
      errorQuda("Order %d with %d reconstruct not supported", dataOr.Order(), dataOr.Reconstruct());

    if (!dataDs.isNative())
      errorQuda("Order %d with %d reconstruct not supported", dataDs.Order(), dataDs.Reconstruct());

    if (dataDs.Precision() == QUDA_SINGLE_PRECISION){
      APEStep<float>(dataDs, dataOr, (float) alpha);
    } else if(dataDs.Precision() == QUDA_DOUBLE_PRECISION) {
      APEStep<double>(dataDs, dataOr, alpha);
    } else {
      errorQuda("Precision %d not supported", dataDs.Precision());
    }
    return;
#else
    errorQuda("Gauge tools are not built");
#endif
  }
}
