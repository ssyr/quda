#include <cstdio>
#include <cstdlib>

#include <tune_quda.h>
#include <gauge_field.h>
#include <color_spinor_field.h>
#include <dslash_quda.h>

#include <jitify_helper.cuh>
#include <kernels/clover_sigma_outer_product.cuh>

namespace quda {

#ifdef GPU_CLOVER_DIRAC

  template <typename Float, typename Arg> class CloverSigmaOprod : public TunableVectorYZ
  {
    Arg &arg;
    const GaugeField &meta;

    unsigned int sharedBytesPerThread() const { return 0; }
    unsigned int sharedBytesPerBlock(const TuneParam &) const { return 0; }

    unsigned int minThreads() const { return arg.length; }
    bool tuneGridDim() const { return false; }

  public:
      CloverSigmaOprod(Arg &arg, const GaugeField &meta) : TunableVectorYZ(2, 6), arg(arg), meta(meta)
      {
        writeAuxString("%s,nvector=%d", meta.AuxString(), arg.nvector);
        // this sets the communications pattern for the packing kernel
#ifdef JITIFY
        create_jitify_program("kernels/clover_sigma_outer_product.cuh");
#endif
      }

      virtual ~CloverSigmaOprod() {}

      void apply(const qudaStream_t &stream)
      {
        if (meta.Location() == QUDA_CUDA_FIELD_LOCATION) {
          TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
#ifdef JITIFY
          using namespace jitify::reflection;
          jitify_error = program->kernel("quda::sigmaOprodKernel")
                             .instantiate(arg.nvector, Type<Float>(), Type<Arg>())
                             .configure(tp.grid, tp.block, tp.shared_bytes, stream)
                             .launch(arg);
#else
          switch (arg.nvector) {
          case 1: qudaLaunchKernel(sigmaOprodKernel<1, Float, Arg>, tp, stream, arg); break;
          default: errorQuda("Unsupported nvector = %d\n", arg.nvector);
          }
#endif
        } else { // run the CPU code
          errorQuda("No CPU support for staggered outer-product calculation\n");
        }
      } // apply

      void preTune() { this->arg.oprod.save(); }
      void postTune() { this->arg.oprod.load(); }

      long long flops() const
      {
        return (2 * (long long)arg.length) * 6
            * ((0 + 144 + 18) * arg.nvector + 18); // spin_mu_nu + spin trace + multiply-add
      }
      long long bytes() const
      {
        return (2 * (long long)arg.length) * 6
            * ((arg.inA[0].Bytes() + arg.inB[0].Bytes()) * arg.nvector + 2 * arg.oprod.Bytes());
      }

      TuneKey tuneKey() const { return TuneKey(meta.VolString(), "CloverSigmaOprod", aux); }
  }; // CloverSigmaOprod

  template<typename Float>
  void computeCloverSigmaOprod(GaugeField& oprod, const std::vector<ColorSpinorField*> &x,
			       const std::vector<ColorSpinorField*> &p, const std::vector<std::vector<double> > &coeff, int nvector)
  {
    // Create the arguments
    CloverSigmaOprodArg<Float, 3> arg(oprod, x, p, coeff, nvector);
    CloverSigmaOprod<Float, decltype(arg)> sigma_oprod(arg, oprod);
    sigma_oprod.apply(0);
  } // computeCloverSigmaOprod

#endif // GPU_CLOVER_FORCE

  void computeCloverSigmaOprod(GaugeField& oprod,
			       std::vector<ColorSpinorField*> &x,
			       std::vector<ColorSpinorField*> &p,
			       std::vector<std::vector<double> > &coeff)
  {

#ifdef GPU_CLOVER_DIRAC
    if (x.size() > MAX_NVECTOR) {
      // divide and conquer
      std::vector<ColorSpinorField*> x0(x.begin(), x.begin()+x.size()/2);
      std::vector<ColorSpinorField*> p0(p.begin(), p.begin()+p.size()/2);
      std::vector<std::vector<double> > coeff0(coeff.begin(), coeff.begin()+coeff.size()/2);
      for (unsigned int i=0; i<coeff0.size(); i++) {
	coeff0[i].reserve(2); coeff0[i][0] = coeff[i][0]; coeff0[i][1] = coeff[i][1];
      }
      computeCloverSigmaOprod(oprod, x0, p0, coeff0);

      std::vector<ColorSpinorField*> x1(x.begin()+x.size()/2, x.end());
      std::vector<ColorSpinorField*> p1(p.begin()+p.size()/2, p.end());
      std::vector<std::vector<double> > coeff1(coeff.begin()+coeff.size()/2, coeff.end());
      for (unsigned int i=0; i<coeff1.size(); i++) {
	coeff1[i].reserve(2); coeff1[i][0] = coeff[coeff.size()/2 + i][0]; coeff1[i][1] = coeff[coeff.size()/2 + i][1];
      }
      computeCloverSigmaOprod(oprod, x1, p1, coeff1);

      return;
    }

    if (oprod.Order() != QUDA_FLOAT2_GAUGE_ORDER) errorQuda("Unsupported output ordering: %d\n", oprod.Order());

    if (checkPrecision(*x[0], *p[0], oprod) == QUDA_DOUBLE_PRECISION) {
      computeCloverSigmaOprod<double>(oprod, x, p, coeff, x.size());
    } else {
      errorQuda("Unsupported precision: %d\n", oprod.Precision());
    }
#else // GPU_CLOVER_DIRAC not defined
    errorQuda("Clover Dirac operator has not been built!");
#endif

  } // computeCloverForce

} // namespace quda
