#include "copy_gauge_inc.cu"
namespace quda {

  // this is the function that is actually called, from here on down we instantiate all required templates
  void copyGenericGaugeSingleIn(GaugeField &out, const GaugeField &in, QudaFieldLocation location, void *Out, void *In,
                                void **ghostOut, void **ghostIn, int type)
  {
#if QUDA_PRECISION & 4
    copyGenericGauge<float>(out, in, location, Out, In, ghostOut, ghostIn, type);
#else
    errorQuda("QUDA_PRECISION=%d does not enable single precision", QUDA_PRECISION);
#endif
  }

} // namespace quda
