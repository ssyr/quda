#include <quda_internal.h>
#include <tune_quda.h>
#include <gauge_field.h>

#include <launch_kernel.cuh>
#include <jitify_helper.cuh>
#include <kernels/gauge_qcharge.cuh>

namespace quda
{

#ifdef GPU_GAUGE_TOOLS

  template <typename Float, typename Arg> class QChargeCompute : TunableLocalParity
  {
    Arg &arg;
    const GaugeField &meta;

private:
    bool tuneGridDim() const { return true; }
    unsigned int minThreads() const { return arg.threads; }

public:
    QChargeCompute(Arg &arg, const GaugeField &meta) : arg(arg), meta(meta)
    {
#ifdef JITIFY
      create_jitify_program("kernels/gauge_qcharge.cuh");
#endif
    }
    virtual ~QChargeCompute() {}

    void apply(const hipStream_t &stream)
    {
      if (meta.Location() == QUDA_CUDA_FIELD_LOCATION) {
        arg.result_h[0] = 0.;
        TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
#ifdef JITIFY
        using namespace jitify::reflection;
        jitify_error = program->kernel("quda::qChargeComputeKernel")
                         .instantiate((int)tp.block.x, Type<Float>(), Type<Arg>())
                         .configure(tp.grid, tp.block, tp.shared_bytes, stream)
                         .launch(arg);
#else
	LAUNCH_KERNEL(qChargeComputeKernel, tp, stream, arg, Float);
#endif
        qudaDeviceSynchronize();
      } else { // run the CPU code
        errorQuda("qChargeComputeKernel not supported on CPU");
      }
    }

    TuneKey tuneKey() const
    {
      std::stringstream aux;
      aux << "threads=" << arg.threads << ",prec=" << sizeof(Float);
      return TuneKey(meta.VolString(), typeid(*this).name(), aux.str().c_str());
    }

    long long flops() const { return 2 * arg.threads * (3 * 198 + 9); }
    long long bytes() const { return 2 * arg.threads * ((6 * 18) + Arg::density) * sizeof(Float); }
  }; // QChargeCompute

  template <typename Float, typename Gauge, bool density>
  void computeQCharge(const Gauge data, const GaugeField &Fmunu, Float *qDensity, Float &qChg)
  {
    QChargeArg<Float, Gauge, density> arg(data, Fmunu, qDensity);
    QChargeCompute<Float, decltype(arg)> qChargeCompute(arg, Fmunu);
    qChargeCompute.apply(0);
    checkCudaError();
    comm_allreduce((double *)arg.result_h);
    qChg = arg.result_h[0];
  }

  template <typename Float, bool density> Float computeQCharge(const GaugeField &Fmunu, Float *qDensity = nullptr)
  {
    Float qChg = 0.0;

    if (!Fmunu.isNative()) errorQuda("Topological charge computation only supported on native ordered fields");

    if (Fmunu.Reconstruct() == QUDA_RECONSTRUCT_NO) {
      typedef typename gauge_mapper<Float, QUDA_RECONSTRUCT_NO>::type Gauge;
      computeQCharge<Float, Gauge, density>(Gauge(Fmunu), Fmunu, qDensity, qChg);
    } else if (Fmunu.Reconstruct() == QUDA_RECONSTRUCT_12) {
      typedef typename gauge_mapper<Float, QUDA_RECONSTRUCT_12>::type Gauge;
      computeQCharge<Float, Gauge, density>(Gauge(Fmunu), Fmunu, qDensity, qChg);
    } else if (Fmunu.Reconstruct() == QUDA_RECONSTRUCT_8) {
      typedef typename gauge_mapper<Float, QUDA_RECONSTRUCT_8>::type Gauge;
      computeQCharge<Float, Gauge, density>(Gauge(Fmunu), Fmunu, qDensity, qChg);
    } else {
      errorQuda("Reconstruction type %d of gauge field not supported", Fmunu.Reconstruct());
    }

    return qChg;
  }
#endif // GPU_GAUGE_TOOLS

  double computeQCharge(const GaugeField &Fmunu)
  {
    double qChg = 0.0;
#ifdef GPU_GAUGE_TOOLS
    if (!Fmunu.isNative()) errorQuda("Order %d with %d reconstruct not supported", Fmunu.Order(), Fmunu.Reconstruct());

    if (Fmunu.Precision() == QUDA_SINGLE_PRECISION) {
      qChg = computeQCharge<float, false>(Fmunu);
    } else if (Fmunu.Precision() == QUDA_DOUBLE_PRECISION) {
      qChg = computeQCharge<double, false>(Fmunu);
    } else {
      errorQuda("Precision %d not supported", Fmunu.Precision());
    }
#else
    errorQuda("Gauge tools are not built");
#endif // GPU_GAUGE_TOOLS
    return qChg;
  }

  double computeQChargeDensity(const GaugeField &Fmunu, void *qDensity)
  {
    double qChg = 0.0;
#ifdef GPU_GAUGE_TOOLS
    if (!Fmunu.isNative()) errorQuda("Order %d with %d reconstruct not supported", Fmunu.Order(), Fmunu.Reconstruct());

    if (Fmunu.Precision() == QUDA_SINGLE_PRECISION) {
      qChg = computeQCharge<float, true>(Fmunu, (float *)qDensity);
    } else if (Fmunu.Precision() == QUDA_DOUBLE_PRECISION) {
      qChg = computeQCharge<double, true>(Fmunu, (double *)qDensity);
    } else {
      errorQuda("Precision %d not supported", Fmunu.Precision());
    }
#else
    errorQuda("Gauge tools are not built");
#endif // GPU_GAUGE_TOOLS
    return qChg;
  }
} // namespace quda
