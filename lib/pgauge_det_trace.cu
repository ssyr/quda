#include "hip/hip_runtime.h"
#include <quda_internal.h>
#include <quda_matrix.h>
#include <tune_quda.h>
#include <gauge_field.h>
#include <gauge_field_order.h>
#include <launch_kernel.cuh>
#include <comm_quda.h>
#include <reduce_helper.h>
#include <index_helper.cuh>
#include <instantiate.h>

namespace quda {

  template <typename Float, int nColor_, QudaReconstructType recon_>
  struct KernelArg : public ReduceArg<double2> {
    static constexpr int nColor = nColor_;
    static constexpr QudaReconstructType recon = recon_;
    using real = typename mapper<Float>::type;
    using Gauge = typename gauge_mapper<real, recon>::type;
    int threads; // number of active threads required
    int X[4]; // grid dimensions
    int border[4];
    Gauge dataOr;

    KernelArg(const GaugeField &data) :
      ReduceArg<double2>(),
      dataOr(data),
      threads(data.LocalVolumeCB())
    {
      for (int dir=0; dir<4; ++dir) {
        border[dir] = data.R()[dir];
        X[dir] = data.X()[dir] - border[dir]*2;
      }
    }
  };

  template <int blockSize, int type, typename Arg>
  __global__ void compute(Arg arg)
  {
    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    int parity = threadIdx.y;

    complex<double> val(0.0, 0.0);
    while (idx < arg.threads) {
      int X[4];
#pragma unroll
      for(int dr=0; dr<4; ++dr) X[dr] = arg.X[dr];

      int x[4];
      getCoords(x, idx, X, parity);
#pragma unroll
      for(int dr=0; dr<4; ++dr) {
        x[dr] += arg.border[dr];
        X[dr] += 2*arg.border[dr];
      }
      idx = linkIndex(x,X);
#pragma unroll
      for (int mu = 0; mu < 4; mu++) {
        Matrix<complex<typename Arg::real>, Arg::nColor> U = arg.dataOr(mu, idx, parity);
        if (type == 0) val += getDeterminant(U);
        else if (type == 1) val += getTrace(U);
      }

      idx += blockDim.x*gridDim.x;
    }

    arg.template reduce2d<blockSize,2>(val);
  }

  template <typename Float, int nColor, QudaReconstructType recon, int type>
  class CalcFunc : TunableLocalParityReduction {
    double2 &result;
    const GaugeField &u;

  public:
    CalcFunc(double2 &result, const GaugeField &u) :
      result(result),
      u(u)
    {
      apply(0);
    }

    void apply(const qudaStream_t &stream)
    {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      KernelArg<Float, nColor, recon> arg(u);
      LAUNCH_KERNEL_LOCAL_PARITY(compute, (*this), tp, stream, arg, type, decltype(arg));
      arg.complete(result, stream);
      if (!activeTuning()) {
        comm_allreduce_array((double*)&result, 2);
        result.x /= (double)(4*u.LocalVolume()*comm_size());
        result.y /= (double)(4*u.LocalVolume()*comm_size());
      }
    }

    TuneKey tuneKey() const { return TuneKey(u.VolString(), typeid(*this).name(), u.AuxString()); }

    long long flops() const {
      if (u.Ncolor()==3 && type == 0) return 264LL*u.LocalVolume();
      else if (type == 1) return 2*u.Geometry()*u.Ncolor()*u.LocalVolume();
      else return 0;
    }

    long long bytes() const { return u.Bytes(); }
  };

  template <typename Float, int nColor, QudaReconstructType recon> struct computeDeterminant {
    computeDeterminant(GaugeField &data, double2 &det)
    {
      CalcFunc<Float, nColor, recon, 0>(det, data);
    }
  };

  template <typename Float, int nColor, QudaReconstructType recon> struct computeTrace {
    computeTrace(GaugeField &data, double2 &trace)
    {
      CalcFunc<Float, nColor, recon, 1>(trace, data);
    }
  };

  /**
   * @brief Calculate the Determinant
   *
   * @param[in] data Gauge field
   * @returns double2 complex Determinant value
   */
  double2 getLinkDeterminant(GaugeField& data)
  {
    double2 det = make_double2(0.0,0.0);
#ifdef GPU_GAUGE_ALG
    instantiate<computeDeterminant>(data, det);
#else
    errorQuda("Pure gauge code has not been built");
#endif // GPU_GAUGE_ALG
    return det;
  }

  /**
   * @brief Calculate the Trace
   *
   * @param[in] data Gauge field
   * @returns double2 complex trace value
   */
  double2 getLinkTrace(GaugeField& data)
  {
    double2 det = make_double2(0.0,0.0);
#ifdef GPU_GAUGE_ALG
    instantiate<computeTrace>(data, det);
#else
    errorQuda("Pure gauge code has not been built");
#endif // GPU_GAUGE_ALG
    return det;
  }

} // namespace quda
