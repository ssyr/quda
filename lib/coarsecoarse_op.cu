#include <transfer.h>
#include <color_spinor_field.h>
#include <gauge_field.h>

#define COARSECOARSE
#ifdef GPU_MULTIGRID
#include <coarse_op.cuh>
#endif
namespace quda {

#ifdef GPU_MULTIGRID

  template <typename Float, typename vFloat, int fineColor, int fineSpin, int coarseColor, int coarseSpin>
  void calculateYcoarse(GaugeField &Y, GaugeField &X, GaugeField &Yatomic, GaugeField &Xatomic,
			ColorSpinorField &uv, const Transfer &T, const GaugeField &g, const GaugeField &clover,
			const GaugeField &cloverInv, double kappa, double mu, double mu_factor, QudaDiracType dirac, QudaMatPCType matpc,
                        bool need_bidirectional) {

    if (Y.Location() == QUDA_CPU_FIELD_LOCATION) {

      constexpr QudaFieldOrder csOrder = QUDA_SPACE_SPIN_COLOR_FIELD_ORDER;
      constexpr QudaGaugeFieldOrder gOrder = QUDA_QDP_GAUGE_ORDER;

      if (T.Vectors(Y.Location()).FieldOrder() != csOrder)
	errorQuda("Unsupported field order %d\n", T.Vectors(Y.Location()).FieldOrder());
      if (g.FieldOrder() != gOrder) errorQuda("Unsupported field order %d\n", g.FieldOrder());

      typedef typename colorspinor::FieldOrderCB<Float,fineSpin,fineColor,coarseColor,csOrder,vFloat> V;
      typedef typename colorspinor::FieldOrderCB<Float,2*fineSpin,fineColor,coarseColor,csOrder,vFloat> F;
      typedef typename gauge::FieldOrder<Float,fineColor*fineSpin,fineSpin,gOrder,true,vFloat> gFine;
      typedef typename gauge::FieldOrder<Float,fineColor*fineSpin,fineSpin,gOrder,true,vFloat> cFine;
      typedef typename gauge::FieldOrder<Float,coarseColor*coarseSpin,coarseSpin,gOrder,true,vFloat> gCoarse;
      typedef typename gauge::FieldOrder<Float,coarseColor*coarseSpin,coarseSpin,gOrder,true,storeType> gCoarseAtomic;

      const ColorSpinorField &v = T.Vectors(Y.Location());

      V vAccessor(const_cast<ColorSpinorField&>(v));
      F uvAccessor(const_cast<ColorSpinorField&>(uv));
      gFine gAccessor(const_cast<GaugeField&>(g));
      cFine cAccessor(const_cast<GaugeField&>(clover));
      cFine cInvAccessor(const_cast<GaugeField&>(cloverInv));
      gCoarse yAccessor(const_cast<GaugeField&>(Y));
      gCoarse xAccessor(const_cast<GaugeField&>(X));
      gCoarseAtomic yAccessorAtomic(const_cast<GaugeField&>(Yatomic));
      gCoarseAtomic xAccessorAtomic(const_cast<GaugeField&>(Xatomic));

      calculateY<true,Float,fineSpin,fineColor,coarseSpin,coarseColor>
	(yAccessor, xAccessor, yAccessorAtomic, xAccessorAtomic,
	 uvAccessor, vAccessor, vAccessor, gAccessor, cAccessor, cInvAccessor,
	 Y, X, Yatomic, Xatomic, uv, const_cast<ColorSpinorField&>(v), v, kappa, mu, mu_factor, dirac, matpc, need_bidirectional,
	 T.fineToCoarse(Y.Location()), T.coarseToFine(Y.Location()));

    } else {

      constexpr QudaFieldOrder csOrder = QUDA_FLOAT2_FIELD_ORDER;
      constexpr QudaGaugeFieldOrder gOrder = QUDA_FLOAT2_GAUGE_ORDER;

      if (T.Vectors(Y.Location()).FieldOrder() != csOrder)
	errorQuda("Unsupported field order %d\n", T.Vectors(Y.Location()).FieldOrder());
      if (g.FieldOrder() != gOrder) errorQuda("Unsupported field order %d\n", g.FieldOrder());

      typedef typename colorspinor::FieldOrderCB<Float,fineSpin,fineColor,coarseColor,csOrder,vFloat> V;
      typedef typename colorspinor::FieldOrderCB<Float,2*fineSpin,fineColor,coarseColor,csOrder,vFloat> F;
      typedef typename gauge::FieldOrder<Float,fineColor*fineSpin,fineSpin,gOrder,true,vFloat> gFine;
      typedef typename gauge::FieldOrder<Float,fineColor*fineSpin,fineSpin,gOrder,true,vFloat> cFine;
      typedef typename gauge::FieldOrder<Float,coarseColor*coarseSpin,coarseSpin,gOrder,true,vFloat> gCoarse;
      typedef typename gauge::FieldOrder<Float,coarseColor*coarseSpin,coarseSpin,gOrder,true,storeType> gCoarseAtomic;

      const ColorSpinorField &v = T.Vectors(Y.Location());

      V vAccessor(const_cast<ColorSpinorField&>(v));
      F uvAccessor(const_cast<ColorSpinorField&>(uv));
      gFine gAccessor(const_cast<GaugeField&>(g));
      cFine cAccessor(const_cast<GaugeField&>(clover));
      cFine cInvAccessor(const_cast<GaugeField&>(cloverInv));
      gCoarse yAccessor(const_cast<GaugeField&>(Y));
      gCoarse xAccessor(const_cast<GaugeField&>(X));
      gCoarseAtomic yAccessorAtomic(const_cast<GaugeField&>(Yatomic));
      gCoarseAtomic xAccessorAtomic(const_cast<GaugeField&>(Xatomic));

      calculateY<true,Float,fineSpin,fineColor,coarseSpin,coarseColor>
	(yAccessor, xAccessor, yAccessorAtomic, xAccessorAtomic,
	 uvAccessor, vAccessor, vAccessor, gAccessor, cAccessor, cInvAccessor,
	 Y, X, Yatomic, Xatomic, uv, const_cast<ColorSpinorField&>(v), v, kappa, mu, mu_factor, dirac, matpc, need_bidirectional,
	 T.fineToCoarse(Y.Location()), T.coarseToFine(Y.Location()));

    }

  }

  // template on the number of coarse degrees of freedom
  template <typename Float, typename vFloat, int fineColor, int fineSpin>
  void calculateYcoarse(GaugeField &Y, GaugeField &X, GaugeField &Yatomic, GaugeField &Xatomic,
			ColorSpinorField &uv, const Transfer &T, const GaugeField &g, const GaugeField &clover,
			const GaugeField &cloverInv, double kappa, double mu, double mu_factor, QudaDiracType dirac, QudaMatPCType matpc, bool need_bidirectional) {
    if (T.Vectors().Nspin()/T.Spin_bs() != 2) 
      errorQuda("Unsupported number of coarse spins %d\n",T.Vectors().Nspin()/T.Spin_bs());
    const int coarseSpin = 2;
    const int coarseColor = Y.Ncolor() / coarseSpin;

    if (coarseColor == 6) {
      calculateYcoarse<Float,vFloat,fineColor,fineSpin,6,coarseSpin>(Y, X, Yatomic, Xatomic, uv, T, g, clover, cloverInv, kappa, mu, mu_factor, dirac, matpc, need_bidirectional);
#if 0
    } else if (coarseColor == 8) {
      calculateYcoarse<Float,vFloat,fineColor,fineSpin,8,coarseSpin>(Y, X, Yatomic, Xatomic, uv, T, g, clover, cloverInv, kappa, mu, mu_factor, dirac, matpc, need_bidirectional);
    } else if (coarseColor == 16) {
      calculateYcoarse<Float,vFloat,fineColor,fineSpin,16,coarseSpin>(Y, X, Yatomic, Xatomic, uv, T, g, clover, cloverInv, kappa, mu, mu_factor, dirac, matpc, need_bidirectional);
#endif
    } else if (coarseColor == 24) {
      calculateYcoarse<Float,vFloat,fineColor,fineSpin,24,coarseSpin>(Y, X, Yatomic, Xatomic, uv, T, g, clover, cloverInv, kappa, mu, mu_factor, dirac, matpc, need_bidirectional);
    } else if (coarseColor == 32) {
      calculateYcoarse<Float,vFloat,fineColor,fineSpin,32,coarseSpin>(Y, X, Yatomic, Xatomic, uv, T, g, clover, cloverInv, kappa, mu, mu_factor, dirac, matpc, need_bidirectional);
    } else {
      errorQuda("Unsupported number of coarse dof %d\n", Y.Ncolor());
    }
  }

  // template on fine spin
  template <typename Float, typename vFloat, int fineColor>
  void calculateYcoarse(GaugeField &Y, GaugeField &X, GaugeField &Yatomic, GaugeField &Xatomic,
			ColorSpinorField &uv, const Transfer &T, const GaugeField &g, const GaugeField &clover,
			const GaugeField &cloverInv, double kappa, double mu, double mu_factor, QudaDiracType dirac, QudaMatPCType matpc, bool need_bidirectional) {
    if (T.Vectors().Nspin() == 2) {
      calculateYcoarse<Float,vFloat,fineColor,2>(Y, X, Yatomic, Xatomic, uv, T, g, clover, cloverInv, kappa, mu, mu_factor, dirac, matpc, need_bidirectional);
    } else {
      errorQuda("Unsupported number of spins %d\n", T.Vectors().Nspin());
    }
  }

  // template on fine colors
  template <typename Float, typename vFloat>
  void calculateYcoarse(GaugeField &Y, GaugeField &X, GaugeField &Yatomic, GaugeField &Xatomic,
			ColorSpinorField &uv, const Transfer &T, const GaugeField &g, const GaugeField &clover,
			const GaugeField &cloverInv, double kappa, double mu, double mu_factor, QudaDiracType dirac, QudaMatPCType matpc, bool need_bidirectional) {
    if (g.Ncolor()/T.Vectors().Nspin() == 6) { // free field Wilson
      calculateYcoarse<Float,vFloat,6>(Y, X, Yatomic, Xatomic, uv, T, g, clover, cloverInv, kappa, mu, mu_factor, dirac, matpc, need_bidirectional);
#if 0
    } else if (g.Ncolor()/T.Vectors().Nspin() == 8) {
      calculateYcoarse<Float,vFloat,8>(Y, X, Yatomic, Xatomic, uv, T, g, clover, cloverInv, kappa, mu, mu_factor, dirac, matpc, need_bidirectional);
    } else if (g.Ncolor()/T.Vectors().Nspin() == 16) {
      calculateYcoarse<Float,vFloat,16>(Y, X, Yatomic, Xatomic, uv, T, g, clover, cloverInv, kappa, mu, mu_factor, dirac, matpc, need_bidirectional);
#endif
    } else if (g.Ncolor()/T.Vectors().Nspin() == 24) {
      calculateYcoarse<Float,vFloat,24>(Y, X, Yatomic, Xatomic, uv, T, g, clover, cloverInv, kappa, mu, mu_factor, dirac, matpc, need_bidirectional);
    } else if (g.Ncolor()/T.Vectors().Nspin() == 32) {
      calculateYcoarse<Float,vFloat,32>(Y, X, Yatomic, Xatomic, uv, T, g, clover, cloverInv, kappa, mu, mu_factor, dirac, matpc, need_bidirectional);
    } else {
      errorQuda("Unsupported number of colors %d\n", g.Ncolor());
    }
  }

  //Does the heavy lifting of creating the coarse color matrices Y
  void calculateYcoarse(GaugeField &Y, GaugeField &X, GaugeField &Yatomic, GaugeField &Xatomic, ColorSpinorField &uv,
			const Transfer &T, const GaugeField &g, const GaugeField &clover, const GaugeField &cloverInv,
			double kappa, double mu, double mu_factor, QudaDiracType dirac, QudaMatPCType matpc, bool need_bidirectional) {
    checkPrecision(X, Y, g, clover, cloverInv, uv, T.Vectors(X.Location()));
    checkPrecision(Xatomic, Yatomic);

    if (getVerbosity() >= QUDA_SUMMARIZE) printfQuda("Computing Y field......\n");
    if (Y.Precision() == QUDA_DOUBLE_PRECISION) {
#ifdef GPU_MULTIGRID_DOUBLE
      if (T.Vectors(X.Location()).Precision() == QUDA_DOUBLE_PRECISION) {
	calculateYcoarse<double,double>(Y, X, Yatomic, Xatomic, uv, T, g, clover, cloverInv, kappa, mu, mu_factor, dirac, matpc, need_bidirectional);
      } else {
	errorQuda("Unsupported precision %d\n", Y.Precision());
      }
#else
      errorQuda("Double precision multigrid has not been enabled");
#endif
    } else if (Y.Precision() == QUDA_SINGLE_PRECISION) {
      if (T.Vectors(X.Location()).Precision() == QUDA_SINGLE_PRECISION) {
	calculateYcoarse<float,float>(Y, X, Yatomic, Xatomic, uv, T, g, clover, cloverInv, kappa, mu, mu_factor, dirac, matpc, need_bidirectional);
      } else {
	errorQuda("Unsupported precision %d\n", T.Vectors(X.Location()).Precision());
      }
    } else if (Y.Precision() == QUDA_HALF_PRECISION) {
      if (T.Vectors(X.Location()).Precision() == QUDA_HALF_PRECISION) {
	calculateYcoarse<float,short>(Y, X, Yatomic, Xatomic, uv, T, g, clover, cloverInv, kappa, mu, mu_factor, dirac, matpc, need_bidirectional);
      } else {
	errorQuda("Unsupported precision %d\n", T.Vectors(X.Location()).Precision());
      }
    } else {
      errorQuda("Unsupported precision %d\n", Y.Precision());
    }
    if (getVerbosity() >= QUDA_SUMMARIZE) printfQuda("....done computing Y field\n");
  }

#endif // GPU_MULTIGRID

  //Calculates the coarse color matrix and puts the result in Y.
  //N.B. Assumes Y, X have been allocated.
  void CoarseCoarseOp(GaugeField &Y, GaugeField &X, const Transfer &T,
		      const GaugeField &gauge, const GaugeField &clover, const GaugeField &cloverInv,
		      double kappa, double mu, double mu_factor, QudaDiracType dirac, QudaMatPCType matpc,
                      bool need_bidirectional) {

#ifdef GPU_MULTIGRID
    QudaPrecision precision = Y.Precision();
    QudaFieldLocation location = checkLocation(X, Y, gauge, clover, cloverInv);

    //Create a field UV which holds U*V.  Has the same similar
    //structure to V but double the number of spins so we can store
    //the four distinct block chiral multiplications in a single UV
    //computation.
    ColorSpinorParam UVparam(T.Vectors(location));
    UVparam.create = QUDA_ZERO_FIELD_CREATE;
    UVparam.location = location;
    UVparam.nSpin *= 2; // so nSpin == 4
    UVparam.setPrecision(T.Vectors(location).Precision());
    UVparam.mem_type = Y.MemType(); // allocate temporaries to match coarse-grid link field

    ColorSpinorField *uv = ColorSpinorField::Create(UVparam);

    GaugeField *Yatomic = &Y;
    GaugeField *Xatomic = &X;
    if (Y.Precision() < QUDA_SINGLE_PRECISION) {
      // we need to coarsen into single precision fields (float or int), so we allocate temporaries for this purpose
      // else we can just coarsen directly into the original fields
      GaugeFieldParam param(X); // use X since we want scalar geometry
      param.location = location;
      param.setPrecision(QUDA_SINGLE_PRECISION, location == QUDA_CUDA_FIELD_LOCATION ? true : false);

      Yatomic = GaugeField::Create(param);
      Xatomic = GaugeField::Create(param);
    }

    calculateYcoarse(Y, X, *Yatomic, *Xatomic, *uv, T, gauge, clover, cloverInv, kappa, mu, mu_factor, dirac, matpc, need_bidirectional);

    if (Yatomic != &Y) delete Yatomic;
    if (Xatomic != &X) delete Xatomic;

    delete uv;
#else
    errorQuda("Multigrid has not been built");
#endif // GPU_MULTIGRID
  }
  
} //namespace quda
