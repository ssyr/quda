#include <copy_color_spinor.cuh>

namespace quda {
  
  void copyGenericColorSpinorHQ(ColorSpinorField &dst, const ColorSpinorField &src, 
				QudaFieldLocation location, void *Dst, void *Src, 
				void *dstNorm, void *srcNorm) {
#if (QUDA_PRECISION & 2) && (QUDA_PRECISION & 1)
    CopyGenericColorSpinor<3>(dst, src, location, (short*)Dst, (int8_t*)Src, (float*)dstNorm, (float*)srcNorm);
#else
    errorQuda("QUDA_PRECISION=%d does not enable precision combination %d %d", QUDA_PRECISION, dst.Precision(), src.Precision());
#endif
  }  

} // namespace quda
