#include "hip/hip_runtime.h"
#include <blas_quda.h>
#include <tune_quda.h>
#include <color_spinor_field_order.h>
#include <jitify_helper.cuh>
#include <kernels/reduce_core.cuh>

namespace quda {

  namespace blas {

    qudaStream_t* getStream();

    template <int block_size, typename real, int len, typename Arg>
    typename std::enable_if<block_size!=32, qudaError_t>::type launch(Arg &arg, const TuneParam &tp, const qudaStream_t &stream)
    {
      if (tp.block.x == block_size)
        return qudaLaunchKernel(reduceKernel<block_size, real, len, Arg>, tp, stream, arg);
      else
        return launch<block_size - 32, real, len>(arg, tp, stream);
    }

    template <int block_size, typename real, int len, typename Arg>
    typename std::enable_if<block_size==32, qudaError_t>::type launch(Arg &arg, const TuneParam &tp, const qudaStream_t &stream)
    {
      if (block_size != tp.block.x) errorQuda("Unexpected block size %d\n", tp.block.x);
      return qudaLaunchKernel(reduceKernel<block_size, real, len, Arg>, tp, stream, arg);
    }

#ifdef QUDA_FAST_COMPILE_REDUCE
    constexpr static unsigned int max_block_size() { return 32; }
#else
    constexpr static unsigned int max_block_size() { return 1024; }
#endif

   /**
       Generic reduction kernel launcher
    */
    template <typename host_reduce_t, typename real, int len, typename Arg>
    auto reduceLaunch(Arg &arg, const TuneParam &tp, const qudaStream_t &stream, Tunable &tunable)
    {
      using device_reduce_t = typename Arg::Reducer::reduce_t;
      if (tp.grid.x > (unsigned int)deviceProp.maxGridSize[0])
        errorQuda("Grid size %d greater than maximum %d\n", tp.grid.x, deviceProp.maxGridSize[0]);

#ifdef JITIFY
      using namespace jitify::reflection;
      tunable.jitifyError() = program->kernel("quda::blas::reduceKernel")
                                  .instantiate((int)tp.block.x, Type<real>(), len, Type<Arg>())
                                  .configure(tp.grid, tp.block, tp.shared_bytes, stream)
                                  .launch(arg);
      arg.launch_error = tunable.jitifyError() == hipSuccess ? QUDA_SUCCESS : QUDA_ERROR;
#else
      arg.launch_error = launch<max_block_size(), real, len>(arg, tp, stream);
#endif

      host_reduce_t result;
      ::quda::zero(result);
      if (!commAsyncReduction()) arg.complete(result, stream);
      return result;
    }

    template <template <typename ReducerType, typename real> class Reducer,
              typename store_t, typename y_store_t, int nSpin, typename coeff_t>
    class Reduce : public Tunable
    {
      using real = typename mapper<y_store_t>::type;
      using host_reduce_t = typename Reducer<double, real>::reduce_t;
      Reducer<device_reduce_t, real> r;
      const int nParity; // for composite fields this includes the number of composites
      host_reduce_t &result;

      const coeff_t &a, &b;
      ColorSpinorField &x, &y, &z, &w, &v;
      QudaFieldLocation location;

      unsigned int sharedBytesPerThread() const { return 0; }
      unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }

      bool advanceSharedBytes(TuneParam &param) const
      {
        TuneParam next(param);
        advanceBlockDim(next); // to get next blockDim
        int nthreads = next.block.x * next.block.y * next.block.z;
        param.shared_bytes = sharedBytesPerThread() * nthreads > sharedBytesPerBlock(param) ?
            sharedBytesPerThread() * nthreads :
            sharedBytesPerBlock(param);
        return false;
      }

      unsigned int maxBlockSize(const TuneParam &param) const { return max_block_size(); }

    public:
      Reduce(const coeff_t &a, const coeff_t &b, const coeff_t &c, ColorSpinorField &x, ColorSpinorField &y,
             ColorSpinorField &z, ColorSpinorField &w, ColorSpinorField &v, host_reduce_t &result) :
        r(a, b),
        nParity((x.IsComposite() ? x.CompositeDim() : 1) * (x.SiteSubset())),
        a(a),
        b(b),
        x(x),
        y(y),
        z(z),
        w(w),
        v(v),
        result(result),
        location(checkLocation(x, y, z, w, v))
      {
        checkLength(x, y, z, w, v);
        auto x_prec = checkPrecision(x, z, w, v);
        auto y_prec = y.Precision();
        auto x_order = checkOrder(x, z, w, v);
        auto y_order = y.FieldOrder();
        if (sizeof(store_t) != x_prec) errorQuda("Expected precision %lu but received %d", sizeof(store_t), x_prec);
        if (sizeof(y_store_t) != y_prec) errorQuda("Expected precision %lu but received %d", sizeof(y_store_t), y_prec);
        if (x_prec == y_prec && x_order != y_order) errorQuda("Orders %d %d do not match", x_order, y_order);

        strcpy(aux, x.AuxString());
        if (x_prec != y_prec) {
          strcat(aux, ",");
          strcat(aux, y.AuxString());
        }
        strcat(aux, nParity == 2 ? ",nParity=2" : ",nParity=1");
        if (location == QUDA_CPU_FIELD_LOCATION) strcat(aux, ",CPU");
        if (commAsyncReduction()) strcat(aux, ",async");

#ifdef JITIFY
        ::quda::create_jitify_program("kernels/reduce_core.cuh");
#endif

        apply(*(blas::getStream()));

        blas::bytes += bytes();
        blas::flops += flops();

        const int Nreduce = sizeof(host_reduce_t) / sizeof(double);
        reduceDoubleArray((double *)&result, Nreduce);
      }

      TuneKey tuneKey() const { return TuneKey(x.VolString(), typeid(r).name(), aux); }

      void apply(const qudaStream_t &stream)
      {
        constexpr bool site_unroll_check = !std::is_same<store_t, y_store_t>::value || isFixed<store_t>::value || decltype(r)::site_unroll;
        if (site_unroll_check && (x.Ncolor() != 3 || x.Nspin() == 2))
          errorQuda("site unroll not supported for nSpin = %d nColor = %d", x.Nspin(), x.Ncolor());

        TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
        if (location == QUDA_CUDA_FIELD_LOCATION) {
          if (site_unroll_check) checkNative(x, y, z, w, v); // require native order when using site_unroll
          using device_store_t = typename device_type_mapper<store_t>::type;
          using device_y_store_t = typename device_type_mapper<y_store_t>::type;
          using device_real_t = typename mapper<device_y_store_t>::type;
          Reducer<device_reduce_t, device_real_t> r_(a, b);

          // redefine site_unroll with device_store types to ensure we have correct N/Ny/M values
          constexpr bool site_unroll = !std::is_same<device_store_t, device_y_store_t>::value || isFixed<device_store_t>::value || decltype(r)::site_unroll;
          constexpr int N = n_vector<device_store_t, true, nSpin, site_unroll>();
          constexpr int Ny = n_vector<device_y_store_t, true, nSpin, site_unroll>();
          constexpr int M = site_unroll ? (nSpin == 4 ? 24 : 6) : N; // real numbers per thread
          const int length = x.Length() / (nParity * M);

          ReductionArg<device_store_t, N, device_y_store_t, Ny, decltype(r_)> arg(x, y, z, w, v, r_, length, nParity, tp);
          result = reduceLaunch<host_reduce_t, device_real_t, M>(arg, tp, stream, *this);
        } else {
          if (checkOrder(x, y, z, w, v) != QUDA_SPACE_SPIN_COLOR_FIELD_ORDER) {
            warningQuda("CPU Blas functions expect AoS field order");
            return;
          }

          using host_store_t = typename host_type_mapper<store_t>::type;
          using host_y_store_t = typename host_type_mapper<y_store_t>::type;
          using host_real_t = typename mapper<host_y_store_t>::type;
          Reducer<double, host_real_t> r_(a, b);

          // redefine site_unroll with host_store types to ensure we have correct N/Ny/M values
          constexpr bool site_unroll = !std::is_same<host_store_t, host_y_store_t>::value || isFixed<host_store_t>::value || decltype(r)::site_unroll;
          constexpr int N = n_vector<host_store_t, false, nSpin, site_unroll>();
          constexpr int Ny = n_vector<host_y_store_t, false, nSpin, site_unroll>();
          constexpr int M = N; // if site unrolling then M=N will be 24/6, e.g., full AoS
          const int length = x.Length() / (nParity * M);

          ReductionArg<host_store_t, N, host_y_store_t, Ny, decltype(r_)> arg(x, y, z, w, v, r_, length, nParity, tp);
          result = reduceCPU<host_real_t, M>(arg);
        }
      }

      void preTune()
      {
        if (r.write.X) x.backup();
        if (r.write.Y) y.backup();
        if (r.write.Z) z.backup();
        if (r.write.W) w.backup();
        if (r.write.V) v.backup();
      }

      void postTune()
      {
        if (r.write.X) x.restore();
        if (r.write.Y) y.restore();
        if (r.write.Z) z.restore();
        if (r.write.W) w.restore();
        if (r.write.V) v.restore();
      }

      bool advanceTuneParam(TuneParam &param) const
      {
        return location == QUDA_CPU_FIELD_LOCATION ? false : Tunable::advanceTuneParam(param);
      }

      void initTuneParam(TuneParam &param) const
      {
        Tunable::initTuneParam(param);
      }

      void defaultTuneParam(TuneParam &param) const
      {
        Tunable::defaultTuneParam(param);
      }

      long long flops() const { return r.flops() * x.Length(); }

      long long bytes() const
      {
        return (r.read.X + r.write.X) * x.Bytes() + (r.read.Y + r.write.Y) * y.Bytes() +
          (r.read.Z + r.write.Z) * z.Bytes() + (r.read.W + r.write.W) * w.Bytes() + (r.read.V + r.write.V) * v.Bytes();
      }

      int tuningIter() const { return 3; }
    };

    template <template <typename reduce_t, typename real> class Functor, bool mixed, typename... Args>
    auto instantiateReduce(Args &&... args)
    {
      using host_reduce_t = typename Functor<double, double>::reduce_t;
      host_reduce_t value;
      ::quda::zero(value); // no default constructor so we need to explicitly zero
      instantiate<Functor, Reduce, mixed>(args..., value);
      return value;
    }

    double norm1(const ColorSpinorField &x)
    {
      ColorSpinorField &y = const_cast<ColorSpinorField &>(x); // FIXME
      return instantiateReduce<Norm1, false>(0.0, 0.0, 0.0, y, y, y, y, y);
    }

    double norm2(const ColorSpinorField &x)
    {
      ColorSpinorField &y = const_cast<ColorSpinorField &>(x);
      return instantiateReduce<Norm2, false>(0.0, 0.0, 0.0, y, y, y, y, y);
    }

    double reDotProduct(ColorSpinorField &x, ColorSpinorField &y)
    {
      return instantiateReduce<Dot, false>(0.0, 0.0, 0.0, x, y, x, x, x);
    }

    double axpbyzNorm(double a, ColorSpinorField &x, double b, ColorSpinorField &y, ColorSpinorField &z)
    {
      return instantiateReduce<axpbyzNorm2, false>(a, b, 0.0, x, y, z, x, x);
    }

    double axpyReDot(double a, ColorSpinorField &x, ColorSpinorField &y)
    {
      return instantiateReduce<AxpyReDot, false>(a, 0.0, 0.0, x, y, x, x, x);
    }

    double caxpyNorm(const Complex &a, ColorSpinorField &x, ColorSpinorField &y)
    {
      return instantiateReduce<caxpyNorm2, false>(a, Complex(0.0), Complex(0.0), x, y, x, x, x);
    }

    double caxpyXmazNormX(const Complex &a, ColorSpinorField &x, ColorSpinorField &y, ColorSpinorField &z)
    {
      return instantiateReduce<caxpyxmaznormx, false>(a, Complex(0.0), Complex(0.0), x, y, z, x, x);
    }

    double cabxpyzAxNorm(double a, const Complex &b, ColorSpinorField &x, ColorSpinorField &y, ColorSpinorField &z)
    {
      return instantiateReduce<cabxpyzaxnorm, false>(Complex(a), b, Complex(0.0), x, y, z, x, x);
    }

    Complex cDotProduct(ColorSpinorField &x, ColorSpinorField &y)
    {
      auto cdot = instantiateReduce<Cdot, false>(0.0, 0.0, 0.0, x, y, x, x, x);
      return Complex(cdot.x, cdot.y);
    }

    Complex caxpyDotzy(const Complex &a, ColorSpinorField &x, ColorSpinorField &y, ColorSpinorField &z)
    {
      auto cdot = instantiateReduce<caxpydotzy, false>(a, Complex(0.0), Complex(0.0), x, y, z, x, x);
      return Complex(cdot.x, cdot.y);
    }

    double3 cDotProductNormA(ColorSpinorField &x, ColorSpinorField &y)
    {
      return instantiateReduce<CdotNormA, false>(0.0, 0.0, 0.0, x, y, x, x, x);
    }

    double3 caxpbypzYmbwcDotProductUYNormY(const Complex &a, ColorSpinorField &x, const Complex &b, ColorSpinorField &y,
                                           ColorSpinorField &z, ColorSpinorField &w, ColorSpinorField &u)
    {
      return instantiateReduce<caxpbypzYmbwcDotProductUYNormY_, true>(a, b, Complex(0.0), x, z, y, w, u);
    }

    Complex axpyCGNorm(double a, ColorSpinorField &x, ColorSpinorField &y)
    {
      double2 cg_norm = instantiateReduce<axpyCGNorm2, true>(a, 0.0, 0.0, x, y, x, x, x);
      return Complex(cg_norm.x, cg_norm.y);
    }

    double3 HeavyQuarkResidualNorm(ColorSpinorField &x, ColorSpinorField &r)
    {
      // in case of x.Ncolor()!=3 (MG mainly) reduce_core do not support this function.
      if (x.Ncolor() != 3) return make_double3(0.0, 0.0, 0.0);
      double3 rtn = instantiateReduce<HeavyQuarkResidualNorm_, false>(0.0, 0.0, 0.0, x, r, r, r, r);
      rtn.z /= (x.Volume()*comm_size());
      return rtn;
    }

    double3 xpyHeavyQuarkResidualNorm(ColorSpinorField &x, ColorSpinorField &y, ColorSpinorField &r)
    {
      // in case of x.Ncolor()!=3 (MG mainly) reduce_core do not support this function.
      if (x.Ncolor()!=3) return make_double3(0.0, 0.0, 0.0);
      double3 rtn = instantiateReduce<xpyHeavyQuarkResidualNorm_, false>(0.0, 0.0, 0.0, x, y, r, r, r);
      rtn.z /= (x.Volume()*comm_size());
      return rtn;
    }

    double3 tripleCGReduction(ColorSpinorField &x, ColorSpinorField &y, ColorSpinorField &z)
    {
      return instantiateReduce<tripleCGReduction_, false>(0.0, 0.0, 0.0, x, y, z, x, x);
    }

    double4 quadrupleCGReduction(ColorSpinorField &x, ColorSpinorField &y, ColorSpinorField &z)
    {
      return instantiateReduce<quadrupleCGReduction_, false>(0.0, 0.0, 0.0, x, y, z, x, x);
    }

    double quadrupleCG3InitNorm(double a, ColorSpinorField &x, ColorSpinorField &y,
                                ColorSpinorField &z, ColorSpinorField &w, ColorSpinorField &v)
    {
      return instantiateReduce<quadrupleCG3InitNorm_, false>(a, 0.0, 0.0, x, y, z, w, v);
    }

    double quadrupleCG3UpdateNorm(double a, double b, ColorSpinorField &x, ColorSpinorField &y,
                                  ColorSpinorField &z, ColorSpinorField &w, ColorSpinorField &v)
    {
      return instantiateReduce<quadrupleCG3UpdateNorm_, false>(a, b, 0.0, x, y, z, w, v);
    }

  } // namespace blas

} // namespace quda
