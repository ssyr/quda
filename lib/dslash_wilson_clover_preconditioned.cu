#include <gauge_field.h>
#include <color_spinor_field.h>
#include <clover_field.h>
#include <dslash.h>
#include <worker.h>

#include <dslash_policy.cuh>
#include <kernels/dslash_wilson_clover_preconditioned.cuh>

/**
   This is the Wilson-clover preconditioned linear operator
*/

namespace quda
{

  /**
     @brief This is a helper class that is used to instantiate the
     correct templated kernel for the dslash.
   */
  template <typename Float, int nDim, int nColor, int nParity, bool dagger, bool xpay, KernelType kernel_type, typename Arg>
  struct WilsonCloverPreconditionedLaunch {
    static constexpr const char *kernel = "quda::wilsonCloverPreconditionedGPU"; // kernel name for jit compilation
    template <typename Dslash>
    inline static void launch(Dslash &dslash, TuneParam &tp, Arg &arg, const hipStream_t &stream)
    {
      static_assert(nParity == 1, "preconditioned wilson-clover operator only defined for nParity=1");
      if (xpay && dagger) errorQuda("xpay operator only defined for not dagger");
      dslash.launch(wilsonCloverPreconditionedGPU < Float, nDim, nColor, nParity, dagger && !xpay, xpay && !dagger,
          kernel_type, Arg >, tp, arg, stream);
    }
  };

  template <typename Float, int nDim, int nColor, typename Arg> class WilsonCloverPreconditioned : public Dslash<Float>
  {

protected:
    Arg &arg;
    const ColorSpinorField &in;

public:
    WilsonCloverPreconditioned(Arg &arg, const ColorSpinorField &out, const ColorSpinorField &in) :
        Dslash<Float>(arg, out, in, "kernels/dslash_wilson_clover_preconditioned.cuh"),
        arg(arg),
        in(in)
    {
    }

    virtual ~WilsonCloverPreconditioned() {}

    void apply(const hipStream_t &stream)
    {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      Dslash<Float>::setParam(arg);
      if (arg.nParity == 1) {
        if (arg.xpay)
          Dslash<Float>::template instantiate<WilsonCloverPreconditionedLaunch, nDim, nColor, 1, true>(tp, arg, stream);
        else
          Dslash<Float>::template instantiate<WilsonCloverPreconditionedLaunch, nDim, nColor, 1, false>(tp, arg, stream);
      } else {
        errorQuda("Preconditioned Wilson-clover operator not defined nParity=%d", arg.nParity);
      }
    }

    long long flops() const
    {
      int clover_flops = 504;
      long long flops = Dslash<Float>::flops();
      switch (arg.kernel_type) {
      case EXTERIOR_KERNEL_X:
      case EXTERIOR_KERNEL_Y:
      case EXTERIOR_KERNEL_Z:
      case EXTERIOR_KERNEL_T: flops += clover_flops * 2 * in.GhostFace()[arg.kernel_type]; break;
      case EXTERIOR_KERNEL_ALL:
        flops += clover_flops * 2 * (in.GhostFace()[0] + in.GhostFace()[1] + in.GhostFace()[2] + in.GhostFace()[3]);
        break;
      case INTERIOR_KERNEL:
      case KERNEL_POLICY:
        flops += clover_flops * in.Volume();

        if (arg.kernel_type == KERNEL_POLICY) break;
        // now correct for flops done by exterior kernel
        long long ghost_sites = 0;
        for (int d = 0; d < 4; d++)
          if (arg.commDim[d]) ghost_sites += 2 * in.GhostFace()[d];
        flops -= clover_flops * ghost_sites;

        break;
      }
      return flops;
    }

    long long bytes() const
    {
      bool isFixed = (in.Precision() == sizeof(short) || in.Precision() == sizeof(char)) ? true : false;
      int clover_bytes = 72 * in.Precision() + (isFixed ? 2 * sizeof(float) : 0);

      long long bytes = Dslash<Float>::bytes();
      switch (arg.kernel_type) {
      case EXTERIOR_KERNEL_X:
      case EXTERIOR_KERNEL_Y:
      case EXTERIOR_KERNEL_Z:
      case EXTERIOR_KERNEL_T: bytes += clover_bytes * 2 * in.GhostFace()[arg.kernel_type]; break;
      case EXTERIOR_KERNEL_ALL:
        bytes += clover_bytes * 2 * (in.GhostFace()[0] + in.GhostFace()[1] + in.GhostFace()[2] + in.GhostFace()[3]);
        break;
      case INTERIOR_KERNEL:
      case KERNEL_POLICY:
        bytes += clover_bytes * in.Volume();

        if (arg.kernel_type == KERNEL_POLICY) break;
        // now correct for bytes done by exterior kernel
        long long ghost_sites = 0;
        for (int d = 0; d < 4; d++)
          if (arg.commDim[d]) ghost_sites += 2 * in.GhostFace()[d];
        bytes -= clover_bytes * ghost_sites;

        break;
      }

      return bytes;
    }

    TuneKey tuneKey() const
    {
      return TuneKey(in.VolString(), typeid(*this).name(), Dslash<Float>::aux[arg.kernel_type]);
    }
  };

  template <typename Float, int nColor, QudaReconstructType recon> struct WilsonCloverPreconditionedApply {

    inline WilsonCloverPreconditionedApply(ColorSpinorField &out, const ColorSpinorField &in, const GaugeField &U,
        const CloverField &A, double a, const ColorSpinorField &x, int parity, bool dagger, const int *comm_override,
        TimeProfile &profile)
    {
      constexpr int nDim = 4;
#ifdef DYNAMIC_CLOVER
      constexpr bool dynamic_clover = true;
#else
      constexpr bool dynamic_clover = false;
#endif
      WilsonCloverArg<Float, nColor, recon, dynamic_clover> arg(out, in, U, A, a, x, parity, dagger, comm_override);
      WilsonCloverPreconditioned<Float, nDim, nColor, WilsonCloverArg<Float, nColor, recon, dynamic_clover>> wilson(
          arg, out, in);

      dslash::DslashPolicyTune<decltype(wilson)> policy(wilson,
          const_cast<cudaColorSpinorField *>(static_cast<const cudaColorSpinorField *>(&in)), in.VolumeCB(),
          in.GhostFaceCB(), profile);
      policy.apply(0);

      checkCudaError();
    }
  };

  // Apply the preconditioned Wilson-clover operator
  // out(x) = M*in = a * A(x)^{-1} (\sum_mu U_{-\mu}(x)in(x+mu) + U^\dagger_mu(x-mu)in(x-mu))
  // Uses the kappa normalization for the Wilson operator.
  void ApplyWilsonCloverPreconditioned(ColorSpinorField &out, const ColorSpinorField &in, const GaugeField &U,
      const CloverField &A, double a, const ColorSpinorField &x, int parity, bool dagger, const int *comm_override,
      TimeProfile &profile)
  {
#ifdef GPU_CLOVER_DIRAC
    if (in.V() == out.V()) errorQuda("Aliasing pointers");
    if (in.FieldOrder() != out.FieldOrder())
      errorQuda("Field order mismatch in = %d, out = %d", in.FieldOrder(), out.FieldOrder());

    // check all precisions match
    checkPrecision(out, in, U, A);

    // check all locations match
    checkLocation(out, in, U, A);

    instantiate<WilsonCloverPreconditionedApply>(out, in, U, A, a, x, parity, dagger, comm_override, profile);
#else
    errorQuda("Clover dslash has not been built");
#endif
  }

} // namespace quda
