#include "hip/hip_runtime.h"
#include <blas_quda.h>
#include <tune_quda.h>
#include <float_vector.h>
#include <color_spinor_field_order.h>
#include <uint_to_char.h>

#include <launch_kernel.cuh>
#include <jitify_helper.cuh>
#include <kernels/multi_reduce_core.cuh>

// work around for Fermi
#if (__COMPUTE_CAPABILITY__ < 300)
#undef MAX_MULTI_BLAS_N
#define MAX_MULTI_BLAS_N 2
#endif

namespace quda {

  namespace blas {

    hipStream_t* getStream();
    hipEvent_t* getReduceEvent();
    bool getFastReduce();
    void initFastReduce(int words);
    void completeFastReduce(int32_t words);

    template <int writeX, int writeY, int writeZ, int writeW>
    struct write {
      static constexpr int X = writeX;
      static constexpr int Y = writeY;
      static constexpr int Z = writeZ;
      static constexpr int W = writeW;
    };

    template <typename doubleN, typename ReduceType, typename FloatN, int M, int NXZ, typename Arg>
    void multiReduceLaunch(doubleN result[], Arg &arg, const TuneParam &tp, const hipStream_t &stream, Tunable &tunable)
    {

      if (tp.grid.x > (unsigned int)deviceProp.maxGridSize[0])
        errorQuda("Grid size %d greater than maximum %d\n", tp.grid.x, deviceProp.maxGridSize[0]);

      const int32_t words = tp.grid.z * NXZ * arg.NYW * sizeof(ReduceType) / sizeof(int32_t);
      if (getFastReduce() && !commAsyncReduction()) initFastReduce(words);

#ifdef WARP_MULTI_REDUCE
#error "Untested - should be reverified"
      // multiReduceKernel<ReduceType,FloatN,M,NXZ><<<tp.grid,tp.block,tp.shared_bytes>>>(arg);
#else
#ifdef JITIFY
      using namespace jitify::reflection;
      tunable.jitifyError() = program->kernel("quda::blas::multiReduceKernel")
                                  .instantiate((int)tp.block.x, Type<ReduceType>(), Type<FloatN>(), M, NXZ, Type<Arg>())
                                  .configure(tp.grid, tp.block, tp.shared_bytes, stream)
                                  .launch(arg);
#else
#if CUDA_VERSION < 9000
      hipMemcpyToSymbolAsync(HIP_SYMBOL(arg_buffer), reinterpret_cast<char *>(&arg), sizeof(arg), 0, hipMemcpyHostToDevice,
                              *getStream());
#endif
      LAUNCH_KERNEL_LOCAL_PARITY(multiReduceKernel, tp, stream, arg, ReduceType, FloatN, M, NXZ);
#endif
#endif

      if (!commAsyncReduction()) {
#if (defined(_MSC_VER) && defined(_WIN64) || defined(__LP64__))
        if (deviceProp.canMapHostMemory) {
          if (getFastReduce()) {
            completeFastReduce(words);
          } else {
            qudaEventRecord(*getReduceEvent(), stream);
            while (hipSuccess != qudaEventQuery(*getReduceEvent())) {}
          }
        } else
#endif
        {
          qudaMemcpy(getHostReduceBuffer(), getMappedHostReduceBuffer(), tp.grid.z * sizeof(ReduceType) * NXZ * arg.NYW,
              hipMemcpyDeviceToHost);
        }
      }

      // need to transpose for same order with vector thread reduction
      for (int i = 0; i < NXZ; i++) {
        for (int j = 0; j < arg.NYW; j++) {
          result[i * arg.NYW + j] = set(((ReduceType *)getHostReduceBuffer())[j * NXZ + i]);
          if (tp.grid.z == 2)
            sum(result[i * arg.NYW + j], ((ReduceType *)getHostReduceBuffer())[NXZ * arg.NYW + j * NXZ + i]);
        }
      }
    }

    namespace detail
    {
      template <unsigned... digits> struct to_chars {
        static const char value[];
      };

      template <unsigned... digits> const char to_chars<digits...>::value[] = {('0' + digits)..., 0};

      template <unsigned rem, unsigned... digits> struct explode : explode<rem / 10, rem % 10, digits...> {
      };

      template <unsigned... digits> struct explode<0, digits...> : to_chars<digits...> {
      };
    } // namespace detail

    template <unsigned num> struct num_to_string : detail::explode<num / 10, num % 10> {
    };

    template <int NXZ, typename doubleN, typename ReduceType, typename FloatN, int M, typename SpinorX,
        typename SpinorY, typename SpinorZ, typename SpinorW, typename Reducer>
    class MultiReduceCuda : public Tunable
    {

  private:
      const int NYW;
      int nParity;
      MultiReduceArg<NXZ, ReduceType, SpinorX, SpinorY, SpinorZ, SpinorW, Reducer> arg;
      doubleN *result;

      std::vector<ColorSpinorField *> &x, &y, &z, &w;

      // don't curry into the Spinors to minimize parameter size
      char *Y_h[MAX_MULTI_BLAS_N], *W_h[MAX_MULTI_BLAS_N], *Ynorm_h[MAX_MULTI_BLAS_N], *Wnorm_h[MAX_MULTI_BLAS_N];

      unsigned int sharedBytesPerThread() const { return 0; }
      unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }

      virtual bool advanceSharedBytes(TuneParam &param) const
      {
        TuneParam next(param);
        advanceBlockDim(next); // to get next blockDim
        int nthreads = next.block.x * next.block.y * next.block.z;
        param.shared_bytes = sharedBytesPerThread() * nthreads > sharedBytesPerBlock(param) ?
            sharedBytesPerThread() * nthreads :
            sharedBytesPerBlock(param);
        return false;
      }

      // we only launch thread blocks up to size 512 since the autoner
      // tuner favours smaller blocks and this helps with compile time
      unsigned int maxBlockSize(const TuneParam &param) const { return deviceProp.maxThreadsPerBlock / 2; }

  public:
      MultiReduceCuda(doubleN result[], SpinorX X[], SpinorY Y[], SpinorZ Z[], SpinorW W[], Reducer &r,
          std::vector<ColorSpinorField *> &x, std::vector<ColorSpinorField *> &y, std::vector<ColorSpinorField *> &z,
          std::vector<ColorSpinorField *> &w, int NYW, int length) :
          NYW(NYW),
          nParity(x[0]->SiteSubset()),
          arg(X, Y, Z, W, r, NYW, length / nParity),
          x(x),
          y(y),
          z(z),
          w(w),
          result(result),
          Y_h(),
          W_h(),
          Ynorm_h(),
          Wnorm_h()
      {
        strcpy(aux, "policy_kernel,");
        strcat(aux, x[0]->AuxString());
        if (getFastReduce()) strcat(aux, ",fast_reduce");

        // since block dot product and block norm use the same functors, we need to distinguish them
        bool is_norm = false;
        if (NXZ == NYW) {
          is_norm = true;
          for (int i = 0; i < NXZ; i++) {
            if (x[i]->V() != y[i]->V() || x[i]->V() != z[i]->V() || x[i]->V() != w[i]->V()) {
              is_norm = false;
              break;
            }
          }
        }
        if (is_norm) strcat(aux, ",norm");

#ifdef JITIFY
        ::quda::create_jitify_program("kernels/multi_reduce_core.cuh");
#endif
      }

      inline TuneKey tuneKey() const
      {
        char name[TuneKey::name_n];
        strcpy(name, num_to_string<NXZ>::value);
        strcat(name, std::to_string(NYW).c_str());
        strcat(name, typeid(arg.r).name());
        return TuneKey(x[0]->VolString(), name, aux);
      }

      void apply(const hipStream_t &stream)
      {
        TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
        multiReduceLaunch<doubleN, ReduceType, FloatN, M, NXZ>(result, arg, tp, stream, *this);
      }

      // Should these be NYW?
#ifdef WARP_MULTI_REDUCE
      /**
         @brief This is a specialized variant of the reducer that only
         assigns an individial warp within a thread block to a given row
         of the reduction.  It's typically slower than CTA-wide reductions
         and spreading the y dimension across blocks rather then within
         the blocks so left disabled.
      */
      bool advanceBlockDim(TuneParam &param) const
      {
        if (param.block.y < NYW) {
          param.block.y++;
          param.grid.y = (NYW + param.block.y - 1) / param.block.y;
          return true;
        } else {
          param.block.y = 1;
          param.grid.y = NYW;
          return false;
        }
      }
#endif

      bool advanceGridDim(TuneParam &param) const
      {
        bool rtn = Tunable::advanceGridDim(param);
        if (NYW > deviceProp.maxGridSize[1]) errorQuda("N=%d is greater than the maximum support grid size", NYW);
        return rtn;
      }

      void initTuneParam(TuneParam &param) const
      {
        Tunable::initTuneParam(param);
        param.block.y = 1;
        param.grid.y = NYW;
        param.grid.z = nParity;
      }

      void defaultTuneParam(TuneParam &param) const
      {
        Tunable::defaultTuneParam(param);
        param.block.y = 1;
        param.grid.y = NYW;
        param.grid.z = nParity;
      }

      void preTune()
      {
        for (int i = 0; i < NYW; ++i) {
          arg.Y[i].backup(&Y_h[i], &Ynorm_h[i], y[i]->Bytes(), y[i]->NormBytes());
          arg.W[i].backup(&W_h[i], &Wnorm_h[i], w[i]->Bytes(), w[i]->NormBytes());
        }
      }

      void postTune()
      {
        for (int i = 0; i < NYW; ++i) {
          arg.Y[i].restore(&Y_h[i], &Ynorm_h[i], y[i]->Bytes(), y[i]->NormBytes());
          arg.W[i].restore(&W_h[i], &Wnorm_h[i], w[i]->Bytes(), w[i]->NormBytes());
        }
      }

      long long flops() const
      {
        return NYW * NXZ * arg.r.flops() * vec_length<FloatN>::value * (long long)arg.length * nParity * M;
      }

      long long bytes() const
      {
        // this will be wrong when mixed precision is added
        return NYW * NXZ * arg.r.streams() * x[0]->Bytes();
      }

      int tuningIter() const { return 3; }
    };

    template <typename doubleN, typename ReduceType, typename RegType, typename StoreType, typename yType, int M, int NXZ,
        template <int MXZ, typename ReducerType, typename Float, typename FloatN> class Reducer, typename write, typename T>
    void multiReduce(doubleN result[], const coeff_array<T> &a, const coeff_array<T> &b, const coeff_array<T> &c,
        std::vector<ColorSpinorField *> &x, std::vector<ColorSpinorField *> &y, std::vector<ColorSpinorField *> &z,
        std::vector<ColorSpinorField *> &w, int length)
    {

      const int NYW = y.size();

      memset(result, 0, NXZ * NYW * sizeof(doubleN));

      const int N_MAX = NXZ > NYW ? NXZ : NYW;
      const int N_MIN = NXZ < NYW ? NXZ : NYW;

      static_assert(MAX_MULTI_BLAS_N * MAX_MULTI_BLAS_N <= QUDA_MAX_MULTI_REDUCE,
          "MAX_MULTI_BLAS_N^2 exceeds maximum number of reductions");
      static_assert(MAX_MULTI_BLAS_N <= 16, "MAX_MULTI_BLAS_N exceeds maximum size 16");
      if (N_MAX > MAX_MULTI_BLAS_N)
        errorQuda("Spinor vector length exceeds max size (%d > %d)", N_MAX, MAX_MULTI_BLAS_N);

      if (NXZ * NYW * sizeof(Complex) > MAX_MATRIX_SIZE)
        errorQuda("A matrix exceeds max size (%lu > %d)", NXZ * NYW * sizeof(Complex), MAX_MATRIX_SIZE);

      for (int i = 0; i < N_MIN; i++) {
        checkSpinor(*x[i], *y[i]);
        checkSpinor(*x[i], *z[i]);
        checkSpinor(*x[i], *w[i]);
        if (!x[i]->isNative()) {
          warningQuda("Reductions on non-native fields are not supported\n");
          return;
        }
      }

      typedef typename scalar<RegType>::type Float;
      typedef typename vector<Float, 2>::type Float2;
      typedef vector<Float, 2> vec2;

#ifdef JITIFY
      // need to get constants pointer from jitify instance
      if (a.use_const || b.use_const || c.use_const)
        errorQuda("Constant memory buffer support not enabled with jitify yet");
#endif

      // FIXME - if NXZ=1 no need to copy entire array
      // FIXME - do we really need strided access here?
      if (a.data && a.use_const) {
        Float2 A[MAX_MATRIX_SIZE / sizeof(Float2)];
        // since the kernel doesn't know the width of them matrix at compile
        // time we stride it and copy the padded matrix to GPU
        for (int i = 0; i < NXZ; i++)
          for (int j = 0; j < NYW; j++) A[MAX_MULTI_BLAS_N * i + j] = make_Float2<Float2>(Complex(a.data[NYW * i + j]));

        hipMemcpyToSymbolAsync(HIP_SYMBOL(Amatrix_d), A, MAX_MATRIX_SIZE, 0, hipMemcpyHostToDevice, *getStream());
        Amatrix_h = reinterpret_cast<signed char *>(const_cast<T *>(a.data));
      }

      if (b.data && b.use_const) {
        Float2 B[MAX_MATRIX_SIZE / sizeof(Float2)];
        // since the kernel doesn't know the width of them matrix at compile
        // time we stride it and copy the padded matrix to GPU
        for (int i = 0; i < NXZ; i++)
          for (int j = 0; j < NYW; j++) B[MAX_MULTI_BLAS_N * i + j] = make_Float2<Float2>(Complex(b.data[NYW * i + j]));

        hipMemcpyToSymbolAsync(HIP_SYMBOL(Bmatrix_d), B, MAX_MATRIX_SIZE, 0, hipMemcpyHostToDevice, *getStream());
        Bmatrix_h = reinterpret_cast<signed char *>(const_cast<T *>(b.data));
      }

      if (c.data && c.use_const) {
        Float2 C[MAX_MATRIX_SIZE / sizeof(Float2)];
        // since the kernel doesn't know the width of them matrix at compile
        // time we stride it and copy the padded matrix to GPU
        for (int i = 0; i < NXZ; i++)
          for (int j = 0; j < NYW; j++) C[MAX_MULTI_BLAS_N * i + j] = make_Float2<Float2>(Complex(c.data[NYW * i + j]));

        hipMemcpyToSymbolAsync(HIP_SYMBOL(Cmatrix_d), C, MAX_MATRIX_SIZE, 0, hipMemcpyHostToDevice, *getStream());
        Cmatrix_h = reinterpret_cast<signed char *>(const_cast<T *>(c.data));
      }

      SpinorTexture<RegType, StoreType, M> X[NXZ];
      Spinor<RegType, yType, M, write::Y> Y[MAX_MULTI_BLAS_N];
      SpinorTexture<RegType, StoreType, M> Z[NXZ];
      Spinor<RegType, StoreType, M, write::W> W[MAX_MULTI_BLAS_N];

      for (int i = 0; i < NXZ; i++) {
        X[i].set(*dynamic_cast<cudaColorSpinorField *>(x[i]));
        Z[i].set(*dynamic_cast<cudaColorSpinorField *>(z[i]));
      }
      for (int i = 0; i < NYW; i++) {
        Y[i].set(*dynamic_cast<cudaColorSpinorField *>(y[i]));
        W[i].set(*dynamic_cast<cudaColorSpinorField *>(w[i]));
      }

      Reducer<NXZ, ReduceType, Float2, RegType> r(a, b, c, NYW);

      MultiReduceCuda<NXZ, doubleN, ReduceType, RegType, M, SpinorTexture<RegType, StoreType, M>,
                      Spinor<RegType, yType, M, write::Y>, SpinorTexture<RegType, StoreType, M>,
                      Spinor<RegType, StoreType, M, write::W>, decltype(r)>
        reduce(result, X, Y, Z, W, r, x, y, z, w, NYW, length);
      reduce.apply(*blas::getStream());

      blas::bytes += reduce.bytes();
      blas::flops += reduce.flops();

      checkCudaError();
    }

    /**
       Driver for multi-reduce with up to four vectors
    */
    template <int NXZ, typename doubleN, typename ReduceType,
        template <int MXZ, typename ReducerType, typename Float, typename FloatN> class Reducer, typename write,
        bool siteUnroll, typename T>
    void multiReduce(doubleN result[], const coeff_array<T> &a, const coeff_array<T> &b, const coeff_array<T> &c,
        CompositeColorSpinorField &x, CompositeColorSpinorField &y, CompositeColorSpinorField &z,
        CompositeColorSpinorField &w)
    {
      const int NYW = y.size();

      int reduce_length = siteUnroll ? x[0]->RealLength() : x[0]->Length();

      QudaPrecision precision = checkPrecision(*x[0], *y[0], *z[0], *w[0]);

      if (precision == QUDA_DOUBLE_PRECISION) {

#if QUDA_PRECISION & 8
        if (x[0]->Nspin() == 4 || x[0]->Nspin() == 2) { // wilson
#if defined(GPU_WILSON_DIRAC) || defined(GPU_DOMAIN_WALL_DIRAC) || defined(GPU_MULTIGRID)
          const int M = siteUnroll ? 12 : 1; // determines how much work per thread to do
          if (x[0]->Nspin() == 2 && siteUnroll) errorQuda("siteUnroll not supported for nSpin==2");
          multiReduce<doubleN, ReduceType, double2, double2, double2, M, NXZ, Reducer, write>(
              result, a, b, c, x, y, z, w, reduce_length / (2 * M));
#else
          errorQuda("blas has not been built for Nspin=%d fields", x[0]->Nspin());
#endif
        } else if (x[0]->Nspin() == 1) {
#ifdef GPU_STAGGERED_DIRAC
          const int M = siteUnroll ? 3 : 1; // determines how much work per thread to do
          multiReduce<doubleN, ReduceType, double2, double2, double2, M, NXZ, Reducer, write>(
              result, a, b, c, x, y, z, w, reduce_length / (2 * M));
#else
          errorQuda("blas has not been built for Nspin=%d field", x[0]->Nspin());
#endif
        } else {
          errorQuda("nSpin=%d is not supported\n", x[0]->Nspin());
        }
#else
        errorQuda("QUDA_PRECISION=%d does not enable precision %d", QUDA_PRECISION, precision);
#endif

      } else if (precision == QUDA_SINGLE_PRECISION) {

#if QUDA_PRECISION & 4
        if (x[0]->Nspin() == 4) { // wilson
#if defined(GPU_WILSON_DIRAC) || defined(GPU_DOMAIN_WALL_DIRAC)
          const int M = siteUnroll ? 6 : 1; // determines how much work per thread to do
          multiReduce<doubleN, ReduceType, float4, float4, float4, M, NXZ, Reducer, write>(
              result, a, b, c, x, y, z, w, reduce_length / (4 * M));
#else
          errorQuda("blas has not been built for Nspin=%d fields", x[0]->Nspin());
#endif
        } else if (x[0]->Nspin() == 1 || x[0]->Nspin() == 2) { // staggered
#if defined(GPU_STAGGERED_DIRAC) || defined(GPU_MULTIGRID)
          const int M = siteUnroll ? 3 : 1;
          if (x[0]->Nspin() == 2 && siteUnroll) errorQuda("siteUnroll not supported for nSpin==2");
          multiReduce<doubleN, ReduceType, float2, float2, float2, M, NXZ, Reducer, write>(
              result, a, b, c, x, y, z, w, reduce_length / (2 * M));
#else
          errorQuda("blas has not been built for Nspin=%d fields", x[0]->Nspin());
#endif
        } else {
          errorQuda("nSpin=%d is not supported\n", x[0]->Nspin());
        }
#else
        errorQuda("QUDA_PRECISION=%d does not enable precision %d", QUDA_PRECISION, precision);
#endif

      } else if (precision == QUDA_HALF_PRECISION) { // half precision

#if QUDA_PRECISION & 2
        if (x[0]->Nspin() == 4) { // wilson
#if defined(GPU_WILSON_DIRAC) || defined(GPU_DOMAIN_WALL_DIRAC)
          const int M = 6;
          multiReduce<doubleN, ReduceType, float4, short4, short4, M, NXZ, Reducer, write>(
              result, a, b, c, x, y, z, w, x[0]->Volume());
#else
          errorQuda("blas has not been built for Nspin=%d fields", x[0]->Nspin());
#endif
        } else if (x[0]->Nspin() == 1) { // staggered
#ifdef GPU_STAGGERED_DIRAC
          const int M = 3;
          multiReduce<doubleN, ReduceType, float2, short2, short2, M, NXZ, Reducer, write>(
              result, a, b, c, x, y, z, w, x[0]->Volume());
#else
          errorQuda("blas has not been built for Nspin=%d fields", x[0]->Nspin());
#endif
        } else {
          errorQuda("nSpin=%d is not supported\n", x[0]->Nspin());
        }
#else
        errorQuda("QUDA_PRECISION=%d does not enable precision %d", QUDA_PRECISION, precision);
#endif

      } else if (precision == QUDA_QUARTER_PRECISION) { // quarter precision

#if QUDA_PRECISION & 1
        if (x[0]->Nspin() == 4) { // wilson
#if defined(GPU_WILSON_DIRAC) || defined(GPU_DOMAIN_WALL_DIRAC)
          const int M = 6;
          multiReduce<doubleN, ReduceType, float4, char4, char4, M, NXZ, Reducer, write>(
              result, a, b, c, x, y, z, w, x[0]->Volume());
#else
          errorQuda("blas has not been built for Nspin=%d fields", x[0]->Nspin());
#endif
        } else if (x[0]->Nspin() == 1) { // staggered
#ifdef GPU_STAGGERED_DIRAC
          const int M = 3;
          multiReduce<doubleN, ReduceType, float2, char2, char2, M, NXZ, Reducer, write>(
              result, a, b, c, x, y, z, w, x[0]->Volume());
#else
          errorQuda("blas has not been built for Nspin=%d fields", x[0]->Nspin());
#endif
        } else {
          errorQuda("nSpin=%d is not supported\n", x[0]->Nspin());
        }
#else
        errorQuda("QUDA_PRECISION=%d does not enable precision %d", QUDA_PRECISION, precision);
#endif
      } else {
        errorQuda("Precision %d not supported\n", precision);
      }
    }

    /**
       Driver for multi-reduce with up to five vectors
    */
    template <int NXZ, typename doubleN, typename ReduceType,
        template <int MXZ, typename ReducerType, typename Float, typename FloatN> class Reducer, typename write,
        bool siteUnroll, typename T>
    void mixedMultiReduce(doubleN result[], const coeff_array<T> &a, const coeff_array<T> &b, const coeff_array<T> &c,
        CompositeColorSpinorField &x, CompositeColorSpinorField &y, CompositeColorSpinorField &z,
        CompositeColorSpinorField &w)
    {
      const int NYW = y.size();

      checkPrecision(*x[0], *z[0]);
      checkPrecision(*y[0], *w[0]);

      assert(siteUnroll == true);
      int reduce_length = siteUnroll ? x[0]->RealLength() : x[0]->Length();

      if (y[0]->Precision() == QUDA_DOUBLE_PRECISION && x[0]->Precision() == QUDA_SINGLE_PRECISION) {

        if (x[0]->Nspin() == 4) { // wilson
#if defined(GPU_WILSON_DIRAC) || defined(GPU_DOMAIN_WALL_DIRAC)
          const int M = 12; // determines how much work per thread to do
          multiReduce<doubleN, ReduceType, double2, float4, double2, M, NXZ, Reducer, write>(
              result, a, b, c, x, y, z, w, reduce_length / (2 * M));
#else
          errorQuda("blas has not been built for Nspin=%d fields", x[0]->Nspin());
#endif
        } else if (x[0]->Nspin() == 1) {
#ifdef GPU_STAGGERED_DIRAC
          const int M = 3; // determines how much work per thread to do
          multiReduce<doubleN, ReduceType, double2, float2, double2, M, NXZ, Reducer, write>(
              result, a, b, c, x, y, z, w, reduce_length / (2 * M));
#else
          errorQuda("blas has not been built for Nspin=%d field", x[0]->Nspin());
#endif
        } else {
          errorQuda("nSpin=%d is not supported\n", x[0]->Nspin());
        }

      } else if (y[0]->Precision() == QUDA_DOUBLE_PRECISION && x[0]->Precision() == QUDA_HALF_PRECISION) {

        if (x[0]->Nspin() == 4) { // wilson
#if defined(GPU_WILSON_DIRAC) || defined(GPU_DOMAIN_WALL_DIRAC)
          const int M = 6; // determines how much work per thread to do
          multiReduce<doubleN, ReduceType, double2, short4, double2, M, NXZ, Reducer, write>(
              result, a, b, c, x, y, z, w, reduce_length / (4 * M));
#else
          errorQuda("blas has not been built for Nspin=%d fields", x[0]->Nspin());
#endif
        } else if (x[0]->Nspin() == 1 || x[0]->Nspin() == 2) { // staggered
#if defined(GPU_STAGGERED_DIRAC)
          const int M = 3;
          multiReduce<doubleN, ReduceType, double2, short2, double2, M, NXZ, Reducer, write>(
              result, a, b, c, x, y, z, w, reduce_length / (2 * M));
#else
          errorQuda("blas has not been built for Nspin=%d fields", x[0]->Nspin());
#endif
        } else {
          errorQuda("nSpin=%d is not supported\n", x[0]->Nspin());
        }

      } else if (y[0]->Precision() == QUDA_SINGLE_PRECISION && x[0]->Precision() == QUDA_HALF_PRECISION) {

        if (x[0]->Nspin() == 4) { // wilson
#if defined(GPU_WILSON_DIRAC) || defined(GPU_DOMAIN_WALL_DIRAC)
          const int M = 6;
          multiReduce<doubleN, ReduceType, float4, short4, float4, M, NXZ, Reducer, write>(
              result, a, b, c, x, y, z, w, x[0]->Volume());
#else
          errorQuda("blas has not been built for Nspin=%d fields", x[0]->Nspin());
#endif
        } else if (x[0]->Nspin() == 1) { // staggered
#ifdef GPU_STAGGERED_DIRAC
          const int M = 3;
          multiReduce<doubleN, ReduceType, float2, short2, float2, M, NXZ, Reducer, write>(
              result, a, b, c, x, y, z, w, x[0]->Volume());
#else
          errorQuda("blas has not been built for Nspin=%d fields", x[0]->Nspin());
#endif
        } else {
          errorQuda("nSpin=%d is not supported\n", x[0]->Nspin());
        }

      } else {
        errorQuda("Precision combination x=%d y=%d not supported\n", x[0]->Precision(), y[0]->Precision());
      }
    }

    template <int NXZ, typename doubleN, typename ReduceType,
        template <int MXZ, typename ReducerType, typename Float, typename FloatN> class ReducerDiagonal, typename writeDiagonal,
        template <int MXZ, typename ReducerType, typename Float, typename FloatN> class ReducerOffDiagonal,
        typename writeOffDiagonal, bool siteUnroll, typename T>
    void multiReduce(doubleN result[], const coeff_array<T> &a, const coeff_array<T> &b, const coeff_array<T> &c,
        CompositeColorSpinorField &x, CompositeColorSpinorField &y, CompositeColorSpinorField &z,
        CompositeColorSpinorField &w, int i, int j)
    {

      if (x[0]->Precision() == y[0]->Precision()) {
        if (i == j) { // we are on the diagonal so invoke the diagonal reducer
          multiReduce<NXZ, doubleN, ReduceType, ReducerDiagonal, writeDiagonal, siteUnroll, T>(
              result, a, b, c, x, y, z, w);
        } else { // we are on the diagonal so invoke the off-diagonal reducer
          multiReduce<NXZ, doubleN, ReduceType, ReducerOffDiagonal, writeOffDiagonal, siteUnroll, T>(
              result, a, b, c, x, y, z, w);
        }
      } else {
        if (i == j) { // we are on the diagonal so invoke the diagonal reducer
          mixedMultiReduce<NXZ, doubleN, ReduceType, ReducerDiagonal, writeDiagonal, true, T>(
              result, a, b, c, x, y, z, w);
        } else { // we are on the diagonal so invoke the off-diagonal reducer
          mixedMultiReduce<NXZ, doubleN, ReduceType, ReducerOffDiagonal, writeOffDiagonal, true, T>(
              result, a, b, c, x, y, z, w);
        }
      }
    }

    void reDotProduct(double* result, std::vector<ColorSpinorField*>& x, std::vector<ColorSpinorField*>& y){
#ifndef SSTEP
    errorQuda("S-step code not built\n");
#else
    switch(x.size()){
      case 1:
        multiReduce<1, double, QudaSumFloat, Dot, 0, 0, 0, 0, false>(
            result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, y);
        break;
      case 2:
        multiReduce<2, double, QudaSumFloat, Dot, 0, 0, 0, 0, false>(
            result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, y);
        break;
      case 3:
        multiReduce<3, double, QudaSumFloat, Dot, 0, 0, 0, 0, false>(
            result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, y);
        break;
      case 4:
        multiReduce<4, double, QudaSumFloat, Dot, 0, 0, 0, 0, false>(
            result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, y);
        break;
      case 5:
        multiReduce<5, double, QudaSumFloat, Dot, 0, 0, 0, 0, false>(
            result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, y);
        break;
      case 6:
        multiReduce<6, double, QudaSumFloat, Dot, 0, 0, 0, 0, false>(
            result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, y);
        break;
      case 7:
        multiReduce<7, double, QudaSumFloat, Dot, 0, 0, 0, 0, false>(
            result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, y);
        break;
      case 8:
        multiReduce<8, double, QudaSumFloat, Dot, 0, 0, 0, 0, false>(
            result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, y);
        break;
      /*case 9:
        multiReduce<9,double,QudaSumFloat,Dot,0,0,0,0,false>
        (result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, y);
        break;
      case 10:
        multiReduce<10,double,QudaSumFloat,Dot,0,0,0,0,false>
        (result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, y);
        break;
      case 11:
        multiReduce<11,double,QudaSumFloat,Dot,0,0,0,0,false>
        (result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, y);
        break;
      case 12:
        multiReduce<12,double,QudaSumFloat,Dot,0,0,0,0,false>
        (result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, y);
        break;
      case 13:
        multiReduce<13,double,QudaSumFloat,Dot,0,0,0,0,false>
        (result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, y);
        break;
      case 14:
        multiReduce<14,double,QudaSumFloat,Dot,0,0,0,0,false>
        (result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, y);
        break;
      case 15:
        multiReduce<15,double,QudaSumFloat,Dot,0,0,0,0,false>
        (result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, y);
        break;
      case 16:
        multiReduce<16,double,QudaSumFloat,Dot,0,0,0,0,false>
        (result, make_double2(0.0, 0.0), make_double2(0.0, 0.0), x, y, x, y);
        break;*/
      default:
        errorQuda("Unsupported vector size");
        break;
    }
#endif // SSTEP
    // do a single multi-node reduction only once we have computed all local dot products
    const int Nreduce = x.size()*y.size();
    reduceDoubleArray((double*)result, Nreduce);
  }


    // This function does the outer product of dot products... in column major.
    // There's a function below called 'cDotProduct' that flips it to row major.
    template <template <int MXZ, typename ReducerType, typename Float, typename FloatN> class ReducerDiagonal, typename writeDiagonal,
	      template <int MXZ, typename ReducerType, typename Float, typename FloatN> class ReducerOffDiagonal, typename writeOffDiagonal>
    void multiReduce_recurse(Complex* result, std::vector<ColorSpinorField*>& x, std::vector<ColorSpinorField*>& y,
			     std::vector<ColorSpinorField*>&z, std::vector<ColorSpinorField*>&w, int i_idx, int j_idx, bool hermitian, unsigned int tile_size) {

      if (y.size() > tile_size) // if greater than max single-kernel size, split and recurse
      {
        // Do the recurse first.
        Complex* result0 = &result[0];
        Complex* result1 = &result[x.size()*(y.size()/2)];
        std::vector<ColorSpinorField*> y0(y.begin(), y.begin() + y.size()/2);
        std::vector<ColorSpinorField*> y1(y.begin() + y.size()/2, y.end());
        std::vector<ColorSpinorField*> w0(w.begin(), w.begin() + w.size()/2);
        std::vector<ColorSpinorField*> w1(w.begin() + w.size()/2, w.end());
        multiReduce_recurse<ReducerDiagonal,writeDiagonal,ReducerOffDiagonal,writeOffDiagonal>(result0, x, y0, z, w0, i_idx, 2*j_idx+0, hermitian, tile_size);
        multiReduce_recurse<ReducerDiagonal,writeDiagonal,ReducerOffDiagonal,writeOffDiagonal>(result1, x, y1, z, w1, i_idx, 2*j_idx+1, hermitian, tile_size);
      }
      else
      {
        double2* cdot = new double2[x.size()*y.size()];

	// if at bottom of recursion, return if on lower left
	if (x.size() <= tile_size && hermitian) {
	  if (j_idx < i_idx) { return; }
	}

        coeff_array<Complex> a, b, c;

        if (x.size() <= tile_size) {
        switch(x.size()){ // COMMENT HERE FOR COMPILE TIME
        case 1:
          multiReduce<1, double2, QudaSumFloat2, ReducerDiagonal, writeDiagonal, ReducerOffDiagonal, writeOffDiagonal, false>(
              cdot, a, b, c, x, y, z, w, i_idx, j_idx);
          break;
#if MAX_MULTI_BLAS_N >= 2
        case 2:
          multiReduce<2, double2, QudaSumFloat2, ReducerDiagonal, writeDiagonal, ReducerOffDiagonal, writeOffDiagonal, false>(
              cdot, a, b, c, x, y, z, w, i_idx, j_idx);
          break;
#if MAX_MULTI_BLAS_N >= 3
        case 3:
          multiReduce<3, double2, QudaSumFloat2, ReducerDiagonal, writeDiagonal, ReducerOffDiagonal, writeOffDiagonal, false>(
              cdot, a, b, c, x, y, z, w, i_idx, j_idx);
          break;
#if MAX_MULTI_BLAS_N >= 4
        case 4:
          multiReduce<4, double2, QudaSumFloat2, ReducerDiagonal, writeDiagonal, ReducerOffDiagonal, writeOffDiagonal, false>(
              cdot, a, b, c, x, y, z, w, i_idx, j_idx);
          break;
#if MAX_MULTI_BLAS_N >= 5
        case 5:
          multiReduce<5, double2, QudaSumFloat2, ReducerDiagonal, writeDiagonal, ReducerOffDiagonal, writeOffDiagonal, false>(
              cdot, a, b, c, x, y, z, w, i_idx, j_idx);
          break;
#if MAX_MULTI_BLAS_N >= 6
        case 6:
          multiReduce<6, double2, QudaSumFloat2, ReducerDiagonal, writeDiagonal, ReducerOffDiagonal, writeOffDiagonal, false>(
              cdot, a, b, c, x, y, z, w, i_idx, j_idx);
          break;
#if MAX_MULTI_BLAS_N >= 7
        case 7:
          multiReduce<7, double2, QudaSumFloat2, ReducerDiagonal, writeDiagonal, ReducerOffDiagonal, writeOffDiagonal, false>(
              cdot, a, b, c, x, y, z, w, i_idx, j_idx);
          break;
#if MAX_MULTI_BLAS_N >= 8
        case 8:
          multiReduce<8, double2, QudaSumFloat2, ReducerDiagonal, writeDiagonal, ReducerOffDiagonal, writeOffDiagonal, false>(
              cdot, a, b, c, x, y, z, w, i_idx, j_idx);
          break;
#if MAX_MULTI_BLAS_N >= 9
	case 9:
          multiReduce<9, double2, QudaSumFloat2, ReducerDiagonal, writeDiagonal, ReducerOffDiagonal, writeOffDiagonal, false>(
              cdot, a, b, c, x, y, z, w, i_idx, j_idx);
          break;
#if MAX_MULTI_BLAS_N >= 10
        case 10:
          multiReduce<10, double2, QudaSumFloat2, ReducerDiagonal, writeDiagonal, ReducerOffDiagonal, writeOffDiagonal, false>(
              cdot, a, b, c, x, y, z, w, i_idx, j_idx);
          break;
#if MAX_MULTI_BLAS_N >= 11
        case 11:
          multiReduce<11, double2, QudaSumFloat2, ReducerDiagonal, writeDiagonal, ReducerOffDiagonal, writeOffDiagonal, false>(
              cdot, a, b, c, x, y, z, w, i_idx, j_idx);
          break;
#if MAX_MULTI_BLAS_N >= 12
        case 12:
          multiReduce<12, double2, QudaSumFloat2, ReducerDiagonal, writeDiagonal, ReducerOffDiagonal, writeOffDiagonal, false>(
              cdot, a, b, c, x, y, z, w, i_idx, j_idx);
          break;
#if MAX_MULTI_BLAS_N >= 13
        case 13:
          multiReduce<13, double2, QudaSumFloat2, ReducerDiagonal, writeDiagonal, ReducerOffDiagonal, writeOffDiagonal, false>(
              cdot, a, b, c, x, y, z, w, i_idx, j_idx);
          break;
#if MAX_MULTI_BLAS_N >= 14
        case 14:
          multiReduce<14, double2, QudaSumFloat2, ReducerDiagonal, writeDiagonal, ReducerOffDiagonal, writeOffDiagonal, false>(
              cdot, a, b, c, x, y, z, w, i_idx, j_idx);
          break;
#if MAX_MULTI_BLAS_N >= 15
        case 15:
          multiReduce<15, double2, QudaSumFloat2, ReducerDiagonal, writeDiagonal, ReducerOffDiagonal, writeOffDiagonal, false>(
              cdot, a, b, c, x, y, z, w, i_idx, j_idx);
          break;
#if MAX_MULTI_BLAS_N >= 16
        case 16:
          multiReduce<16, double2, QudaSumFloat2, ReducerDiagonal, writeDiagonal, ReducerOffDiagonal, writeOffDiagonal, false>(
              cdot, a, b, c, x, y, z, w, i_idx, j_idx);
          break;
#endif //16
#endif //15
#endif //14
#endif //13
#endif //12
#endif //11
#endif //10
#endif // 9
#endif // 8
#endif // 7
#endif // 6
#endif // 5
#endif // 4
#endif // 3
#endif // 2
	}
	} else {
          // split the problem and recurse. Splitting in x requires
          // memory reshuffling (unless y = 1).
          // Use a few temporary variables.

          Complex* tmpmajor = new Complex[x.size()*y.size()];
          Complex* result0 = &tmpmajor[0];
          Complex* result1 = &tmpmajor[(x.size()/2)*y.size()];
          std::vector<ColorSpinorField*> x0(x.begin(), x.begin() + x.size()/2);
          std::vector<ColorSpinorField*> x1(x.begin() + x.size()/2, x.end());
          std::vector<ColorSpinorField*> z0(z.begin(), z.begin() + z.size()/2);
          std::vector<ColorSpinorField*> z1(z.begin() + z.size()/2, z.end());

          multiReduce_recurse<ReducerDiagonal,writeDiagonal,ReducerOffDiagonal,writeOffDiagonal>(result0, x0, y, z0, w, 2*i_idx+0, j_idx, hermitian, tile_size);
          multiReduce_recurse<ReducerDiagonal,writeDiagonal,ReducerOffDiagonal,writeOffDiagonal>(result1, x1, y, z1, w, 2*i_idx+1, j_idx, hermitian, tile_size);

          const unsigned int xlen0 = x.size()/2;
          const unsigned int xlen1 = x.size() - xlen0;
          const unsigned int ylen = y.size();

          // Copy back into result.
          int count = 0, count0 = 0, count1 = 0;
          for (unsigned int i = 0; i < ylen; i++)
          {
            for (unsigned int j = 0; j < xlen0; j++)
              result[count++] = result0[count0++];
            for (unsigned int j = 0; j < xlen1; j++)
              result[count++] = result1[count1++];
          }

          delete[] tmpmajor;
        }

	// we are at the leaf of the binary tree (e.g., we ran the kernel): perform the row-to-column-major transpose here.
        if (x.size() <= tile_size)
        {
          const unsigned int xlen = x.size();
          const unsigned int ylen = y.size();
          for (unsigned int j = 0; j < xlen; j++)
            for (unsigned int i = 0; i < ylen; i++)
              result[i*xlen+j] = Complex(cdot[j*ylen + i].x, cdot[j*ylen+i].y);
        }
        delete[] cdot;
      }
    }


    template <template <int MXZ, typename ReducerType, typename Float, typename FloatN> class ReducerDiagonal,
	      typename writeDiagonal,
	      template <int MXZ, typename ReducerType, typename Float, typename FloatN> class ReducerOffDiagonal,
	      typename writeOffDiagonal>
    class TileSizeTune : public Tunable {
      typedef std::vector<ColorSpinorField*> vec;
      Complex *result;
      vec &x, &y, &z, &w;
      bool hermitian;
      bool Anorm;

      unsigned int sharedBytesPerThread() const { return 0; }
      unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }

      unsigned int max_tile_size;

    public:
      TileSizeTune(Complex *result, vec &x, vec &y, vec &z, vec &w, bool hermitian, bool Anorm = false)
	: result(result), x(x), y(y), z(z), w(w), hermitian(hermitian), Anorm(Anorm), max_tile_size(1)
      {
      	strcpy(aux, "policy,");
      	strcat(aux, x[0]->AuxString());
      	strcat(aux, ",");
      	strcat(aux, y[0]->AuxString());
        if (hermitian) strcat(aux, ",hermitian");
        if (Anorm) strcat(aux, ",Anorm");
	strcat(aux,",n=");
	char size[8];
	u64toa(size, x.size());
	strcat(aux,size);
	strcat(aux,",m=");
	u64toa(size, y.size());
	strcat(aux,size);
        u64toa(size, MAX_MULTI_BLAS_N);
        strcat(aux, ",multi-blas-n=");
        strcat(aux, size);

        // before we do policy tuning we must ensure the kernel
        // constituents have been tuned since we can't do nested tuning
        // FIXME this will break if the kernels are destructive - which they aren't here
        if (getTuning() && getTuneCache().find(tuneKey()) == getTuneCache().end()) {
          disableProfileCount(); // purely for profiling reasons, don't want to profile tunings.

          if (x.size() == 1 || y.size() == 1) { // 1-d reduction

            max_tile_size = std::min(MAX_MULTI_BLAS_N, (int)std::max(x.size(), y.size()));

            // Make sure constituents are tuned.
	    for ( unsigned int tile_size=1; tile_size <= max_tile_size; tile_size++) {
	      multiReduce_recurse<ReducerDiagonal,writeDiagonal,ReducerOffDiagonal,writeOffDiagonal>
		(result, x, y, z, w, 0, 0, hermitian, tile_size);
	    }

          } else { // 2-d reduction

            // max_tile_size should be set to the largest power of 2 less than
            // MAX_MULTI_BLAS_N, since we have a requirement that the
            // tile size is a power of 2.
            unsigned int max_count = 0;
	    unsigned int tile_size_tmp = MAX_MULTI_BLAS_N;
	    while (tile_size_tmp != 1) { tile_size_tmp = tile_size_tmp >> 1; max_count++; }
	    tile_size_tmp = 1;
	    for (unsigned int i = 0; i < max_count; i++) { tile_size_tmp = tile_size_tmp << 1; }
	    max_tile_size = tile_size_tmp;

	    // Make sure constituents are tuned.
	    for ( unsigned int tile_size=1; tile_size <= max_tile_size && tile_size <= x.size() &&
		    (tile_size <= y.size() || y.size()==1) ; tile_size*=2) {
	      multiReduce_recurse<ReducerDiagonal,writeDiagonal,ReducerOffDiagonal,writeOffDiagonal>
		(result, x, y, z, w, 0, 0, hermitian, tile_size);
	    }

            // also test case using a single kernel if both dimensions
            // are less than MAX_MULTI_BLAS_N
            if (x.size() <= MAX_MULTI_BLAS_N && y.size() <= MAX_MULTI_BLAS_N) {
	      multiReduce_recurse<ReducerDiagonal,writeDiagonal,ReducerOffDiagonal,writeOffDiagonal>
		(result, x, y, z, w, 0, 0, hermitian, MAX_MULTI_BLAS_N);
            }
          }

          enableProfileCount();
          setPolicyTuning(true);
        }
      }

      virtual ~TileSizeTune() { setPolicyTuning(false); }

      void apply(const hipStream_t &stream) {
        TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());

        // tp.aux.x is where the tile size is stored. "tp" is the tuning struct.
        // it contains blocksize, grid size, etc. Since we're only tuning
        // a policy, we don't care about those sizes. That's why we only
        // tune "aux.x", which is the tile size.
        multiReduce_recurse<ReducerDiagonal,writeDiagonal,ReducerOffDiagonal,writeOffDiagonal>
          (result, x, y, z, w, 0, 0, hermitian, tp.aux.x);
      }

      // aux.x is the tile size
      bool advanceAux(TuneParam &param) const
      {
	if ( x.size()==1 || y.size()==1 ) { // 1-d reduction

	  param.aux.x++;
	  if ( (unsigned int)param.aux.x <= max_tile_size ) {
	    return true;
	  } else {
	    param.aux.x = 1;
	    return false;
	  }

	} else { // 2-d reduction

	  if ( (unsigned int)(2*param.aux.x) <= max_tile_size &&
               (unsigned int)(2*param.aux.x) <= x.size() &&
	       (unsigned int)(2*param.aux.x) <= y.size() ) {
            param.aux.x *= 2; // only tune powers of two
	    return true;
	  } else if (x.size() <= MAX_MULTI_BLAS_N && y.size() <= MAX_MULTI_BLAS_N && param.aux.x < MAX_MULTI_BLAS_N) {
            // we've run out of power of two tiles to try, but before
            // we finish, try a single kernel if it fits
            param.aux.x = MAX_MULTI_BLAS_N;
            return true;
          } else {
	    param.aux.x = 1; // reset to the beginning (which we'd need for multi-dimensional tuning)
	    return false;
	  }

	}
      }

      bool advanceTuneParam(TuneParam &param) const { return advanceAux(param); }

      void initTuneParam(TuneParam &param) const  {
      	Tunable::initTuneParam(param);
      	param.aux.x = 1; param.aux.y = 0; param.aux.z = 0; param.aux.w = 0;
      }

      void defaultTuneParam(TuneParam &param) const  {
      	Tunable::defaultTuneParam(param); // default is max tile size
        // max_tile_size is MAX_MULTI_BLAS_N rounded down to the nearest power of 2.
      	param.aux.x = max_tile_size; param.aux.y = 0; param.aux.z = 0; param.aux.w = 0;
      }

      TuneKey tuneKey() const {
        return TuneKey(x[0]->VolString(), typeid(*this).name(), aux);
      }

      long long flops() const { return 0; } // FIXME
      long long bytes() const { return 0; } // FIXME

      void preTune() { } // FIXME - use write to determine what needs to be saved
      void postTune() { } // FIXME - use write to determine what needs to be saved
    };

    void cDotProduct(Complex* result, std::vector<ColorSpinorField*>& x, std::vector<ColorSpinorField*>& y){
      if (x.size() == 0 || y.size() == 0) errorQuda("vector.size() == 0");
      Complex* result_tmp = new Complex[x.size()*y.size()];
      for (unsigned int i = 0; i < x.size()*y.size(); i++) result_tmp[i] = 0.0;

      // cDotProduct_recurse returns a column-major matrix.
      // To be consistent with the multi-blas functions, we should
      // switch this to row-major.
      TileSizeTune<Cdot,write<0,0,0,0>,Cdot,write<0,0,0,0> > tile(result_tmp, x, y, x, y, false);
      tile.apply(0);

      // do a single multi-node reduction only once we have computed all local dot products
      const int Nreduce = 2*x.size()*y.size();
      reduceDoubleArray((double*)result_tmp, Nreduce);

      // Switch from col-major to row-major
      const unsigned int xlen = x.size();
      const unsigned int ylen = y.size();
      for (unsigned int j = 0; j < xlen; j++)
        for (unsigned int i = 0; i < ylen; i++)
          result[j*ylen+i] = result_tmp[i*xlen + j];

      delete[] result_tmp;
    }

    void hDotProduct(Complex* result, std::vector<ColorSpinorField*>& x, std::vector<ColorSpinorField*>& y){
      if (x.size() == 0 || y.size() == 0) errorQuda("vector.size() == 0");
      if (x.size() != y.size()) errorQuda("Cannot call Hermitian block dot product on non-square inputs");

      Complex* result_tmp = new Complex[x.size()*y.size()];
      for (unsigned int i = 0; i < x.size()*y.size(); i++) result_tmp[i] = 0.0;

      TileSizeTune<Cdot,write<0,0,0,0>,Cdot,write<0,0,0,0> > tile(result_tmp, x, y, x, y, true, false); // last false is b/c L2 norm
      tile.apply(0);

      // do a single multi-node reduction only once we have computed all local dot products
      const int Nreduce = 2*x.size()*y.size();
      reduceDoubleArray((double*)result_tmp, Nreduce); // FIXME - could optimize this for Hermiticity as well

      // Switch from col-major to row-major
      const unsigned int xlen = x.size();
      const unsigned int ylen = y.size();
      for (unsigned int j = 0; j < xlen; j++)
        for (unsigned int i = j; i < ylen; i++) {
          result[j*ylen+i] = result_tmp[i*xlen + j];
          result[i*ylen+j] = conj(result_tmp[i*xlen + j]);
	}

      delete[] result_tmp;
    }

    // for (p, Ap) norms in CG which are Hermitian.
    void hDotProduct_Anorm(Complex* result, std::vector<ColorSpinorField*>& x, std::vector<ColorSpinorField*>& y){
      if (x.size() == 0 || y.size() == 0) errorQuda("vector.size() == 0");
      if (x.size() != y.size()) errorQuda("Cannot call Hermitian block A-norm dot product on non-square inputs");

      Complex* result_tmp = new Complex[x.size()*y.size()];
      for (unsigned int i = 0; i < x.size()*y.size(); i++) result_tmp[i] = 0.0;

      TileSizeTune<Cdot,write<0,0,0,0>,Cdot,write<0,0,0,0> > tile(result_tmp, x, y, x, y, true, true); // last true is b/c A norm
      tile.apply(0);

      // do a single multi-node reduction only once we have computed all local dot products
      const int Nreduce = 2*x.size()*y.size();
      reduceDoubleArray((double*)result_tmp, Nreduce); // FIXME - could optimize this for Hermiticity as well

      // Switch from col-major to row-major
      const unsigned int xlen = x.size();
      const unsigned int ylen = y.size();
      for (unsigned int j = 0; j < xlen; j++)
        for (unsigned int i = j; i < ylen; i++) {
          result[j*ylen+i] = result_tmp[i*xlen + j];
          result[i*ylen+j] = conj(result_tmp[i*xlen + j]);
  }

      delete[] result_tmp;
    }

    // takes the outer product of inner products between and y and copies y into z
    void cDotProductCopy(Complex* result, std::vector<ColorSpinorField*>& x, std::vector<ColorSpinorField*>& y,
			 std::vector<ColorSpinorField*>&z){

#if 0
      if (x.size() == 0 || y.size() == 0) errorQuda("vector.size() == 0");
      if (y.size() != z.size()) errorQuda("Cannot copy input y of size %lu into z of size %lu\n", y.size(), z.size());

      Complex* result_tmp = new Complex[x.size()*y.size()];
      for (unsigned int i = 0; i < x.size()*y.size(); i++) result_tmp[i] = 0.0;

      // When recursing, only the diagonal tiles will do the copy, the rest just do the outer product
      TileSizeTune<CdotCopy,write<0,0,0,1>,Cdot,write<0,0,0,0> > tile(result_tmp, x, y, x, y, true);
      tile.apply(0);

      // do a single multi-node reduction only once we have computed all local dot products
      const int Nreduce = 2*x.size()*y.size();
      reduceDoubleArray((double*)result_tmp, Nreduce);

      // Switch from col-major to row-major.
      const unsigned int xlen = x.size();
      const unsigned int ylen = y.size();
      for (unsigned int j = 0; j < xlen; j++)
        for (unsigned int i = 0; i < ylen; i++)
          result[j*ylen+i] = result_tmp[i*xlen + j];

      delete[] result_tmp;
#else
      errorQuda("cDotProductCopy not enabled");
#endif
    }

   } // namespace blas

} // namespace quda
