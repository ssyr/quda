#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <cstring> // needed for memset

#include <tune_quda.h>
#include <blas_quda.h>
#include <color_spinor_field.h>

#include <jitify_helper.cuh>
#include <kernels/multi_blas_core.cuh>

namespace quda {

  namespace blas {

    qudaStream_t* getStream();

    template <template <typename ...> class Functor, typename store_t, typename y_store_t, int nSpin, typename T>
    class MultiBlas : public TunableVectorY
    {
      using real = typename mapper<y_store_t>::type;
      const int NXZ;
      const int NYW;
      Functor<real> f;
      int max_warp_split;
      mutable int warp_split; // helper used to keep track of current warp splitting
      const int nParity;
      const T &a, &b, &c;
      std::vector<ColorSpinorField *> &x, &y, &z, &w;
      const QudaFieldLocation location;

      bool tuneSharedBytes() const { return false; }

      // for these streaming kernels, there is no need to tune the grid size, just use max
      unsigned int minGridSize() const { return maxGridSize(); }

    public:
      MultiBlas(const T &a, const T &b, const T &c, const ColorSpinorField &x_meta, const ColorSpinorField &y_meta,
                std::vector<ColorSpinorField *> &x, std::vector<ColorSpinorField *> &y,
                std::vector<ColorSpinorField *> &z, std::vector<ColorSpinorField *> &w) :
        TunableVectorY(y.size()),
        NXZ(x.size()),
        NYW(y.size()),
        f(NXZ, NYW),
        warp_split(1),
        nParity(x[0]->SiteSubset()),
        a(a),
        b(b),
        c(c),
        x(x),
        y(y),
        z(z),
        w(w),
        location(checkLocation(*x[0], *y[0], *z[0], *w[0]))
      {
        checkLength(*x[0], *y[0], *z[0], *w[0]);
        auto x_prec = checkPrecision(*x[0], *z[0], *w[0]);
        auto y_prec = y[0]->Precision();
        auto x_order = checkOrder(*x[0], *z[0], *w[0]);
        auto y_order = y[0]->FieldOrder();
        if (sizeof(store_t) != x_prec) errorQuda("Expected precision %lu but received %d", sizeof(store_t), x_prec);
        if (sizeof(y_store_t) != y_prec) errorQuda("Expected precision %lu but received %d", sizeof(y_store_t), y_prec);
        if (x_prec == y_prec && x_order != y_order) errorQuda("Orders %d %d do not match", x_order, y_order);

        // heuristic for enabling if we need the warp-splitting optimization
        const int gpu_size = 2 * deviceProp.maxThreadsPerBlock * deviceProp.multiProcessorCount;
        switch (gpu_size / (x[0]->Length() * NYW)) {
        case 0: max_warp_split = 1; break; // we have plenty of work, no need to split
        case 1: max_warp_split = 2; break; // double the thread count
        case 2:                            // quadruple the thread count
        default: max_warp_split = 4;
        }
        max_warp_split = std::min(NXZ, max_warp_split); // ensure we only split if valid

        Amatrix_h = reinterpret_cast<signed char *>(const_cast<typename T::type *>(a.data));
        Bmatrix_h = reinterpret_cast<signed char *>(const_cast<typename T::type *>(b.data));
        Cmatrix_h = reinterpret_cast<signed char *>(const_cast<typename T::type *>(c.data));

        strcpy(aux, x[0]->AuxString());
        if (x_prec != y_prec) {
          strcat(aux, ",");
          strcat(aux, y[0]->AuxString());
        }

#ifdef JITIFY
        ::quda::create_jitify_program("kernels/multi_blas_core.cuh");
#endif

        apply(*getStream());

        blas::bytes += bytes();
        blas::flops += flops();
      }

      TuneKey tuneKey() const
      {
        char name[TuneKey::name_n];
        char NXZ_str[8];
        char NYW_str[8];
        u32toa(NXZ_str, NXZ);
        u32toa(NYW_str, NYW);
        strcpy(name, "Nxz");
        strcat(name, NXZ_str);
        strcat(name, "Nyw");
        strcat(name, NYW_str);
        strcat(name, typeid(f).name());
        return TuneKey(x[0]->VolString(), name, aux);
      }

      template <bool multi_1d, typename device_buffer_t, typename Arg> typename std::enable_if<multi_1d, void>::type
      set_param(device_buffer_t &&buf_d, Arg &arg, char select, const T &h, const qudaStream_t &stream)
      {
        using coeff_t = typename decltype(arg.f)::coeff_t;
        coeff_t *buf_arg = nullptr;
        switch (select) {
        case 'a': buf_arg = arg.f.a; break;
        case 'b': buf_arg = arg.f.b; break;
        case 'c': buf_arg = arg.f.c; break;
        default: errorQuda("Unknown buffer %c", select);
        }
        const auto N = std::max(NXZ,NYW);
        for (int i = 0; i < N; i++) buf_arg[i] = coeff_t(h.data[i]);
      }

      template <bool multi_1d, typename device_buffer_t, typename Arg> typename std::enable_if<!multi_1d, void>::type
      set_param(device_buffer_t &&buf_d, Arg &arg, char dummy, const T &h, const qudaStream_t &stream)
      {
        using coeff_t = typename decltype(arg.f)::coeff_t;
        constexpr size_t n_coeff = MAX_MATRIX_SIZE / sizeof(coeff_t);

        coeff_t tmp[n_coeff];
        for (int i = 0; i < NXZ; i++)
          for (int j = 0; j < NYW; j++) tmp[NYW * i + j] = coeff_t(h.data[NYW * i + j]);

#ifdef JITIFY
        hipMemcpyHtoDAsync(buf_d, tmp, NXZ * NYW * sizeof(coeff_t), stream);
#else
        hipMemcpyToSymbolAsync(HIP_SYMBOL(buf_d), tmp, NXZ * NYW * sizeof(coeff_t), 0, hipMemcpyHostToDevice, stream);
#endif
      }

      template <int NXZ> void compute(const qudaStream_t &stream)
      {
        staticCheck<NXZ, store_t, y_store_t, decltype(f)>(f, x, y);

        constexpr bool site_unroll_check = !std::is_same<store_t, y_store_t>::value || isFixed<store_t>::value;
        if (site_unroll_check && (x[0]->Ncolor() != 3 || x[0]->Nspin() == 2))
          errorQuda("site unroll not supported for nSpin = %d nColor = %d", x[0]->Nspin(), x[0]->Ncolor());

        TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());

        if (location == QUDA_CUDA_FIELD_LOCATION) {
          if (site_unroll_check) checkNative(*x[0], *y[0], *z[0], *w[0]); // require native order when using site_unroll
          using device_store_t = typename device_type_mapper<store_t>::type;
          using device_y_store_t = typename device_type_mapper<y_store_t>::type;
          using device_real_t = typename mapper<device_y_store_t>::type;
          Functor<device_real_t> f_(NXZ, NYW);

          // redefine site_unroll with device_store types to ensure we have correct N/Ny/M values
          constexpr bool site_unroll = !std::is_same<device_store_t, device_y_store_t>::value || isFixed<device_store_t>::value;
          constexpr int N = n_vector<device_store_t, true, nSpin, site_unroll>();
          constexpr int Ny = n_vector<device_y_store_t, true, nSpin, site_unroll>();
          constexpr int M = site_unroll ? (nSpin == 4 ? 24 : 6) : N; // real numbers per thread
          const int length = x[0]->Length() / (nParity * M);

          tp.block.x *= tp.aux.x; // include warp-split factor

          MultiBlasArg<NXZ, device_store_t, N, device_y_store_t, Ny, decltype(f_)> arg(x, y, z, w, f_, NYW, length);
#ifdef JITIFY
          using namespace jitify::reflection;
          auto instance = program->kernel("quda::blas::multiBlasKernel")
            .instantiate(Type<device_real_t>(), M, NXZ, tp.aux.x, Type<decltype(arg)>());

          if (a.data) set_param<decltype(f_)::multi_1d>(instance.get_constant_ptr("quda::blas::Amatrix_d"), arg, 'a', a, stream);
          if (b.data) set_param<decltype(f_)::multi_1d>(instance.get_constant_ptr("quda::blas::Bmatrix_d"), arg, 'b', b, stream);
          if (c.data) set_param<decltype(f_)::multi_1d>(instance.get_constant_ptr("quda::blas::Cmatrix_d"), arg, 'c', c, stream);

          jitify_error = instance.configure(tp.grid, tp.block, tp.shared_bytes, stream).launch(arg);
#else
          if (a.data) { set_param<decltype(f_)::multi_1d>(Amatrix_d, arg, 'a', a, stream); }
          if (b.data) { set_param<decltype(f_)::multi_1d>(Bmatrix_d, arg, 'b', b, stream); }
          if (c.data) { set_param<decltype(f_)::multi_1d>(Cmatrix_d, arg, 'c', c, stream); }
          switch (tp.aux.x) {
          case 1: qudaLaunchKernel(multiBlasKernel<device_real_t, M, NXZ, 1, decltype(arg)>, tp, stream, arg); break;
#ifdef WARP_SPLIT
          case 2: qudaLaunchKernel(multiBlasKernel<device_real_t, M, NXZ, 2, decltype(arg)>, tp, stream, arg); break;
          case 4: qudaLaunchKernel(multiBlasKernel<device_real_t, M, NXZ, 4, decltype(arg)>, tp, stream, arg); break;
#endif
          default: errorQuda("warp-split factor %d not instantiated", tp.aux.x);
          }
#endif

          tp.block.x /= tp.aux.x; // restore block size
        } else {
          errorQuda("Only implemented for GPU fields");
        }
      }

      template <int n> typename std::enable_if<n!=1, void>::type instantiateLinear(const qudaStream_t &stream)
      {
        if (NXZ == n) compute<n>(stream);
        else instantiateLinear<n-1>(stream);
      }

      template <int n> typename std::enable_if<n==1, void>::type instantiateLinear(const qudaStream_t &stream)
      {
        compute<1>(stream);
      }

      template <int n> typename std::enable_if<n!=1, void>::type instantiatePow2(const qudaStream_t &stream)
      {
        if (NXZ == n) compute<n>(stream);
        else instantiatePow2<n/2>(stream);
      }

      template <int n> typename std::enable_if<n==1, void>::type instantiatePow2(const qudaStream_t &stream)
      {
        compute<1>(stream);
      }

      // instantiate the loop unrolling template
      template <int NXZ_max> typename std::enable_if<NXZ_max!=1, void>::type instantiate(const qudaStream_t &stream)
      {
        // if multi-1d then constrain the templates to no larger than max-1d size
        constexpr int pow2_max = !decltype(f)::multi_1d ? max_NXZ_power2<false, isFixed<store_t>::value>() :
          std::min(max_N_multi_1d(), max_NXZ_power2<false, isFixed<store_t>::value>());
        constexpr int linear_max = !decltype(f)::multi_1d ? MAX_MULTI_BLAS_N : std::min(max_N_multi_1d(), MAX_MULTI_BLAS_N);

        if (NXZ <= pow2_max && is_power2(NXZ)) instantiatePow2<pow2_max>(stream);
        else if (NXZ <= linear_max) instantiateLinear<linear_max>(stream);
        else errorQuda("x.size %lu greater than maximum supported size (pow2 = %d, linear = %d)", x.size(), pow2_max, linear_max);
      }

      template <int NXZ_max> typename std::enable_if<NXZ_max==1, void>::type instantiate(const qudaStream_t &stream)
      {
        compute<1>(stream);
      }

      void apply(const qudaStream_t &stream) { instantiate<decltype(f)::NXZ_max>(stream); }

      void preTune()
      {
        for (int i = 0; i < NYW; ++i) {
          if (f.write.X) x[i]->backup();
          if (f.write.Y) y[i]->backup();
          if (f.write.Z) z[i]->backup();
          if (f.write.W) w[i]->backup();
        }
      }

      void postTune()
      {
        for (int i = 0; i < NYW; ++i) {
          if (f.write.X) x[i]->restore();
          if (f.write.Y) y[i]->restore();
          if (f.write.Z) z[i]->restore();
          if (f.write.W) w[i]->restore();
        }
      }

      bool advanceAux(TuneParam &param) const
      {
#ifdef WARP_SPLIT
        if (2 * param.aux.x <= max_warp_split) {
          param.aux.x *= 2;
          warp_split = param.aux.x;
          return true;
        } else {
          param.aux.x = 1;
          warp_split = param.aux.x;
          // reset the block dimension manually here to pick up the warp_split parameter
          resetBlockDim(param);
          return false;
        }
#else
        warp_split = 1;
        return false;
#endif
      }

      int blockStep() const { return deviceProp.warpSize / warp_split; }
      int blockMin() const { return deviceProp.warpSize / warp_split; }

      void initTuneParam(TuneParam &param) const
      {
        TunableVectorY::initTuneParam(param);
        param.grid.z = nParity;
        param.aux = make_int4(1, 0, 0, 0); // warp-split parameter
      }

      void defaultTuneParam(TuneParam &param) const
      {
        TunableVectorY::defaultTuneParam(param);
        param.grid.z = nParity;
        param.aux = make_int4(1, 0, 0, 0); // warp-split parameter
      }

      long long flops() const
      {
        return NYW * NXZ * f.flops() * x[0]->Length();
      }

      long long bytes() const
      {
        // X and Z reads are repeated (and hopefully cached) across NYW
        // each Y and W read/write is done once
        return NYW * NXZ * (f.read.X + f.write.X) * x[0]->Bytes() +
          NYW * (f.read.Y + f.write.Y) * y[0]->Bytes() +
          NYW * NXZ * (f.read.Z + f.write.Z) * z[0]->Bytes() +
          NYW * (f.read.W + f.write.W) * w[0]->Bytes();
      }

      int tuningIter() const { return 3; }
    };

    using range = std::pair<size_t,size_t>;

    template <template <typename...> class Functor, typename T>
    void axpy_recurse(const T *a_, std::vector<ColorSpinorField *> &x, std::vector<ColorSpinorField *> &y,
                      const range &range_x, const range &range_y, int upper, int coeff_width)
    {
      // if greater than max single-kernel size, recurse
      if (y.size() > (size_t)max_YW_size(x.size(), x[0]->Precision(), y[0]->Precision(), false, false, coeff_width, false)) {
        // We need to split up 'a' carefully since it's row-major.
        T *tmpmajor = new T[x.size() * y.size()];
        T *tmpmajor0 = &tmpmajor[0];
        T *tmpmajor1 = &tmpmajor[x.size() * (y.size() / 2)];
        std::vector<ColorSpinorField*> y0(y.begin(), y.begin() + y.size()/2);
        std::vector<ColorSpinorField*> y1(y.begin() + y.size()/2, y.end());

        const unsigned int xlen = x.size();
        const unsigned int ylen0 = y.size()/2;
        const unsigned int ylen1 = y.size() - y.size()/2;

        int count = 0, count0 = 0, count1 = 0;
        for (unsigned int i = 0; i < xlen; i++)
        {
          for (unsigned int j = 0; j < ylen0; j++)
            tmpmajor0[count0++] = a_[count++];
          for (unsigned int j = 0; j < ylen1; j++)
            tmpmajor1[count1++] = a_[count++];
        }

        axpy_recurse<Functor>(tmpmajor0, x, y0, range_x, range(range_y.first, range_y.first + y0.size()), upper, coeff_width);
        axpy_recurse<Functor>(tmpmajor1, x, y1, range_x, range(range_y.first + y0.size(), range_y.second), upper, coeff_width);

        delete[] tmpmajor;
      } else {
        // if at the bottom of recursion,
        if (is_valid_NXZ(x.size(), false, x[0]->Precision() < QUDA_SINGLE_PRECISION)) {
          // since tile range is [first,second), e.g., [first,second-1], we need >= here
          // if upper triangular and upper-right tile corner is below diagonal return
          if (upper == 1 && range_y.first >= range_x.second) { return; }
          // if lower triangular and lower-left tile corner is above diagonal return
          if (upper == -1 && range_x.first >= range_y.second) { return; }

          // mark true since we will copy the "a" matrix into constant memory
          coeff_array<T> a(a_), b, c;
          constexpr bool mixed = true;
          instantiate<Functor, MultiBlas, mixed>(a, b, c, *x[0], *y[0], x, y, x, x);
        } else {
          // split the problem in half and recurse
          const T *a0 = &a_[0];
          const T *a1 = &a_[(x.size() / 2) * y.size()];

          std::vector<ColorSpinorField *> x0(x.begin(), x.begin() + x.size() / 2);
          std::vector<ColorSpinorField *> x1(x.begin() + x.size() / 2, x.end());

          axpy_recurse<Functor>(a0, x0, y, range(range_x.first, range_x.first + x0.size()), range_y, upper, coeff_width);
          axpy_recurse<Functor>(a1, x1, y, range(range_x.first + x0.size(), range_x.second), range_y, upper, coeff_width);
        }
      } // end if (y.size() > max_YW_size())
    }

    void caxpy(const Complex *a_, std::vector<ColorSpinorField*> &x, std::vector<ColorSpinorField*> &y) {
      // Enter a recursion.
      // Pass a, x, y. (0,0) indexes the tiles. false specifies the matrix is unstructured.
      axpy_recurse<multicaxpy_>(a_, x, y, range(0,x.size()), range(0,y.size()), 0, 2);
    }

    void caxpy_U(const Complex *a_, std::vector<ColorSpinorField*> &x, std::vector<ColorSpinorField*> &y) {
      // Enter a recursion.
      // Pass a, x, y. (0,0) indexes the tiles. 1 indicates the matrix is upper-triangular,
      //                                         which lets us skip some tiles.
      if (x.size() != y.size())
      {
        errorQuda("An optimal block caxpy_U with non-square 'a' has not yet been implemented. Use block caxpy instead");
      }
      axpy_recurse<multicaxpy_>(a_, x, y, range(0,x.size()), range(0,y.size()), 1, 2);
    }

    void caxpy_L(const Complex *a_, std::vector<ColorSpinorField*> &x, std::vector<ColorSpinorField*> &y) {
      // Enter a recursion.
      // Pass a, x, y. (0,0) indexes the tiles. -1 indicates the matrix is lower-triangular
      //                                         which lets us skip some tiles.
      if (x.size() != y.size())
      {
        errorQuda("An optimal block caxpy_L with non-square 'a' has not yet been implemented. Use block caxpy instead");
      }
      axpy_recurse<multicaxpy_>(a_, x, y, range(0,x.size()), range(0,y.size()), -1, 2);
    }

    void caxpy(const Complex *a, ColorSpinorField &x, ColorSpinorField &y) { caxpy(a, x.Components(), y.Components()); }

    void caxpy_U(const Complex *a, ColorSpinorField &x, ColorSpinorField &y) { caxpy_U(a, x.Components(), y.Components()); }

    void caxpy_L(const Complex *a, ColorSpinorField &x, ColorSpinorField &y) { caxpy_L(a, x.Components(), y.Components()); }

    void caxpyz_recurse(const Complex *a_, std::vector<ColorSpinorField*> &x, std::vector<ColorSpinorField*> &y,
                        std::vector<ColorSpinorField*> &z, const range &range_x, const range &range_y,
                        int pass, int upper)
    {
      // if greater than max single-kernel size, recurse
      if (y.size() > (size_t)max_YW_size(x.size(), x[0]->Precision(), y[0]->Precision(), false, true, 2, false)) {
        // We need to split up 'a' carefully since it's row-major.
        Complex* tmpmajor = new Complex[x.size()*y.size()];
        Complex* tmpmajor0 = &tmpmajor[0];
        Complex* tmpmajor1 = &tmpmajor[x.size()*(y.size()/2)];
        std::vector<ColorSpinorField*> y0(y.begin(), y.begin() + y.size()/2);
        std::vector<ColorSpinorField*> y1(y.begin() + y.size()/2, y.end());

        std::vector<ColorSpinorField*> z0(z.begin(), z.begin() + z.size()/2);
        std::vector<ColorSpinorField*> z1(z.begin() + z.size()/2, z.end());

        const unsigned int xlen = x.size();
        const unsigned int ylen0 = y.size()/2;
        const unsigned int ylen1 = y.size() - y.size()/2;

        int count = 0, count0 = 0, count1 = 0;
        for (unsigned int i_ = 0; i_ < xlen; i_++)
        {
          for (unsigned int j = 0; j < ylen0; j++)
            tmpmajor0[count0++] = a_[count++];
          for (unsigned int j = 0; j < ylen1; j++)
            tmpmajor1[count1++] = a_[count++];
        }

        caxpyz_recurse(tmpmajor0, x, y0, z0, range_x, range(range_y.first, range_y.first + y0.size()), pass, upper);
        caxpyz_recurse(tmpmajor1, x, y1, z1, range_x, range(range_y.first + y0.size(), range_y.second), pass, upper);

        delete[] tmpmajor;
      } else {
        // if at bottom of recursion check where we are
        if (is_valid_NXZ(x.size(), false, x[0]->Precision() < QUDA_SINGLE_PRECISION)) {
          // check if tile straddles diagonal
          bool is_diagonal = (range_x.first < range_y.second) && (range_y.first < range_x.second);
          if (pass==1) {
            if (!is_diagonal) {
              // if upper triangular and upper-right tile corner is below diagonal return
              if (upper == 1 && range_y.first >= range_x.second) { return; }
              // if lower triangular and lower-left tile corner is above diagonal return
              if (upper == -1 && range_x.first >= range_y.second) { return; }
              caxpy(a_, x, z); return;  // off diagonal
            }
            return;
      	  } else {
            if (!is_diagonal) return; // We're on the first pass, so we only want to update the diagonal.
          }

          coeff_array<Complex> a(a_), b, c;
          constexpr bool mixed = false;
          instantiate<multicaxpyz_, MultiBlas, mixed>(a, b, c, *x[0], *y[0], x, y, x, z);
        } else {
          // split the problem in half and recurse
          const Complex *a0 = &a_[0];
          const Complex *a1 = &a_[(x.size() / 2) * y.size()];

          std::vector<ColorSpinorField *> x0(x.begin(), x.begin() + x.size() / 2);
          std::vector<ColorSpinorField *> x1(x.begin() + x.size() / 2, x.end());

          caxpyz_recurse(a0, x0, y, z, range(range_x.first, range_x.first + x0.size()), range_y, pass, upper);
          caxpyz_recurse(a1, x1, y, z, range(range_x.first + x0.size(), range_x.second), range_y, pass, upper);
        }
      } // end if (y.size() > max_YW_size())
    }

    void caxpyz(const Complex *a, std::vector<ColorSpinorField*> &x, std::vector<ColorSpinorField*> &y, std::vector<ColorSpinorField*> &z)
    {
      // first pass does the caxpyz on the diagonal
      caxpyz_recurse(a, x, y, z, range(0, x.size()), range(0, y.size()), 0, 0);
      // second pass does caxpy on the off diagonals
      caxpyz_recurse(a, x, y, z, range(0, x.size()), range(0, y.size()), 1, 0);
    }

    void caxpyz_U(const Complex *a, std::vector<ColorSpinorField*> &x, std::vector<ColorSpinorField*> &y, std::vector<ColorSpinorField*> &z)
    {
      // a is upper triangular.
      // first pass does the caxpyz on the diagonal
      caxpyz_recurse(a, x, y, z, range(0, x.size()), range(0, y.size()), 0, 1);
      // second pass does caxpy on the off diagonals
      caxpyz_recurse(a, x, y, z, range(0, x.size()), range(0, y.size()), 1, 1);
    }

    void caxpyz_L(const Complex *a, std::vector<ColorSpinorField*> &x, std::vector<ColorSpinorField*> &y, std::vector<ColorSpinorField*> &z)
    {
      // a is upper triangular.
      // first pass does the caxpyz on the diagonal
      caxpyz_recurse(a, x, y, z, range(0, x.size()), range(0, y.size()), 0, -1);
      // second pass does caxpy on the off diagonals
      caxpyz_recurse(a, x, y, z, range(0, x.size()), range(0, y.size()), 1, -1);
    }


    void caxpyz(const Complex *a, ColorSpinorField &x, ColorSpinorField &y, ColorSpinorField &z)
    {
      caxpyz(a, x.Components(), y.Components(), z.Components());
    }

    void caxpyz_U(const Complex *a, ColorSpinorField &x, ColorSpinorField &y, ColorSpinorField &z)
    {
      caxpyz_U(a, x.Components(), y.Components(), z.Components());
    }

    void caxpyz_L(const Complex *a, ColorSpinorField &x, ColorSpinorField &y, ColorSpinorField &z)
    {
      caxpyz_L(a, x.Components(), y.Components(), z.Components());
    }

    void axpyBzpcx(const double *a_, std::vector<ColorSpinorField *> &x_, std::vector<ColorSpinorField *> &y_,
                   const double *b_, ColorSpinorField &z_, const double *c_)
    {
      if (y_.size() <= (size_t)max_N_multi_1d()) {
        // swizzle order since we are writing to x_ and y_, but the
	// multi-blas only allow writing to y and w, and moreover the
	// block width of y and w must match, and x and z must match.
	std::vector<ColorSpinorField*> &y = y_;
	std::vector<ColorSpinorField*> &w = x_;

	// wrap a container around the third solo vector
	std::vector<ColorSpinorField*> x;
	x.push_back(&z_);

        coeff_array<double> a(a_), b(b_), c(c_);
        constexpr bool mixed = true;
        instantiate<multi_axpyBzpcx_, MultiBlas, mixed>(a, b, c, *x[0], *y[0], x, y, x, w);
      } else {
        // split the problem in half and recurse
	const double *a0 = &a_[0];
	const double *b0 = &b_[0];
	const double *c0 = &c_[0];

	std::vector<ColorSpinorField*> x0(x_.begin(), x_.begin() + x_.size()/2);
	std::vector<ColorSpinorField*> y0(y_.begin(), y_.begin() + y_.size()/2);

	axpyBzpcx(a0, x0, y0, b0, z_, c0);

	const double *a1 = &a_[y_.size()/2];
	const double *b1 = &b_[y_.size()/2];
	const double *c1 = &c_[y_.size()/2];

	std::vector<ColorSpinorField*> x1(x_.begin() + x_.size()/2, x_.end());
	std::vector<ColorSpinorField*> y1(y_.begin() + y_.size()/2, y_.end());

	axpyBzpcx(a1, x1, y1, b1, z_, c1);
      }
    }

    void caxpyBxpz(const Complex *a_, std::vector<ColorSpinorField*> &x_, ColorSpinorField &y_,
		   const Complex *b_, ColorSpinorField &z_)
    {
      if (x_.size() <= (size_t)max_N_multi_1d() &&
          is_valid_NXZ(x_.size(), false, x_[0]->Precision() < QUDA_SINGLE_PRECISION)) // only split if we have to.
      {
        // swizzle order since we are writing to y_ and z_, but the
        // multi-blas only allow writing to y and w, and moreover the
        // block width of y and w must match, and x and z must match.
        // Also, wrap a container around them.
        std::vector<ColorSpinorField*> y;
        y.push_back(&y_);
        std::vector<ColorSpinorField*> w;
        w.push_back(&z_);

        // we're reading from x
        std::vector<ColorSpinorField*> &x = x_;

        coeff_array<Complex> a(a_), b(b_), c;
        constexpr bool mixed = true;
        instantiate<multi_caxpyBxpz_, MultiBlas, mixed>(a, b, c, *x[0], *y[0], x, y, x, w);
      } else {
        // split the problem in half and recurse
        const Complex *a0 = &a_[0];
        const Complex *b0 = &b_[0];

        std::vector<ColorSpinorField*> x0(x_.begin(), x_.begin() + x_.size()/2);

        caxpyBxpz(a0, x0, y_, b0, z_);

        const Complex *a1 = &a_[x_.size()/2];
        const Complex *b1 = &b_[x_.size()/2];

        std::vector<ColorSpinorField*> x1(x_.begin() + x_.size()/2, x_.end());

        caxpyBxpz(a1, x1, y_, b1, z_);
      }
    }

    void axpy(const double *a_, std::vector<ColorSpinorField*> &x, std::vector<ColorSpinorField*> &y)
    {
      // Enter a recursion.
      // Pass a, x, y. (0,0) indexes the tiles. false specifies the matrix is unstructured.
      axpy_recurse<multiaxpy_>(a_, x, y, range(0, x.size()), range(0, y.size()), 0, 1);
    }

    void axpy_U(const double *a_, std::vector<ColorSpinorField*> &x, std::vector<ColorSpinorField*> &y)
    {
      // Enter a recursion.
      // Pass a, x, y. (0,0) indexes the tiles. 1 indicates the matrix is upper-triangular,
      //                                         which lets us skip some tiles.
      if (x.size() != y.size())
      {
        errorQuda("An optimal block axpy_U with non-square 'a' has not yet been implemented. Use block axpy instead");
      }
      axpy_recurse<multiaxpy_>(a_, x, y, range(0, x.size()), range(0, y.size()), 1, 1);
    }

    void axpy_L(const double *a_, std::vector<ColorSpinorField*> &x, std::vector<ColorSpinorField*> &y)
    {
      // Enter a recursion.
      // Pass a, x, y. (0,0) indexes the tiles. -1 indicates the matrix is lower-triangular
      //                                         which lets us skip some tiles.
      if (x.size() != y.size())
      {
        errorQuda("An optimal block axpy_L with non-square 'a' has not yet been implemented. Use block axpy instead");
      }
      axpy_recurse<multiaxpy_>(a_, x, y, range(0, x.size()), range(0, y.size()), -1, 1);
    }

    // Composite field version
    void axpy(const double *a, ColorSpinorField &x, ColorSpinorField &y) { axpy(a, x.Components(), y.Components()); }

    void axpy_U(const double *a, ColorSpinorField &x, ColorSpinorField &y) { axpy_U(a, x.Components(), y.Components()); }

    void axpy_L(const double *a, ColorSpinorField &x, ColorSpinorField &y) { axpy_L(a, x.Components(), y.Components()); }

  } // namespace blas

} // namespace quda
