#include "hip/hip_runtime.h"
#include <quda_internal.h>
#include <quda_matrix.h>
#include <gauge_field.h>
#include <gauge_field_order.h>
#include <index_helper.cuh>
#include <color_spinor.h>
#include <color_spinor_field.h>
#include <color_spinor_field_order.h>
#include <tune_quda.h>
#include <mpi.h>

namespace quda {

  template <typename Float, int Ns, int Nc, QudaReconstructType gRecon>
  struct WuppertalSmearingArg {
    typedef typename colorspinor_mapper<Float,Ns,Nc>::type F;
    typedef typename gauge_mapper<Float,gRecon>::type G;

    F out;                // output vector field
    const F in;           // input vector field
    const G U;            // the gauge field
    const Float aW[4];    // alpha-Wuppertal parameter, can be different in each direction
    const Float bW;       // some general factor multiplying the diagonal term of the smearing function
    const int parity;     // only use this for single parity fields
    const int nParity;    // number of parities we're working on
    const int nFace;      // hard code to 1 for now
    const int dim[5];     // full lattice dimensions
    const int commDim[4]; // whether a given dimension is partitioned or not
    const int volumeCB;   // checkerboarded volume

  WuppertalSmearingArg(ColorSpinorField &out, const ColorSpinorField &in, int parity, const GaugeField &U, const Float *aW, const Float bW)
  : out(out), in(in), U(U), aW{aW[0],aW[1],aW[2],aW[3]}, bW(bW), parity(parity), nParity(in.SiteSubset()), nFace(1),
      dim{ (3-nParity) * in.X(0), in.X(1), in.X(2), in.X(3), 1 },
      commDim{comm_dim_partitioned(0), comm_dim_partitioned(1), comm_dim_partitioned(2), comm_dim_partitioned(3)},
      volumeCB(in.VolumeCB())
    {      
      if (in.FieldOrder() != QUDA_FLOAT2_FIELD_ORDER || !U.isNative())
        errorQuda("Unsupported field order colorspinor=%d gauge=%d combination\n", in.FieldOrder(), U.FieldOrder());
    }
  };

  /**
     Computes out = sum_mu U_mu(x)in(x+d) + U^\dagger_mu(x-d)in(x-d)
     @param[out] out The out result field
     @param[in] U The gauge field
     @param[in] in The input field
     @param[in] x_cb The checkerboarded site index
     @param[in] parity The site parity
  */
  template <typename Float, int Nc, typename Vector, typename Arg>
  __device__ __host__ inline void computeNeighborSum(Vector &out, Arg &arg, int x_cb, int parity) {

    typedef Matrix<complex<Float>,Nc> Link;
    const int their_spinor_parity = (arg.nParity == 2) ? 1-parity : 0;

    int coord[5];
    getCoords(coord, x_cb, arg.dim, parity);
    coord[4] = 0;

    //-C.K: for-loop runs over all directions now.
    //-If smearing is not desired in any direction(s) "dir",
    //-this is controlled by setting aW[dir] = 0
#pragma unroll
    for (int dir=0; dir<4; dir++) {

      if( fabs(arg.aW[dir]) < 1e-8 ) continue;  //-C.K: Skip this direction if aW[dir] is zero
      
      //Forward gather - compute fwd offset for vector fetch
      const int fwd_idx = linkIndexP1(coord, arg.dim, dir);

      if ( arg.commDim[dir] && (coord[dir] + arg.nFace >= arg.dim[dir]) ) {
        const int ghost_idx = ghostFaceIndex<1>(coord, arg.dim, dir, arg.nFace);

        const Link U = arg.U(dir, x_cb, parity);
	const Vector in = arg.in.Ghost(dir, 1, ghost_idx, their_spinor_parity);

        out += arg.aW[dir]*U * in;
      } else {
        const Link U = arg.U(dir, x_cb, parity);
	const Vector in = arg.in(fwd_idx, their_spinor_parity);

        out += arg.aW[dir]*U * in;
      }

      //Backward gather - compute back offset for spinor and gauge fetch
      const int back_idx = linkIndexM1(coord, arg.dim, dir);
      const int gauge_idx = back_idx;

      if ( arg.commDim[dir] && (coord[dir] - arg.nFace < 0) ) {
        const int ghost_idx = ghostFaceIndex<0>(coord, arg.dim, dir, arg.nFace);

        const Link U = arg.U.Ghost(dir, ghost_idx, 1-parity);
	const Vector in = arg.in.Ghost(dir, 0, ghost_idx, their_spinor_parity);

        out += arg.aW[dir]*conj(U) * in;
      } else {
        const Link U = arg.U(dir, gauge_idx, 1-parity);
	const Vector in = arg.in(back_idx, their_spinor_parity);

        out += arg.aW[dir]*conj(U) * in;
      }
      
    }//-dir for-loop
    
  } //-function closes

  // out(x) =   ( bW * in(x) + computeNeighborSum(out, x, aW[mu]) )
  template <typename Float, int Ns, int Nc, typename Arg>
  __device__ __host__ inline void computeWupperalStep(Arg &arg, int x_cb, int parity)
  {
    typedef ColorSpinor<Float,Nc,Ns> Vector;
    Vector out;

    computeNeighborSum<Float,Nc>(out, arg, x_cb, parity);

    Vector in = arg.in(x_cb, parity) ;
    out = arg.bW * in + out ;
    
    arg.out(x_cb, parity) = out;
  }

  // CPU kernel for applying a wuppertal smearing step to a vector
  template <typename Float, int Ns, int Nc, typename Arg>
  void wuppertalStepCPU(Arg arg)
  {

    for (int parity= 0; parity < arg.nParity; parity++) {
      // for full fields then set parity from loop else use arg setting
      parity = (arg.nParity == 2) ? parity : arg.parity;

      for (int x_cb = 0; x_cb < arg.volumeCB; x_cb++) { // 4-d volume
        computeWupperalStep<Float,Ns,Nc>(arg, x_cb, parity);
      } // 4-d volumeCB
    } // parity

  }

  // GPU Kernel for applying a wuppertal smearing step to a vector
  template <typename Float, int Ns, int Nc, typename Arg>
  __global__ void wuppertalStepGPU(Arg arg)
  {
    int x_cb = blockIdx.x*blockDim.x + threadIdx.x;

    // for full fields set parity from y thread index else use arg setting
    int parity = blockDim.y*blockIdx.y + threadIdx.y;

    if (x_cb >= arg.volumeCB) return;
    if (parity >= arg.nParity) return;
    parity = (arg.nParity == 2) ? parity : arg.parity;

    computeWupperalStep<Float,Ns,Nc>(arg, x_cb, parity);
  }

  template <typename Float, int Ns, int Nc, typename Arg>
  class WuppertalSmearing : public TunableVectorY {

  protected:
    Arg &arg;
    const ColorSpinorField &meta;

    long long flops() const
    {
      return (2*3*Ns*Nc*(8*Nc-2) + 2*3*Nc*Ns )*arg.nParity*(long long)meta.VolumeCB();
    }
    long long bytes() const
    {
      return arg.out.Bytes() + (2*3+1)*arg.in.Bytes() + arg.nParity*2*3*arg.U.Bytes()*meta.VolumeCB();
    }
    bool tuneGridDim() const { return false; }
    unsigned int minThreads() const { return arg.volumeCB; }

  public:
    WuppertalSmearing(Arg &arg, const ColorSpinorField &meta) : TunableVectorY(arg.nParity), arg(arg), meta(meta)
    {
      strcpy(aux, meta.AuxString());
      strcat(aux, comm_dim_partitioned_string());
    }
    virtual ~WuppertalSmearing() { }

    void apply(const hipStream_t &stream) {
      if (meta.Location() == QUDA_CPU_FIELD_LOCATION) {
        wuppertalStepCPU<Float,Ns,Nc>(arg);
      } else {
        TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());

        wuppertalStepGPU<Float,Ns,Nc> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg);
      }
    }

    TuneKey tuneKey() const { return TuneKey(meta.VolString(), typeid(*this).name(), aux); }
  };

  template<typename Float, int Ns, int Nc, QudaReconstructType gRecon>
  void wuppertalStep(ColorSpinorField &out, const ColorSpinorField &in, int parity,
		     const GaugeField& U, const double *aW, const double bW)
  {
    WuppertalSmearingArg<Float,Ns,Nc,gRecon> arg(out, in, parity, U, (Float*) aW, (Float) bW);
    WuppertalSmearing<Float,Ns,Nc,WuppertalSmearingArg<Float,Ns,Nc,gRecon> > wuppertal(arg, in);
    wuppertal.apply(0);
  }

  // template on the gauge reconstruction
  template<typename Float, int Ns, int Nc>
  void wuppertalStep(ColorSpinorField &out, const ColorSpinorField &in, int parity,
		     const GaugeField& U, const double *aW, const double bW)
  {
    if (U.Reconstruct() == QUDA_RECONSTRUCT_NO) {
      wuppertalStep<Float,Ns,Nc,QUDA_RECONSTRUCT_NO>(out, in, parity, U, aW, bW);
    } else if(U.Reconstruct() == QUDA_RECONSTRUCT_12) {
      wuppertalStep<Float,Ns,Nc,QUDA_RECONSTRUCT_12>(out, in, parity, U, aW, bW);
    } else if(U.Reconstruct() == QUDA_RECONSTRUCT_8) {
      wuppertalStep<Float,Ns,Nc,QUDA_RECONSTRUCT_8>(out, in, parity, U, aW, bW);
    } else {
      errorQuda("Reconstruction type %d of origin gauge field not supported", U.Reconstruct());
    }
  }


  // template on the number of colors
  template<typename Float, int Ns>
  void wuppertalStep(ColorSpinorField &out, const ColorSpinorField &in, int parity,
		     const GaugeField& U, const double *aW, const double bW)
  {
    if (out.Ncolor() != in.Ncolor()) {
      errorQuda("Orign and destination fields must have the same number of colors\n");
    }

    if (out.Ncolor() == 3 ) {
      wuppertalStep<Float,Ns,3>(out, in, parity, U, aW, bW);
    } else {
      errorQuda(" is not implemented for Ncolor!=3");
    }
  }

  // template on the number of spins
  template<typename Float>
  void wuppertalStep(ColorSpinorField &out, const ColorSpinorField &in, int parity,
		     const GaugeField& U, const double *aW, const double bW)
  {
    if(out.Nspin() != in.Nspin()) {
      errorQuda("Orign and destination fields must have the same number of spins\n");
    }

    if (out.Nspin() == 4 ){
      wuppertalStep<Float,4>(out, in, parity, U, aW, bW);
    }else if (in.Nspin() == 1 ){
      wuppertalStep<Float,1>(out, in, parity, U, aW, bW);
    }else{
      errorQuda("Nspin %d not supported", out.Nspin());
    }
  }


  // template on the precision
  /**
     Apply Wuppertal smearing step as
     out(x) = bW * in(x)  + \sum_mu alpha_\mu (U_{-\mu}(x)in(x+mu) + U^\dagger_mu(x-mu)in(x-mu)))
     @param[out] out The out result field
     @param[in] in The in spinor field
     @param[in] U The gauge field
     @param[in] alpha_\mu The smearing parameter, can be different in each direction \mu
     @param[in] bW A general factor that multiplies the local term. If bW -> (bW - 2 sum_\mu \alpha_\mu) then
     the hopping term operation generalizes to the full Laplacian operator.
  */
  void wuppertalStep(ColorSpinorField &out, const ColorSpinorField &in, int parity,
		     const GaugeField& U, const double *aW, const double bW)
  {
    if (in.V() == out.V()) {
      errorQuda("Orign and destination fields must be different pointers");
    }

    // check precisions match
    checkPrecision(out, in, U);

    // check all locations match
    checkLocation(out, in, U);

    const int nFace = 1;
    in.exchangeGhost((QudaParity)(1-parity), nFace, 0); // last parameter is dummy

    if (out.Precision() == QUDA_SINGLE_PRECISION){
      wuppertalStep<float>(out, in, parity, U, aW, bW);
    } else if(out.Precision() == QUDA_DOUBLE_PRECISION) {
      wuppertalStep<double>(out, in, parity, U, aW, bW);
    } else {
      errorQuda("Precision %d not supported", out.Precision());
    }

    in.bufferIndex = (1 - in.bufferIndex);
    return;
  }

} // namespace quda
