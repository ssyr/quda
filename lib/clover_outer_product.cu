#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>

#include <tune_quda.h>
#include <gauge_field_order.h>
#include <color_spinor_field_order.h>
#include <quda_matrix.h>
#include <color_spinor.h>
#include <dslash_quda.h>

namespace quda {

  enum OprodKernelType { OPROD_INTERIOR_KERNEL, OPROD_EXTERIOR_KERNEL };

  template<typename Float, int nColor_, QudaReconstructType recon>
  struct CloverForceArg {
    typedef typename mapper<Float>::type real;
    static constexpr int nColor = nColor_;
    static constexpr int nSpin = 4;
    static constexpr int spin_project = true;
    using F = typename colorspinor_mapper<Float, nSpin, nColor, spin_project>::type;
    using Gauge = typename gauge_mapper<Float, recon, 18>::type;
    using Force = typename gauge_mapper<Float, QUDA_RECONSTRUCT_NO, 18>::type;

    const F inA;
    const F inB;
    const F inC;
    const F inD;
    Gauge  gauge;
    Force force;
    unsigned int length;
    int X[4];
    unsigned int parity;
    unsigned int dir;
    unsigned int displacement;
    OprodKernelType kernelType;
    bool partitioned[4];
    Float coeff;

    CloverForceArg(GaugeField &force, const GaugeField &gauge, const ColorSpinorField &inA, const ColorSpinorField &inB,
                   const ColorSpinorField &inC, const ColorSpinorField &inD, const unsigned int parity, const double coeff) :
      inA(inA),
      inB(inB),
      inC(inC),
      inD(inD),
      gauge(gauge),
      force(force),
      length(gauge.VolumeCB()),
      parity(parity),
      dir(5),
      displacement(1),
      kernelType(OPROD_INTERIOR_KERNEL),
      coeff(coeff)
    {
      for (int i=0; i<4; ++i) this->X[i] = gauge.X()[i];
      for (int i=0; i<4; ++i) this->partitioned[i] = commDimPartitioned(i) ? true : false;
    }
  };

  template<typename real, typename Arg> __global__ void interiorOprodKernel(Arg arg)
  {
    typedef complex<real> Complex;
    int idx = blockIdx.x*blockDim.x + threadIdx.x;

    ColorSpinor<real, Arg::nColor, 4> A, B_shift, C, D_shift;
    Matrix<Complex, Arg::nColor> U, result, temp;

    while (idx<arg.length) {
      A = arg.inA(idx, 0);
      C = arg.inC(idx, 0);

#pragma unroll
      for (int dim=0; dim<4; ++dim) {
	int shift[4] = {0,0,0,0};
	shift[dim] = 1;
	const int nbr_idx = neighborIndex(idx, shift, arg.partitioned, arg.parity, arg.X);

	if (nbr_idx >= 0) {
	  B_shift = arg.inB(nbr_idx, 0);
	  D_shift = arg.inD(nbr_idx, 0);

	  B_shift = (B_shift.project(dim,1)).reconstruct(dim,1);
	  result = outerProdSpinTrace(B_shift,A);

	  D_shift = (D_shift.project(dim,-1)).reconstruct(dim,-1);
	  result += outerProdSpinTrace(D_shift,C);

	  temp = arg.force(dim, idx, arg.parity);
	  U = arg.gauge(dim, idx, arg.parity);
	  result = temp + U*result*arg.coeff;
	  arg.force(dim, idx, arg.parity) = result;
	}
      } // dim

      idx += gridDim.x*blockDim.x;
    }
  } // interiorOprodKernel

  template<int dim, typename real, typename Arg> __global__ void exteriorOprodKernel(Arg arg)
  {
    typedef complex<real> Complex;
    int cb_idx = blockIdx.x*blockDim.x + threadIdx.x;

    ColorSpinor<real, Arg::nColor, 4> A, B_shift, C, D_shift;
    ColorSpinor<real, Arg::nColor, 2> projected_tmp;
    Matrix<Complex, Arg::nColor> U, result, temp;

    int x[4];
    while (cb_idx<arg.length) {
      coordsFromIndexExterior(x, cb_idx, arg.X, dim, arg.displacement, arg.parity);
      const unsigned int bulk_cb_idx = ((((x[3]*arg.X[2] + x[2])*arg.X[1] + x[1])*arg.X[0] + x[0]) >> 1);
      A = arg.inA(bulk_cb_idx, 0);
      C = arg.inC(bulk_cb_idx, 0);

      projected_tmp = arg.inB.Ghost(dim, 1, cb_idx, 0);
      B_shift = projected_tmp.reconstruct(dim, 1);
      result = outerProdSpinTrace(B_shift,A);

      projected_tmp = arg.inD.Ghost(dim, 1, cb_idx, 0);
      D_shift = projected_tmp.reconstruct(dim,-1);
      result += outerProdSpinTrace(D_shift,C);

      temp = arg.force(dim, bulk_cb_idx, arg.parity);
      U = arg.gauge(dim, bulk_cb_idx, arg.parity);
      result = temp + U*result*arg.coeff;
      arg.force(dim, bulk_cb_idx, arg.parity) = result;

      cb_idx += gridDim.x*blockDim.x;
    }
  } // exteriorOprodKernel

  template<typename Float, typename Arg>
  class CloverForce : public Tunable {
    Arg &arg;
    const GaugeField &meta;

    unsigned int sharedBytesPerThread() const { return 0; }
    unsigned int sharedBytesPerBlock(const TuneParam &) const { return 0; }

    unsigned int minThreads() const { return arg.length; }
    bool tuneGridDim() const { return false; }

  public:
    CloverForce(Arg &arg, GaugeField &meta) :
      arg(arg), meta(meta) {
      writeAuxString(meta.AuxString());
      // this sets the communications pattern for the packing kernel
      int comms[QUDA_MAX_DIM] = { commDimPartitioned(0), commDimPartitioned(1), commDimPartitioned(2), commDimPartitioned(3) };
      setPackComms(comms);
    }

    void apply(const qudaStream_t &stream) {
      if (meta.Location() == QUDA_CUDA_FIELD_LOCATION) {
	// Disable tuning for the time being
	TuneParam tp = tuneLaunch(*this,getTuning(),getVerbosity());

	if (arg.kernelType == OPROD_INTERIOR_KERNEL) {
	  qudaLaunchKernel(interiorOprodKernel<Float, Arg>, tp, stream, arg);
        } else if (arg.kernelType == OPROD_EXTERIOR_KERNEL) {
          if (arg.dir == 0)      qudaLaunchKernel(exteriorOprodKernel<0,Float,Arg>, tp, stream, arg);
	  else if (arg.dir == 1) qudaLaunchKernel(exteriorOprodKernel<1,Float,Arg>, tp, stream, arg);
	  else if (arg.dir == 2) qudaLaunchKernel(exteriorOprodKernel<2,Float,Arg>, tp, stream, arg);
          else if (arg.dir == 3) qudaLaunchKernel(exteriorOprodKernel<3,Float,Arg>, tp, stream, arg);
        } else {
          errorQuda("Kernel type not supported\n");
        }
      }else{ // run the CPU code
	errorQuda("No CPU support for staggered outer-product calculation\n");
      }
    } // apply

    void preTune() {
      this->arg.force.save();
    }
    void postTune() {
      this->arg.force.load();
    }

    long long flops() const {
      if (arg.kernelType == OPROD_INTERIOR_KERNEL) {
	return ((long long)arg.length)*4*(24 + 144 + 234); // spin project + spin trace + multiply-add
      } else {
	return ((long long)arg.length)*(144 + 234); // spin trace + multiply-add
      }
    }
    long long bytes() const {
      if (arg.kernelType == OPROD_INTERIOR_KERNEL) {
	return arg.length*(arg.inA.Bytes() + arg.inC.Bytes() + 4*(arg.inB.Bytes() + arg.inD.Bytes() + 2*arg.force.Bytes() + arg.gauge.Bytes()));
      } else {
	return arg.length*(arg.inA.Bytes() + arg.inB.Bytes()/2 + arg.inC.Bytes() + arg.inD.Bytes()/2 + 2*arg.force.Bytes() + arg.gauge.Bytes());
      }
    }

    TuneKey tuneKey() const {
      char new_aux[TuneKey::aux_n];
      strcpy(new_aux, aux);
      if (arg.kernelType == OPROD_INTERIOR_KERNEL) {
	strcat(new_aux, ",interior");
      } else {
	strcat(new_aux, ",exterior");
	if (arg.dir==0) strcat(new_aux, ",dir=0");
	else if (arg.dir==1) strcat(new_aux, ",dir=1");
	else if (arg.dir==2) strcat(new_aux, ",dir=2");
	else if (arg.dir==3) strcat(new_aux, ",dir=3");
      }
      return TuneKey(meta.VolString(), "CloverForce", new_aux);
    }
  }; // CloverForce

  void exchangeGhost(cudaColorSpinorField &a, int parity, int dag) {
    // need to enable packing in temporal direction to get spin-projector correct
    pushKernelPackT(true);

    // first transfer src1
    qudaDeviceSynchronize();

    MemoryLocation location[2*QUDA_MAX_DIM] = {Device, Device, Device, Device, Device, Device, Device, Device};
    a.pack(1, 1-parity, dag, Nstream-1, location, Device);

    qudaDeviceSynchronize();

    for (int i=3; i>=0; i--) {
      if (commDimPartitioned(i)) {
	// Initialize the host transfer from the source spinor
	a.gather(1, dag, 2*i);
      } // commDim(i)
    } // i=3,..,0

    qudaDeviceSynchronize(); comm_barrier();

    for (int i=3; i>=0; i--) {
      if (commDimPartitioned(i)) {
	a.commsStart(1, 2*i, dag);
      }
    }

    for (int i=3; i>=0; i--) {
      if (commDimPartitioned(i)) {
	a.commsWait(1, 2*i, dag);
	a.scatter(1, dag, 2*i);
      }
    }

    qudaDeviceSynchronize();
    popKernelPackT(); // restore packing state

    a.bufferIndex = (1 - a.bufferIndex);
    comm_barrier();
  }

  template <typename Float, QudaReconstructType recon>
  void computeCloverForce(GaugeField &force, const GaugeField &gauge, const ColorSpinorField& inA, const ColorSpinorField& inB,
                          const ColorSpinorField& inC, const ColorSpinorField& inD, int parity, const double coeff)
  {
    // Create the arguments for the interior kernel
    CloverForceArg<Float, 3, recon> arg(force, gauge, inA, inB, inC, inD, parity, coeff);
    CloverForce<Float,decltype(arg)> oprod(arg, force);

    arg.kernelType = OPROD_INTERIOR_KERNEL;
    arg.length = inA.VolumeCB();
    oprod.apply(0);

    for (int i=3; i>=0; i--) {
      if (commDimPartitioned(i)) {
        // update parameters for this exterior kernel
        arg.kernelType = OPROD_EXTERIOR_KERNEL;
        arg.dir = i;
        arg.length = inA.GhostFaceCB()[i];
        arg.displacement = 1; // forwards displacement
        oprod.apply(0);
      }
    } // i=3,..,0
  } // computeCloverForce

  void computeCloverForce(GaugeField &force, const GaugeField &U, std::vector<ColorSpinorField *> &x,
                          std::vector<ColorSpinorField *> &p, std::vector<double> &coeff)
  {
#ifdef GPU_CLOVER_DIRAC
    if (force.Order() != QUDA_FLOAT2_GAUGE_ORDER) errorQuda("Unsupported output ordering: %d\n", force.Order());
    checkPrecision(*x[0], *p[0], force, U);

    int dag = 1;

    for (unsigned int i=0; i<x.size(); i++) {
      static_cast<cudaColorSpinorField&>(x[i]->Even()).allocateGhostBuffer(1);
      static_cast<cudaColorSpinorField&>(x[i]->Odd()).allocateGhostBuffer(1);
      static_cast<cudaColorSpinorField&>(p[i]->Even()).allocateGhostBuffer(1);
      static_cast<cudaColorSpinorField&>(p[i]->Odd()).allocateGhostBuffer(1);

      for (int parity=0; parity<2; parity++) {

	ColorSpinorField& inA = (parity&1) ? p[i]->Odd() : p[i]->Even();
	ColorSpinorField& inB = (parity&1) ? x[i]->Even(): x[i]->Odd();
	ColorSpinorField& inC = (parity&1) ? x[i]->Odd() : x[i]->Even();
	ColorSpinorField& inD = (parity&1) ? p[i]->Even(): p[i]->Odd();

	if (x[0]->Precision() == QUDA_DOUBLE_PRECISION) {
          exchangeGhost(static_cast<cudaColorSpinorField&>(inB), parity, dag);
          exchangeGhost(static_cast<cudaColorSpinorField&>(inD), parity, 1-dag);

	  if (U.Reconstruct() == QUDA_RECONSTRUCT_NO) {
	    computeCloverForce<double, QUDA_RECONSTRUCT_NO>(force, U, inA, inB, inC, inD, parity, coeff[i]);
	  } else if (U.Reconstruct() == QUDA_RECONSTRUCT_12) {
	    computeCloverForce<double, QUDA_RECONSTRUCT_12>(force, U, inA, inB, inC, inD, parity, coeff[i]);
	  } else {
	    errorQuda("Unsupported recontruction type");
	  }
	} else {
	  errorQuda("Unsupported precision: %d\n", x[0]->Precision());
	}
      }
    }
#else // GPU_CLOVER_DIRAC not defined
   errorQuda("Clover Dirac operator has not been built!");
#endif

  } // computeCloverForce

} // namespace quda
