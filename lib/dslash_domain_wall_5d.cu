#include <gauge_field.h>
#include <color_spinor_field.h>
#include <dslash.h>
#include <worker.h>

#include <dslash_policy.cuh>
#include <kernels/dslash_domain_wall_5d.cuh>

/**
   This is the gauged domain-wall 5-d preconditioned operator.
*/

namespace quda
{

  /**
     @brief This is a helper class that is used to instantiate the
     correct templated kernel for the dslash.
   */
  template <typename Float, int nDim, int nColor, int nParity, bool dagger, bool xpay, KernelType kernel_type, typename Arg>
  struct DomainWall5DLaunch {
    static constexpr const char *kernel = "quda::domainWall5DGPU"; // kernel name for jit compilation
    template <typename Dslash>
    inline static void launch(Dslash &dslash, TuneParam &tp, Arg &arg, const hipStream_t &stream)
    {
      dslash.launch(domainWall5DGPU<Float, nDim, nColor, nParity, dagger, xpay, kernel_type, Arg>, tp, arg, stream);
    }
  };

  template <typename Float, int nDim, int nColor, typename Arg> class DomainWall5D : public Dslash<Float>
  {

protected:
    Arg &arg;
    const ColorSpinorField &in;

public:
    DomainWall5D(Arg &arg, const ColorSpinorField &out, const ColorSpinorField &in) :
        Dslash<Float>(arg, out, in, "kernels/dslash_domain_wall_5d.cuh"),
        arg(arg),
        in(in)
    {
      TunableVectorYZ::resizeVector(in.X(4), arg.nParity);
    }

    virtual ~DomainWall5D() {}

    void apply(const hipStream_t &stream)
    {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      Dslash<Float>::setParam(arg);
      Dslash<Float>::template instantiate<DomainWall5DLaunch, nDim, nColor>(tp, arg, stream);
    }

    long long flops() const
    {
      long long flops = Dslash<Float>::flops();
      switch (arg.kernel_type) {
      case EXTERIOR_KERNEL_X:
      case EXTERIOR_KERNEL_Y:
      case EXTERIOR_KERNEL_Z:
      case EXTERIOR_KERNEL_T:
      case EXTERIOR_KERNEL_ALL: break; // 5-d flops are in the interior kernel
      case INTERIOR_KERNEL:
      case KERNEL_POLICY:
        int Ls = in.X(4);
        long long bulk = (Ls - 2) * (in.Volume() / Ls);
        long long wall = 2 * (in.Volume() / Ls);
        flops += 96ll * bulk + 120ll * wall;
        break;
      }
      return flops;
    }

    long long bytes() const
    {
      bool isFixed = (in.Precision() == sizeof(short) || in.Precision() == sizeof(char)) ? true : false;
      int spinor_bytes = 2 * in.Ncolor() * in.Nspin() * in.Precision() + (isFixed ? sizeof(float) : 0);
      long long bytes = Dslash<Float>::bytes();
      switch (arg.kernel_type) {
      case EXTERIOR_KERNEL_X:
      case EXTERIOR_KERNEL_Y:
      case EXTERIOR_KERNEL_Z:
      case EXTERIOR_KERNEL_T:
      case EXTERIOR_KERNEL_ALL: break;
      case INTERIOR_KERNEL:
      case KERNEL_POLICY: bytes += 2 * spinor_bytes * in.VolumeCB(); break;
      }
      return bytes;
    }

    TuneKey tuneKey() const
    {
      return TuneKey(in.VolString(), typeid(*this).name(), Dslash<Float>::aux[arg.kernel_type]);
    }
  };

  template <typename Float, int nColor, QudaReconstructType recon> struct DomainWall5DApply {

    inline DomainWall5DApply(ColorSpinorField &out, const ColorSpinorField &in, const GaugeField &U, double a,
        double m_f, const ColorSpinorField &x, int parity, bool dagger, const int *comm_override, TimeProfile &profile)
    {
      constexpr int nDim = 5;
      DomainWall5DArg<Float, nColor, recon> arg(out, in, U, a, m_f, a != 0.0, x, parity, dagger, comm_override);
      DomainWall5D<Float, nDim, nColor, DomainWall5DArg<Float, nColor, recon>> twisted(arg, out, in);

      dslash::DslashPolicyTune<decltype(twisted)> policy(twisted,
          const_cast<cudaColorSpinorField *>(static_cast<const cudaColorSpinorField *>(&in)),
          in.getDslashConstant().volume_4d_cb, in.getDslashConstant().ghostFaceCB, profile);
      policy.apply(0);

      checkCudaError();
    }
  };

  // Apply the 4-d preconditioned domain-wall Dslash operator
  // out(x) = M*in = in(x) + a*\sum_mu U_{-\mu}(x)in(x+mu) + U^\dagger_mu(x-mu)in(x-mu)
  void ApplyDomainWall5D(ColorSpinorField &out, const ColorSpinorField &in, const GaugeField &U, double a, double m_f,
      const ColorSpinorField &x, int parity, bool dagger, const int *comm_override, TimeProfile &profile)
  {
#ifdef GPU_DOMAIN_WALL_DIRAC
    if (in.V() == out.V()) errorQuda("Aliasing pointers");
    if (in.FieldOrder() != out.FieldOrder())
      errorQuda("Field order mismatch in = %d, out = %d", in.FieldOrder(), out.FieldOrder());

    // check all precisions match
    checkPrecision(out, in, x, U);

    // check all locations match
    checkLocation(out, in, x, U);

    // with 5-d checkerboarding we must use kernel packing
    pushKernelPackT(true);

    instantiate<DomainWall5DApply>(out, in, U, a, m_f, x, parity, dagger, comm_override, profile);

    popKernelPackT();
#else
    errorQuda("Domain-wall dslash has not been built");
#endif // GPU_DOMAIN_WALL_DIRAC
  }

} // namespace quda
